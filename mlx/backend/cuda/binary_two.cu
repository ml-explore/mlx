#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/binary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/binary_ops.cuh"
#include "mlx/backend/cuda/device/cucomplex_math.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void
binary_ss(const In* a, const In* b, Out* out_a, Out* out_b, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto out = Op{}(a[0], b[0]);
    out_a[0] = out[0];
    out_b[0] = out[1];
  }
}

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void
binary_sv(const In* a, const In* b, Out* out_a, Out* out_b, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto out = Op{}(a[0], b[index]);
    out_a[index] = out[0];
    out_b[index] = out[1];
  }
}

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void
binary_vs(const In* a, const In* b, Out* out_a, Out* out_b, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto out = Op{}(a[index], b[0]);
    out_a[index] = out[0];
    out_b[index] = out[1];
  }
}

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void
binary_vv(const In* a, const In* b, Out* out_a, Out* out_b, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto out = Op{}(a[index], b[index]);
    out_a[index] = out[0];
    out_b[index] = out[1];
  }
}

template <typename Op, typename In, typename Out, typename IdxT, int NDIM>
__global__ void binary_g_nd(
    const In* a,
    const In* b,
    Out* out_a,
    Out* out_b,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> a_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> b_strides) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx] = elem_to_loc_nd<NDIM>(
        index, shape.data(), a_strides.data(), b_strides.data());
    auto out = Op{}(a[a_idx], b[b_idx]);
    out_a[index] = out[0];
    out_b[index] = out[1];
  }
}

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void binary_g(
    const In* a,
    const In* b,
    Out* out_a,
    Out* out_b,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides a_strides,
    const __grid_constant__ Strides b_strides,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx] = elem_to_loc_4d(
        index, shape.data(), a_strides.data(), b_strides.data(), ndim);
    auto out = Op{}(a[a_idx], b[b_idx]);
    out_a[index] = out[0];
    out_b[index] = out[1];
  }
}

template <typename Op, typename In, typename Out>
constexpr bool supports_binary_op() {
  if (std::is_same_v<Op, DivMod>) {
    return std::is_same_v<In, Out> &&
        (std::is_integral_v<Out> || is_floating_v<Out>);
  }
  return false;
}

} // namespace cu

template <typename Op>
void binary_op_gpu_inplace(
    const std::vector<array>& inputs,
    std::vector<array>& outputs,
    std::string_view op,
    const Stream& s) {
  assert(inputs.size() > 1);
  const auto& a = inputs[0];
  const auto& b = inputs[1];
  auto& out_a = outputs[0];
  auto& out_b = outputs[1];
  auto bopt = get_binary_op_type(a, b);
  set_binary_op_output_data(a, b, out_a, bopt);
  set_binary_op_output_data(a, b, out_b, bopt);

  if (out_a.size() == 0) {
    return;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_output_array(out_a);
  encoder.set_output_array(out_b);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(a.dtype(), CTYPE_IN, {
      MLX_SWITCH_ALL_TYPES(out_a.dtype(), CTYPE_OUT, {
        if constexpr (cu::supports_binary_op<Op, CTYPE_IN, CTYPE_OUT>()) {
          using InType = cuda_type_t<CTYPE_IN>;
          using OutType = cuda_type_t<CTYPE_OUT>;

          auto bopt = get_binary_op_type(a, b);
          if (bopt == BinaryOpType::General) {
            auto [shape, strides] = collapse_contiguous_dims(a, b, out_a);
            auto& a_strides = strides[0];
            auto& b_strides = strides[1];
            bool large = a.data_size() > INT32_MAX ||
                b.data_size() > INT32_MAX || out_a.data_size() > INT32_MAX;
            MLX_SWITCH_BOOL(large, LARGE, {
              using IdxT = std::conditional_t<LARGE, int64_t, int32_t>;
              int ndim = shape.size();
              if (ndim <= 3) {
                MLX_SWITCH_1_2_3(ndim, NDIM, {
                  auto kernel =
                      &cu::binary_g_nd<Op, InType, OutType, IdxT, NDIM>;
                  auto [num_blocks, block_dims] =
                      get_launch_args(kernel, out_a, large);
                  kernel<<<num_blocks, block_dims, 0, stream>>>(
                      a.data<InType>(),
                      b.data<InType>(),
                      out_a.data<OutType>(),
                      out_b.data<OutType>(),
                      out_a.size(),
                      const_param<NDIM>(shape),
                      const_param<NDIM>(a_strides),
                      const_param<NDIM>(b_strides));
                });
              } else {
                auto kernel = cu::binary_g<Op, InType, OutType, IdxT>;
                auto [num_blocks, block_dims] =
                    get_launch_args(kernel, out_a, large);
                kernel<<<num_blocks, block_dims, 0, stream>>>(
                    a.data<InType>(),
                    b.data<InType>(),
                    out_a.data<OutType>(),
                    out_b.data<OutType>(),
                    out_a.size(),
                    const_param(shape),
                    const_param(a_strides),
                    const_param(b_strides),
                    ndim);
              }
            });
          } else {
            MLX_SWITCH_BOOL(out_a.data_size() > UINT32_MAX, LARGE, {
              using IdxT = std::conditional_t<LARGE, int64_t, uint32_t>;
              auto kernel = cu::binary_ss<Op, InType, OutType, IdxT>;
              if (bopt == BinaryOpType::ScalarVector) {
                kernel = cu::binary_sv<Op, InType, OutType, IdxT>;
              } else if (bopt == BinaryOpType::VectorScalar) {
                kernel = cu::binary_vs<Op, InType, OutType, IdxT>;
              } else if (bopt == BinaryOpType::VectorVector) {
                kernel = cu::binary_vv<Op, InType, OutType, IdxT>;
              }
              auto [num_blocks, block_dims] = get_launch_args(
                  kernel,
                  out_a.data_size(),
                  out_a.shape(),
                  out_a.strides(),
                  LARGE);
              kernel<<<num_blocks, block_dims, 0, stream>>>(
                  a.data<InType>(),
                  b.data<InType>(),
                  out_a.data<OutType>(),
                  out_b.data<OutType>(),
                  out_a.data_size());
            });
          }
        } else {
          throw std::runtime_error(fmt::format(
              "Can not do binary op {} on inputs of {} with result of {}.",
              op,
              dtype_to_string(a.dtype()),
              dtype_to_string(out_a.dtype())));
        }
      });
    });
  });
}

template <typename Op>
void binary_op_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs,
    std::string_view op,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto bopt = get_binary_op_type(a, b);
  set_binary_op_output_data(a, b, outputs[0], bopt);
  set_binary_op_output_data(a, b, outputs[1], bopt);
  binary_op_gpu_inplace<Op>(inputs, outputs, op, s);
}

void DivMod::eval_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs) {
  nvtx3::scoped_range r("DivMod::eval_gpu");
  auto& s = outputs[0].primitive().stream();
  binary_op_gpu<cu::DivMod>(inputs, outputs, get_primitive_string(this), s);
}

} // namespace mlx::core
