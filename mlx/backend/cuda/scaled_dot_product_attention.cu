#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/config.h"
#include "mlx/backend/cuda/device/utils.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/lru_cache.h"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"
#include "mlx/transforms_impl.h"

#include <nvtx3/nvtx3.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

#define PRAGMA_LOOP_UNROLL #pragma unroll

struct AttnParams {
  int B;
  int H;
  int D;

  int qL;
  int kL;

  int gqa_factor;
  float scale;

  int64_t Q_strides[3];
  int64_t K_strides[3];
  int64_t V_strides[3];
  int64_t O_strides[3];
};

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_1pass(
    const T* Q,
    const T* K,
    const T* V,
    T* O,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 32;
  constexpr int BD = 32;

  constexpr int v_per_thread = D / BD;

  const int inner_k_stride = BN * int(params.K_strides[2]);
  const int inner_v_stride = BN * int(params.V_strides[2]);

  typedef float U;

  U q[v_per_thread];
  U k[v_per_thread];
  U o[v_per_thread];

  __shared__ U outputs[BN][BD + 1];
  __shared__ U max_scores[BN];
  __shared__ U sum_exp_scores[BN];

  const U scale_log2 = params.scale * 1.44269504089f;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z;
  const int head_idx = blockIdx.x;
  const int kv_head_idx = head_idx / params.gqa_factor;

  const int q_seq_idx = blockIdx.y;
  const int kv_seq_idx = warp_idx;

  Q += batch_idx * params.Q_strides[0] + // Batch
      head_idx * params.Q_strides[1] + // Head
      q_seq_idx * params.Q_strides[2]; // Sequence

  K += batch_idx * params.K_strides[0] + // Batch
      kv_head_idx * params.K_strides[1] + // Head
      kv_seq_idx * params.K_strides[2]; // Sequence

  V += batch_idx * params.V_strides[0] + // Batch
      kv_head_idx * params.V_strides[1] + // Head
      kv_seq_idx * params.V_strides[2]; // Sequence

  O += batch_idx * params.O_strides[0] + // Batch
      head_idx * params.O_strides[1] + // Head
      q_seq_idx * params.O_strides[2]; // Sequence

  // Read the query and 0 the output accumulator
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    q[i] = scale_log2 * static_cast<U>(Q[v_per_thread * lane_idx + i]);
  }

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = 0.f;
  }

  U max_score = -INFINITY;
  U sum_exp_score = 0.f;

  // For each key
  for (int i = kv_seq_idx; i < params.kL; i += BN) {
    bool use_key = true;
    if constexpr (do_causal) {
      use_key = i <= (params.kL - params.qL + q_seq_idx);
    }

    if (use_key) {
      // Read the key
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        k[j] = K[v_per_thread * lane_idx + j];
      }

      // Compute the i-th score
      U score = 0.f;
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        score += q[j] * k[j];
      }

      // Warp sum
      score = cg::reduce(warp, score, cg::plus<U>());

      // Update the accumulators
      U new_max = max(max_score, score);
      U factor = exp2f(max_score - new_max);
      U exp_score = exp2f(score - new_max);

      max_score = new_max;
      sum_exp_score = sum_exp_score * factor + exp_score;

      // Update the output accumulator
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        o[j] = o[j] * factor +
            exp_score * static_cast<U>(V[v_per_thread * lane_idx + j]);
      }
    }

    // Move the pointers to the next kv
    K += inner_k_stride;
    V += inner_v_stride;
  }

  if (lane_idx == 0) {
    max_scores[warp_idx] = max_score;
    sum_exp_scores[warp_idx] = sum_exp_score;
  }
  block.sync();

  max_score = max_scores[lane_idx];
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  sum_exp_score =
      cg::reduce(warp, sum_exp_scores[lane_idx] * factor, cg::plus<U>());
  sum_exp_score = __frcp_rn(sum_exp_score);

  // Now we need to aggregate all the outputs
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[lane_idx][warp_idx] = o[i];
    block.sync();
    U ot = outputs[warp_idx][lane_idx] * factor;
    o[i] = cg::reduce(warp, ot, cg::plus<U>()) * sum_exp_score;
    block.sync();
  }

  // And write the output
  if (lane_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      O[v_per_thread * warp_idx + i] = static_cast<T>(o[i]);
    }
  }
}

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_2pass_1(
    const T* Q,
    const T* K,
    const T* V,
    float* partials,
    float* sums,
    float* maxs,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 8;
  constexpr int BD = 32;
  constexpr int blocks = 32;

  constexpr int v_per_thread = D / BD;

  const int inner_k_stride = blocks * BN * int(params.K_strides[2]);
  const int inner_v_stride = blocks * BN * int(params.V_strides[2]);

  typedef float U;

  U q[v_per_thread];
  U k[v_per_thread];
  U o[v_per_thread];

  __shared__ U outputs[BN][BD + 1];
  __shared__ U max_scores[BN];
  __shared__ U sum_exp_scores[BN];

  const U scale_log2 = params.scale * 1.44269504089f;

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z / blocks;
  const int block_idx = blockIdx.z % blocks;
  const int head_idx = blockIdx.x;
  const int kv_head_idx = head_idx / params.gqa_factor;

  const int q_seq_idx = blockIdx.y;
  const int kv_seq_idx = block_idx * BN + warp_idx;

  Q += batch_idx * params.Q_strides[0] + // Batch
      head_idx * params.Q_strides[1] + // Head
      q_seq_idx * params.Q_strides[2]; // Sequence

  K += batch_idx * params.K_strides[0] + // Batch
      kv_head_idx * params.K_strides[1] + // Head
      kv_seq_idx * params.K_strides[2]; // Sequence

  V += batch_idx * params.V_strides[0] + // Batch
      kv_head_idx * params.V_strides[1] + // Head
      kv_seq_idx * params.V_strides[2]; // Sequence

  const int p_stride_s = blocks;
  const int p_stride_h = params.qL * p_stride_s;
  const int p_stride_b = params.H * p_stride_h;
  const int p_offset = batch_idx * p_stride_b + // Batch
      head_idx * p_stride_h + // Head
      q_seq_idx * p_stride_s + // Sequence
      block_idx; // Block

  partials += p_offset * D;
  sums += p_offset;
  maxs += p_offset;

  // Read the query and 0 the output accumulator
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    q[i] = scale_log2 * static_cast<U>(Q[v_per_thread * lane_idx + i]);
  }

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = 0.f;
  }

  U max_score = -1e9;
  U sum_exp_score = 0.f;

  // For each key
  for (int i = kv_seq_idx; i < params.kL; i += blocks * BN) {
    bool use_key = true;
    if constexpr (do_causal) {
      use_key = i <= (params.kL - params.qL + q_seq_idx);
    }

    if (use_key) {
      // Read the key
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        k[j] = K[v_per_thread * lane_idx + j];
      }

      // Compute the i-th score
      U score = 0.f;
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        score += q[j] * k[j];
      }

      // Warp sum
      score = cg::reduce(warp, score, cg::plus<U>());

      // Update the accumulators
      U new_max = max(max_score, score);
      U factor = exp2f(max_score - new_max);
      U exp_score = exp2f(score - new_max);

      max_score = new_max;
      sum_exp_score = sum_exp_score * factor + exp_score;

      // Update the output accumulator
      PRAGMA_LOOP_UNROLL
      for (int j = 0; j < v_per_thread; j++) {
        o[j] = o[j] * factor +
            exp_score * static_cast<U>(V[v_per_thread * lane_idx + j]);
      }
    }

    // Move the pointers to the next kv
    K += inner_k_stride;
    V += inner_v_stride;
  }

  if (lane_idx == 0) {
    max_scores[warp_idx] = max_score;
    sum_exp_scores[warp_idx] = sum_exp_score;
  }

  block.sync();

  max_score = (lane_idx < BN) ? max_scores[lane_idx] : -1e9;
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  sum_exp_score = (lane_idx < BN) ? sum_exp_scores[lane_idx] : 0.f;
  sum_exp_score = cg::reduce(warp, sum_exp_score * factor, cg::plus<U>());

  // Write the sum and new max
  if (warp_idx == 0) {
    sums[0] = sum_exp_score;
    maxs[0] = new_max;
  }

  // Now we need to aggregate all the outputs
  auto ff = exp2f(max_scores[warp_idx] - new_max);
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[warp_idx][lane_idx] = o[i] * ff;
    block.sync();

    if (warp_idx == 0) {
      U ot = outputs[0][lane_idx];
      PRAGMA_LOOP_UNROLL
      for (int j = 1; j < BN; j++) {
        ot += outputs[j][lane_idx];
        warp.sync();
      }
      o[i] = ot;
    }
    block.sync();
  }

  if (warp_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      partials[v_per_thread * lane_idx + i] = o[i];
    }
  }
}

template <typename T, bool do_causal, int D>
__global__ void kernel_sdpav_2pass_2(
    const float* partials,
    const float* sums,
    const float* maxs,
    T* O,
    __grid_constant__ const AttnParams params) {
  constexpr int BN = 32;
  constexpr int BD = 32;
  constexpr int blocks = 32;

  constexpr int v_per_thread = D / BD;

  typedef float U;

  U o[v_per_thread];
  __shared__ U outputs[BN][BD + 1];

  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(block);

  const int lane_idx = warp.thread_rank();
  const int warp_idx = warp.meta_group_rank();

  // Adjust to thread block and thread
  const int batch_idx = blockIdx.z;
  const int head_idx = blockIdx.x;
  const int q_seq_idx = blockIdx.y;

  const int p_stride_s = blocks;
  const int p_stride_h = params.qL * p_stride_s;
  const int p_stride_b = params.H * p_stride_h;
  const int p_offset = batch_idx * p_stride_b + // Batch
      head_idx * p_stride_h + // Head
      q_seq_idx * p_stride_s; // Sequence

  partials += p_offset * D + warp_idx * D;
  sums += p_offset;
  maxs += p_offset;

  O += batch_idx * params.O_strides[0] + // Batch
      head_idx * params.O_strides[1] + // Head
      q_seq_idx * params.O_strides[2]; // Sequence

  U max_score = maxs[lane_idx];
  U new_max = cg::reduce(warp, max_score, cg::greater<U>());
  U factor = exp2f(max_score - new_max);
  U sum_exp_score = cg::reduce(warp, sums[lane_idx] * factor, cg::plus<U>());
  sum_exp_score = __frcp_rn(sum_exp_score);

  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    o[i] = partials[v_per_thread * lane_idx + i];
  }

  // Now we need to aggregate all the outputs
  PRAGMA_LOOP_UNROLL
  for (int i = 0; i < v_per_thread; i++) {
    outputs[lane_idx][warp_idx] = o[i];
    block.sync();
    U ot = outputs[warp_idx][lane_idx] * factor;
    o[i] = cg::reduce(warp, ot, cg::plus<U>()) * sum_exp_score;
    block.sync();
  }

  // And write the output
  if (lane_idx == 0) {
    PRAGMA_LOOP_UNROLL
    for (int i = 0; i < v_per_thread; i++) {
      O[v_per_thread * warp_idx + i] = static_cast<T>(o[i]);
    }
  }
}

} // namespace cu

namespace {

template <typename F>
void dispatch_headdim(int n, F&& f) {
  switch (n) {
    case 64:
      f(std::integral_constant<int, 64>{});
      break;
    case 96:
      f(std::integral_constant<int, 96>{});
      break;
    case 128:
      f(std::integral_constant<int, 128>{});
      break;
  }
}

void sdpa_vector_1pass_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  encoder.set_input_array(q);
  encoder.set_input_array(k);
  encoder.set_input_array(v);
  encoder.set_output_array(o);

  cu::AttnParams params{
      /* int B = */ q.shape(0),
      /* int H = */ q.shape(1),
      /* int D = */ q.shape(3),

      /* int qL = */ q.shape(2),
      /* int kL = */ k.shape(2),

      /* int gqa_factor = */ q.shape(1) / k.shape(1),
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)}};

  dim3 grid_dim(params.H, params.qL, params.B);
  dim3 block_dim(1024, 1, 1);

  dispatch_float_types(o.dtype(), "kernel_sdpav_1pass", [&](auto type_tag) {
    dispatch_bool(do_causal_, [&](auto do_causal) {
      dispatch_headdim(params.D, [&](auto headdim) {
        using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

        auto kernel =
            cu::kernel_sdpav_1pass<DataType, do_causal.value, headdim.value>;
        encoder.add_kernel_node(
            kernel,
            grid_dim,
            block_dim,
            0,
            q.data<DataType>(),
            k.data<DataType>(),
            v.data<DataType>(),
            o.data<DataType>(),
            params);
      });
    });
  });
}

void sdpa_vector_2pass_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  cu::AttnParams params{
      /* int B = */ q.shape(0),
      /* int H = */ q.shape(1),
      /* int D = */ q.shape(3),

      /* int qL = */ q.shape(2),
      /* int kL = */ k.shape(2),

      /* int gqa_factor = */ q.shape(1) / k.shape(1),
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)}};

  // Allocate the intermediates
  int blocks = 32;

  Shape intermediate_shape;
  intermediate_shape.reserve(o.ndim() + 1);
  intermediate_shape.insert(
      intermediate_shape.end(), o.shape().begin(), o.shape().end() - 1);
  intermediate_shape.push_back(blocks);
  intermediate_shape.push_back(o.shape().back());

  array intermediate(intermediate_shape, float32, nullptr, {});
  intermediate_shape.pop_back();
  array sums(intermediate_shape, float32, nullptr, {});
  array maxs(std::move(intermediate_shape), float32, nullptr, {});

  intermediate.set_data(allocator::malloc(intermediate.nbytes()));
  sums.set_data(allocator::malloc(sums.nbytes()));
  maxs.set_data(allocator::malloc(maxs.nbytes()));

  encoder.add_temporary(intermediate);
  encoder.add_temporary(sums);
  encoder.add_temporary(maxs);

  dispatch_float_types(o.dtype(), "kernel_sdpav_2pass", [&](auto type_tag) {
    dispatch_bool(do_causal_, [&](auto do_causal) {
      dispatch_headdim(params.D, [&](auto headdim) {
        using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

        {
          auto kernel = cu::
              kernel_sdpav_2pass_1<DataType, do_causal.value, headdim.value>;

          encoder.set_input_array(q);
          encoder.set_input_array(k);
          encoder.set_input_array(v);
          encoder.set_output_array(intermediate);
          encoder.set_output_array(sums);
          encoder.set_output_array(maxs);

          dim3 grid_dim(params.H, params.qL, params.B * 32);
          dim3 block_dim(8 * 32, 1, 1);

          encoder.add_kernel_node(
              kernel,
              grid_dim,
              block_dim,
              0,
              q.data<DataType>(),
              k.data<DataType>(),
              v.data<DataType>(),
              intermediate.data<float>(),
              sums.data<float>(),
              maxs.data<float>(),
              params);
        }

        {
          auto kernel = cu::
              kernel_sdpav_2pass_2<DataType, do_causal.value, headdim.value>;

          encoder.set_input_array(intermediate);
          encoder.set_input_array(sums);
          encoder.set_input_array(maxs);
          encoder.set_output_array(o);

          dim3 grid_dim(params.H, params.qL, params.B);
          dim3 block_dim(1024, 1, 1);

          encoder.add_kernel_node(
              kernel,
              grid_dim,
              block_dim,
              0,
              intermediate.data<float>(),
              sums.data<float>(),
              maxs.data<float>(),
              o.data<DataType>(),
              params);
        }
      });
    });
  });
}

void sdpa_vector_fallback(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  int kL = k.shape(2);

  if (kL > 1024) {
    return sdpa_vector_2pass_fallback(
        s, encoder, q, k, v, scale, o, do_causal_);
  } else {
    return sdpa_vector_1pass_fallback(
        s, encoder, q, k, v, scale, o, do_causal_);
  }
}

} // namespace

namespace fast {

bool ScaledDotProductAttention::use_fallback(
    const array& q,
    const array& k,
    const array& v,
    bool has_mask,
    bool has_arr_mask,
    bool do_causal,
    Stream s) {
  if (detail::in_grad_tracing()) {
    return true;
  }
  if (s.device == Device::cpu) {
    return true;
  }

  const int value_head_dim = v.shape(-1);
  const int query_head_dim = q.shape(-1);
  const int query_sequence_length = q.shape(2);
  const int key_sequence_length = k.shape(2);

  const bool sdpa_supported_head_dim = query_head_dim == value_head_dim &&
      (query_head_dim == 64 || query_head_dim == 96 || query_head_dim == 128);

  const bool supported_vector_config =
      sdpa_supported_head_dim && query_sequence_length < 4;

  const bool supported_config = supported_vector_config;

  return has_arr_mask || !supported_config;
}

void ScaledDotProductAttention::eval_gpu(
    const std::vector<array>& inputs,
    array& out) {
  nvtx3::scoped_range r("ScaledDotProductAttention::eval_gpu");

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);

  auto& q_pre = inputs[0];
  auto& k_pre = inputs[1];
  auto& v_pre = inputs[2];
  auto& o = out;

  std::vector<array> copies;

  // Define some copy functions to ensure the layout of the inputs is as
  // expected.
  copies.reserve(3);
  auto copy_unless = [&copies, &s](
                         auto predicate, const array& arr) -> const array& {
    if (!predicate(arr)) {
      array arr_copy = contiguous_copy_gpu(arr, s);
      copies.push_back(std::move(arr_copy));
      return copies.back();
    } else {
      return arr;
    }
  };

  // We are in vector mode ie single query
  if (q_pre.shape(2) < 4) {
    auto q_copy_unless = [](const array& arr) {
      if (arr.flags().row_contiguous) {
        return true;
      }
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (shape[0] == 1 || shape[1] == 1) {
        // If either the batch or head dimension is a singleton, the other can
        // be transposed with the sequence dimension
        auto bidx = shape[0] == 1 ? 1 : 0;
        return (strides[3] == 1) && (strides[2] == shape[3] * shape[bidx]) &&
            (strides[bidx] == shape[3]);
      }
      return false;
    };

    auto kv_copy_unless = [](const array& arr) {
      // keys and values should be copied if:
      // - the last dimension is not contiguous
      // - the batch and head dim are not contiguous
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (strides.back() != 1) {
        return false;
      }
      if (shape[0] == 1 || shape[1] == 1) {
        return true;
      }
      return (strides[0] == strides[1] * shape[1]);
    };

    const auto& q = copy_unless(q_copy_unless, q_pre);
    const auto& k = copy_unless(kv_copy_unless, k_pre);
    const auto& v = copy_unless(kv_copy_unless, v_pre);

    for (const auto& cp : copies) {
      encoder.add_temporary(cp);
    }

    // Donate the query if possible
    if (q.is_donatable() && q.flags().row_contiguous && q.size() == o.size()) {
      o.copy_shared_buffer(q);
    } else {
      int64_t str_oD = 1;
      int64_t str_oH = o.shape(3);
      int64_t str_oL = o.shape(1) * str_oH;
      int64_t str_oB = o.shape(2) * str_oL;
      size_t data_size = o.shape(0) * str_oB;

      array::Flags flags{
          /* bool contiguous = */ 1,
          /* bool row_contiguous = */ o.shape(2) == 1,
          /* bool col_contiguous = */ 0,
      };

      o.set_data(
          allocator::malloc(o.nbytes()),
          data_size,
          {str_oB, str_oH, str_oL, str_oD},
          flags);
    }

    return sdpa_vector_fallback(s, encoder, q, k, v, scale_, o, do_causal_);
  }

  // Full attention mode should never reach here
  else {
    throw std::runtime_error("Doesn't support matrix yet.");
  }
}

} // namespace fast

} // namespace mlx::core
