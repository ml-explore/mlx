// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/config.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/lru_cache.h"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"

// cudnn_frontend.h redefines this macro.
#undef CHECK_CUDA_ERROR

#include <cudnn_frontend.h>
#include <fmt/format.h>
#include <nvtx3/nvtx3.hpp>

namespace fe = cudnn_frontend;

namespace mlx::core {

namespace cu {} // namespace cu

namespace {

struct SDPACacheKey {
  int device_id;
  fe::DataType_t cudnn_type;

  int B;
  int H;
  int D;

  int qL;
  int kL;

  int gqa_factor;
  float scale;

  int64_t Q_strides[3];
  int64_t K_strides[3];
  int64_t V_strides[3];
  int64_t O_strides[3];

  bool generate_stats;
  bool causal_mask;
};

auto& sdpa_cache() {
  static LRUBytesKeyCache<SDPACacheKey, std::shared_ptr<fe::graph::Graph>>
      cache(
          /* capacity */ 128);
  return cache;
}

#define Q_UID 1
#define K_UID 2
#define V_UID 3
#define O_UID 4
#define STATS_UID 5

std::shared_ptr<fe::graph::Graph> get_sdpa_forward_graph(
    cu::CommandEncoder& encoder,
    const SDPACacheKey& cache_key) {
  // Check if graph has already been fully built
  if (auto it = sdpa_cache().find(cache_key); it != sdpa_cache().end()) {
    return it->second;
  }

  // Set up new graph
  auto graph = std::make_shared<fe::graph::Graph>();

  graph->set_io_data_type(cache_key.cudnn_type)
      .set_intermediate_data_type(fe::DataType_t::FLOAT)
      .set_compute_data_type(fe::DataType_t::FLOAT);

  auto Q = graph->tensor(
      fe::graph::Tensor_attributes()
          .set_name("Q")
          .set_uid(Q_UID)
          .set_dim({cache_key.B, cache_key.H, cache_key.qL, cache_key.D})
          .set_stride(
              {cache_key.Q_strides[0],
               cache_key.Q_strides[1],
               cache_key.Q_strides[2],
               1}));

  int h_kv = cache_key.H / cache_key.gqa_factor;
  auto K =
      graph->tensor(fe::graph::Tensor_attributes()
                        .set_name("K")
                        .set_uid(K_UID)
                        .set_dim({cache_key.B, h_kv, cache_key.kL, cache_key.D})
                        .set_stride(
                            {cache_key.K_strides[0],
                             cache_key.K_strides[1],
                             cache_key.V_strides[2],
                             1}));

  auto V =
      graph->tensor(fe::graph::Tensor_attributes()
                        .set_name("V")
                        .set_uid(V_UID)
                        .set_dim({cache_key.B, h_kv, cache_key.kL, cache_key.D})
                        .set_stride(
                            {cache_key.V_strides[0],
                             cache_key.V_strides[1],
                             cache_key.V_strides[2],
                             1}));

  auto sdpa_options = fe::graph::SDPA_attributes()
                          .set_name("flash_attention")
                          .set_is_inference(!cache_key.generate_stats)
                          .set_attn_scale(cache_key.scale);

  if (cache_key.causal_mask && cache_key.qL > 1) {
    sdpa_options.set_diagonal_alignment(fe::DiagonalAlignment_t::TOP_LEFT)
        .set_diagonal_band_right_bound(0);
  }

  auto [O, Stats] = graph->sdpa(Q, K, V, sdpa_options);

  O->set_output(true)
      .set_uid(O_UID)
      .set_dim({cache_key.B, cache_key.H, cache_key.qL, cache_key.D})
      .set_stride(
          {cache_key.O_strides[0],
           cache_key.O_strides[1],
           cache_key.O_strides[2],
           1});

  if (cache_key.generate_stats) {
    Stats->set_output(true)
        .set_data_type(fe::DataType_t::FLOAT)
        .set_uid(STATS_UID);
  }

  // Build and Validate cudnn graph

  auto handle = encoder.device().cudnn_handle();

  // cuDNN only supports native CUDA graphs for sdpa in 9.6 or above.
  if (hipdnnGetVersion() < 90600) {
    auto build_status = graph->build(handle, {fe::HeurMode_t::A});
    if (!build_status.is_good()) {
      throw std::runtime_error(
          "Unable to build cudnn graph for attention."
          " Failed with message: " +
          build_status.get_message());
    }

  } else {
    auto val_status = graph->validate();
    auto op_status = graph->build_operation_graph(handle);

    auto plan_stauts =
        graph->create_execution_plans({cudnn_frontend::HeurMode_t::A});
    if (!plan_stauts.is_good()) {
      throw std::runtime_error(
          "Unable to create exec plan for cudnn attention."
          " Failed with message: " +
          plan_stauts.get_message());
    }

    graph->select_behavior_notes(
        {cudnn_frontend::BehaviorNote_t::SUPPORTS_CUDA_GRAPH_NATIVE_API});

    auto support_status = graph->check_support(handle);
    if (!support_status.is_good()) {
      throw std::runtime_error(
          "No cuda graph support for cudnn attention."
          " Failed with message: " +
          support_status.get_message());
    }

    auto build_status = graph->build_plans(handle);
    if (!build_status.is_good()) {
      throw std::runtime_error(
          "Unable to build cudnn graph for attention."
          " Failed with message: " +
          build_status.get_message());
    }
  }

  auto [it, _] = sdpa_cache().emplace(cache_key, graph);

  return it->second;
}

inline fe::DataType_t dtype_to_cudnn_type(Dtype dtype) {
  switch (dtype) {
    case int8:
      return fe::DataType_t::INT8;
    case int32:
      return fe::DataType_t::INT32;
    case uint8:
      return fe::DataType_t::UINT8;
    case float16:
      return fe::DataType_t::HALF;
    case bfloat16:
      return fe::DataType_t::BFLOAT16;
    case float32:
      return fe::DataType_t::FLOAT;
    case float64:
      return fe::DataType_t::DOUBLE;
    default:
      throw std::runtime_error(fmt::format(
          "Unsupported dtype in SDPA: {}.", dtype_to_string(dtype)));
  }
}

void sdpa_cudnn(
    const Stream& s,
    cu::CommandEncoder& encoder,
    const array& q,
    const array& k,
    const array& v,
    const float scale,
    array& o,
    bool do_causal_ = false) {
  encoder.set_input_array(q);
  encoder.set_input_array(k);
  encoder.set_input_array(v);
  encoder.set_output_array(o);

  auto cudnn_type = dtype_to_cudnn_type(q.dtype());

  int B = q.shape(0);
  int H = q.shape(1);
  int D = q.shape(3);
  int gqa_factor = q.shape(1) / k.shape(1);

  int qL = q.shape(2);
  int kL = k.shape(2);

  SDPACacheKey cache_key{
      /* int device_id = */ encoder.device().cuda_device(),
      /* fe::DataType_t cudnn_type = */ cudnn_type,

      /* int B = */ B,
      /* int H = */ H,
      /* int D = */ D,

      /* int qL = */ qL,
      /* int kL = */ kL,

      /* int gqa_factor = */ gqa_factor,
      /* float scale = */ scale,

      /* int64_t Q_strides[3] = */ {q.strides(0), q.strides(1), q.strides(2)},
      /* int64_t K_strides[3] = */ {k.strides(0), k.strides(1), k.strides(2)},
      /* int64_t V_strides[3] = */ {v.strides(0), v.strides(1), v.strides(2)},
      /* int64_t O_strides[3] = */ {o.strides(0), o.strides(1), o.strides(2)},

      /* bool generate_stats = */ false,
      /* bool causal_mask = */ do_causal_};

  auto graph = get_sdpa_forward_graph(encoder, cache_key);

  int64_t workspace_size = 0;
  auto workspace_status = graph->get_workspace_size(workspace_size);
  if (!workspace_status.is_good()) {
    throw std::runtime_error("Unable to get workspace for cudnn attention.");
  }

  array workspace(
      allocator::malloc(workspace_size), {int(workspace_size)}, uint8);
  auto workspace_ptr = workspace.data<void>();

  std::unordered_map<int64_t, void*> variant_pack = {
      {Q_UID, const_cast<void*>(q.data<void>())},
      {K_UID, const_cast<void*>(k.data<void>())},
      {V_UID, const_cast<void*>(v.data<void>())},
      {O_UID, o.data<void>()}};

  auto handle = encoder.device().cudnn_handle();
  hipdnnSetStream(handle, encoder.stream());

  // cuDNN only supports native CUDA graphs for sdpa in 9.6 or above.
  if (hipdnnGetVersion() < 90600) {
    auto capture = encoder.capture_context();
    auto exec_status = graph->execute(handle, variant_pack, workspace_ptr);

    if (!exec_status.is_good()) {
      capture.discard = true;
      throw std::runtime_error(
          "Unable to execute cudnn attention."
          " Failed with message: " +
          exec_status.get_message());
    }
  } else {
    hipGraph_t cu_graph;
    hipGraphCreate(&cu_graph, 0);

    std::unique_ptr<hipGraph_t, void (*)(hipGraph_t*)> graph_freer(
        &cu_graph, [](hipGraph_t* p) { hipGraphDestroy(*p); });

    auto cu_graph_status = graph->populate_cuda_graph(
        handle, variant_pack, workspace_ptr, cu_graph);

    if (!cu_graph_status.is_good()) {
      throw std::runtime_error(
          "Unable to add cuda graph for cudnn attention."
          " Failed with message: " +
          cu_graph_status.get_message());
    }

    encoder.add_graph_node(cu_graph);
  }

  encoder.add_temporary(workspace);
}

} // namespace

namespace fast {

bool ScaledDotProductAttention::use_fallback(
    const array& q,
    const array& k,
    const array& v,
    bool has_mask,
    bool has_arr_mask,
    bool do_causal,
    Stream s) {
  if (s.device == Device::cpu) {
    return true;
  }

  auto& cu_device = cu::device(s.device);
  if (cu_device.compute_capability_major() < 8) {
    return true;
  }

  const int value_head_dim = v.shape(-1);
  const int query_head_dim = q.shape(-1);
  const int query_sequence_length = q.shape(2);
  const int key_sequence_length = k.shape(2);

  const bool sdpa_supported_head_dim = query_head_dim == value_head_dim &&
      (query_head_dim == 64 || query_head_dim == 96 || query_head_dim == 128);

  const bool supported_dtype = q.dtype() == float16 || q.dtype() == bfloat16;

  const bool supported_config = supported_dtype && sdpa_supported_head_dim;

  return has_arr_mask || !supported_config;
}

void ScaledDotProductAttention::eval_gpu(
    const std::vector<array>& inputs,
    array& out) {
  nvtx3::scoped_range r("ScaledDotProductAttention::eval_gpu");

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);

  auto& q_pre = inputs[0];
  auto& k_pre = inputs[1];
  auto& v_pre = inputs[2];
  auto& o = out;

  std::vector<array> copies;

  // Define some copy functions to ensure the layout of the inputs is as
  // expected.
  copies.reserve(3);
  auto copy_unless = [&copies, &s](
                         auto predicate, const array& arr) -> const array& {
    if (!predicate(arr)) {
      array arr_copy = contiguous_copy_gpu(arr, s);
      copies.push_back(std::move(arr_copy));
      return copies.back();
    } else {
      return arr;
    }
  };

  auto is_matrix_contiguous = [](const array& arr) {
    return arr.strides(-1) == 1;
  };

  // We are in vector mode ie single query
  if (q_pre.shape(2) <= 1) {
    auto q_copy_unless = [](const array& arr) {
      if (arr.flags().row_contiguous) {
        return true;
      }
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (shape[0] == 1 || shape[1] == 1) {
        // If either the batch or head dimension is a singleton, the other can
        // be transposed with the sequence dimension
        auto bidx = shape[0] == 1 ? 1 : 0;
        return (strides[3] == 1) && (strides[2] == shape[3] * shape[bidx]) &&
            (strides[bidx] == shape[3]);
      }
      return false;
    };

    auto kv_copy_unless = [](const array& arr) {
      // keys and values should be copied if:
      // - the last dimension is not contiguous
      // - the batch and head dim are not contiguous
      auto& strides = arr.strides();
      auto& shape = arr.shape();
      if (strides.back() != 1) {
        return false;
      }
      if (shape[0] == 1 || shape[1] == 1) {
        return true;
      }
      return (strides[0] == strides[1] * shape[1]);
    };

    const auto& q = copy_unless(q_copy_unless, q_pre);
    const auto& k = copy_unless(kv_copy_unless, k_pre);
    const auto& v = copy_unless(kv_copy_unless, v_pre);

    for (const auto& cp : copies) {
      encoder.add_temporary(cp);
    }

    // Donate the query if possible
    if (q.is_donatable() && q.flags().row_contiguous && q.size() == o.size()) {
      o.copy_shared_buffer(q);
    } else {
      o.set_data(allocator::malloc(o.nbytes()));
    }

    return sdpa_cudnn(s, encoder, q, k, v, scale_, o, do_causal_);
  }

  // Full attention mode
  else {
    const auto& q = copy_unless(is_matrix_contiguous, q_pre);
    const auto& k = copy_unless(is_matrix_contiguous, k_pre);
    const auto& v = copy_unless(is_matrix_contiguous, v_pre);

    int64_t str_oD = 1;
    int64_t str_oH = o.shape(3);
    int64_t str_oL = o.shape(1) * str_oH;
    int64_t str_oB = o.shape(2) * str_oL;
    size_t data_size = o.shape(0) * str_oB;

    array::Flags flags{
        /* bool contiguous = */ 1,
        /* bool row_contiguous = */ 0,
        /* bool col_contiguous = */ 0,
    };

    o.set_data(
        allocator::malloc(o.nbytes()),
        data_size,
        {str_oB, str_oH, str_oL, str_oD},
        flags);

    return sdpa_cudnn(s, encoder, q, k, v, scale_, o, do_causal_);
  }
}

} // namespace fast

} // namespace mlx::core