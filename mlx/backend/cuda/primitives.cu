// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/kernels/arange.cuh"
#include "mlx/backend/cuda/kernels/fp16_math.cuh"
#include "mlx/distributed/primitives.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <cassert>

namespace mlx::core {

void Arange::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Arange::eval_gpu");
  assert(inputs.size() == 0);
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }
  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_output_array(out);
  encoder.launch_kernel([&, this](hipStream_t stream) {
    MLX_SWITCH_INT_FLOAT_TYPES_CHECKED(out.dtype(), "Arange", CTYPE, {
      using OutType = cuda_type_t<CTYPE>;
      CTYPE step =
          static_cast<CTYPE>(start_ + step_) - static_cast<CTYPE>(start_);
      thrust::transform(
          cu::thrust_policy(stream),
          thrust::counting_iterator<uint32_t>(0),
          thrust::counting_iterator<uint32_t>(out.data_size()),
          thrust::device_pointer_cast(out.data<OutType>()),
          cu::Arange<OutType>{
              static_cast<OutType>(start_), static_cast<OutType>(step)});
    });
  });
}

bool fast::ScaledDotProductAttention::use_fallback(
    const array& q,
    const array& k,
    const array& v,
    bool has_mask,
    bool has_arr_mask,
    bool do_causal,
    Stream s) {
  return true;
}

#define NO_GPU_MULTI(func)                                             \
  void func::eval_gpu(                                                 \
      const std::vector<array>& inputs, std::vector<array>& outputs) { \
    throw std::runtime_error(#func " has no CUDA implementation.");    \
  }

#define NO_GPU_USE_FALLBACK(func)     \
  bool func::use_fallback(Stream s) { \
    return true;                      \
  }                                   \
  NO_GPU_MULTI(func)

#define NO_GPU(func)                                                  \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) { \
    throw std::runtime_error(#func " has no CUDA implementation.");   \
  }

NO_GPU(ArgPartition)
NO_GPU(ArgReduce)
NO_GPU(ArgSort)
NO_GPU(BlockMaskedMM)
NO_GPU_MULTI(Compiled)
NO_GPU(Convolution)
NO_GPU_MULTI(DivMod)
NO_GPU(DynamicSlice)
NO_GPU(DynamicSliceUpdate)
NO_GPU(FFT)
NO_GPU(Gather)
NO_GPU(GatherAxis)
NO_GPU(GatherMM)
NO_GPU(GatherQMM)
NO_GPU(Hadamard)
NO_GPU(Load)
NO_GPU(LogSumExp)
NO_GPU_MULTI(LUF)
NO_GPU(Partition)
NO_GPU_MULTI(QRF)
NO_GPU(QuantizedMatmul)
NO_GPU(RandomBits)
NO_GPU(Reduce)
NO_GPU(Scan)
NO_GPU(Scatter)
NO_GPU(ScatterAxis)
NO_GPU(Select)
NO_GPU(SliceUpdate)
NO_GPU(Softmax)
NO_GPU(Sort)
NO_GPU_MULTI(SVD)
NO_GPU(Inverse)
NO_GPU(Cholesky)
NO_GPU_MULTI(Eig)
NO_GPU_MULTI(Eigh)

namespace fast {
NO_GPU_USE_FALLBACK(LayerNorm)
NO_GPU_MULTI(LayerNormVJP)
NO_GPU_USE_FALLBACK(RMSNorm)
NO_GPU_MULTI(RMSNormVJP)
NO_GPU_USE_FALLBACK(RoPE)
NO_GPU(ScaledDotProductAttention)
NO_GPU_MULTI(AffineQuantize)
NO_GPU_MULTI(CustomKernel)
} // namespace fast

namespace distributed {
NO_GPU_MULTI(AllReduce)
NO_GPU_MULTI(AllGather)
NO_GPU_MULTI(Send)
NO_GPU_MULTI(Recv)
} // namespace distributed

} // namespace mlx::core
