// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/dtype_utils.cuh"
#include "mlx/backend/cuda/kernels/arange.cuh"
#include "mlx/backend/cuda/kernels/fp16_math.cuh"
#include "mlx/distributed/primitives.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <cassert>

namespace mlx::core {

void Arange::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Arange::eval_gpu");
  assert(inputs.size() == 0);
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }
  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_output_array(out);
  encoder.launch_kernel([&, this](hipStream_t stream) {
    MLX_SWITCH_INT_FLOAT_TYPES_CHECKED(out.dtype(), "Arange", CTYPE, {
      using OutType = cuda_type_t<CTYPE>;
      CTYPE step =
          static_cast<CTYPE>(start_ + step_) - static_cast<CTYPE>(start_);
      thrust::transform(
          cu::thrust_policy(stream),
          thrust::counting_iterator<uint32_t>(0),
          thrust::counting_iterator<uint32_t>(out.data_size()),
          thrust::device_pointer_cast(out.data<OutType>()),
          cu::Arange<OutType>{
              static_cast<OutType>(start_), static_cast<OutType>(step)});
    });
  });
}

#define NO_GPU_MULTI(func)                                             \
  void func::eval_gpu(                                                 \
      const std::vector<array>& inputs, std::vector<array>& outputs) { \
    throw std::runtime_error(#func " has no CUDA implementation.");    \
  }

#define NO_GPU(func)                                                  \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) { \
    throw std::runtime_error(#func " has no CUDA implementation.");   \
  }

NO_GPU(Abs)
NO_GPU(Add)
NO_GPU(AddMM)
NO_GPU(ArcCos)
NO_GPU(ArcCosh)
NO_GPU(ArcSin)
NO_GPU(ArcSinh)
NO_GPU(ArcTan)
NO_GPU(ArcTan2)
NO_GPU(ArcTanh)
NO_GPU(ArgPartition)
NO_GPU(ArgReduce)
NO_GPU(ArgSort)
NO_GPU(BitwiseBinary)
NO_GPU(BitwiseInvert)
NO_GPU(BlockMaskedMM)
NO_GPU(Ceil)
NO_GPU_MULTI(Compiled)
NO_GPU(Conjugate)
NO_GPU(Convolution)
NO_GPU(Cos)
NO_GPU(Cosh)
NO_GPU(Divide)
NO_GPU_MULTI(DivMod)
NO_GPU(DynamicSlice)
NO_GPU(DynamicSliceUpdate)
NO_GPU(Remainder)
NO_GPU(Equal)
NO_GPU(Erf)
NO_GPU(ErfInv)
NO_GPU(Exp)
NO_GPU(Expm1)
NO_GPU(FFT)
NO_GPU(Floor)
NO_GPU(Gather)
NO_GPU(GatherAxis)
NO_GPU(GatherMM)
NO_GPU(GatherQMM)
NO_GPU(Greater)
NO_GPU(GreaterEqual)
NO_GPU(Hadamard)
NO_GPU(Imag)
NO_GPU(Less)
NO_GPU(LessEqual)
NO_GPU(Load)
NO_GPU(Log)
NO_GPU(Log1p)
NO_GPU(LogicalNot)
NO_GPU(LogicalAnd)
NO_GPU(LogicalOr)
NO_GPU(LogAddExp)
NO_GPU(LogSumExp)
NO_GPU_MULTI(LUF)
NO_GPU(Matmul)
NO_GPU(Maximum)
NO_GPU(Minimum)
NO_GPU(Multiply)
NO_GPU(Negative)
NO_GPU(NotEqual)
NO_GPU(Partition)
NO_GPU(Power)
NO_GPU_MULTI(QRF)
NO_GPU(QuantizedMatmul)
NO_GPU(RandomBits)
NO_GPU(Real)
NO_GPU(Reduce)
NO_GPU(Round)
NO_GPU(Scan)
NO_GPU(Scatter)
NO_GPU(ScatterAxis)
NO_GPU(Select)
NO_GPU(Sigmoid)
NO_GPU(Sign)
NO_GPU(Sin)
NO_GPU(Sinh)
NO_GPU(SliceUpdate)
NO_GPU(Softmax)
NO_GPU(Sort)
NO_GPU(Square)
NO_GPU(Sqrt)
NO_GPU(Subtract)
NO_GPU_MULTI(SVD)
NO_GPU(Tan)
NO_GPU(Tanh)
NO_GPU(Inverse)
NO_GPU(Cholesky)
NO_GPU_MULTI(Eigh)

namespace fast {
NO_GPU_MULTI(LayerNorm)
NO_GPU_MULTI(LayerNormVJP)
NO_GPU_MULTI(RMSNorm)
NO_GPU_MULTI(RMSNormVJP)
NO_GPU_MULTI(RoPE)
NO_GPU(ScaledDotProductAttention)
NO_GPU_MULTI(AffineQuantize)
NO_GPU_MULTI(CustomKernel)
} // namespace fast

namespace distributed {
NO_GPU_MULTI(AllReduce)
NO_GPU_MULTI(AllGather)
NO_GPU_MULTI(Send)
NO_GPU_MULTI(Recv)
} // namespace distributed

} // namespace mlx::core
