// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/kernels/arange.cuh"
#include "mlx/backend/cuda/kernels/fp16_math.cuh"
#include "mlx/backend/cuda/kernels/random.cuh"
#include "mlx/distributed/primitives.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

#include <cassert>

namespace mlx::core {

void Arange::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Arange::eval_gpu");
  assert(inputs.size() == 0);
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }
  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_output_array(out);
  encoder.launch_kernel([&, this](hipStream_t stream) {
    MLX_SWITCH_INT_FLOAT_TYPES_CHECKED(out.dtype(), "Arange", CTYPE, {
      using OutType = cuda_type_t<CTYPE>;
      CTYPE step =
          static_cast<CTYPE>(start_ + step_) - static_cast<CTYPE>(start_);
      thrust::transform(
          cu::thrust_policy(stream),
          thrust::counting_iterator<uint32_t>(0),
          thrust::counting_iterator<uint32_t>(out.data_size()),
          thrust::device_pointer_cast(out.data<OutType>()),
          cu::Arange<OutType>{
              static_cast<OutType>(start_), static_cast<OutType>(step)});
    });
  });
}

void RandomBits::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("RandomBits::eval_gpu");
  assert(inputs.size() == 1);

  // keys has shape (N1, ..., NK, 2)
  // out has shape (N1, ..., NK, M1, M2, ...)
  auto& keys = inputs[0];
  size_t num_keys = keys.size() / 2;

  size_t elems_per_key = out.size() / num_keys;
  size_t bytes_per_key = out.itemsize() * elems_per_key;
  out.set_data(allocator::malloc(out.nbytes()));
  if (out.size() == 0) {
    return;
  }

  size_t out_per_key = (bytes_per_key + 4 - 1) / 4;
  size_t half_size = out_per_key / 2;
  bool odd = out_per_key % 2;

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(keys);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    dim3 grid_dim{
        static_cast<uint32_t>(num_keys),
        static_cast<uint32_t>(half_size + odd)};
    dim3 block_dim = get_block_dims(grid_dim.x, grid_dim.y, 1);
    dim3 num_blocks{
        cuda::ceil_div(grid_dim.x, block_dim.x),
        cuda::ceil_div(grid_dim.y, block_dim.y)};
    if (keys.flags().row_contiguous) {
      cu::rbitsc<<<num_blocks, block_dim, 0, stream>>>(
          keys.data<uint32_t>(),
          out.data<uint8_t>(),
          grid_dim,
          odd,
          bytes_per_key);
    } else {
      cu::rbits<<<num_blocks, block_dim, 0, stream>>>(
          keys.data<uint32_t>(),
          out.data<uint8_t>(),
          grid_dim,
          odd,
          bytes_per_key,
          keys.ndim(),
          const_param(keys.shape()),
          const_param(keys.strides()));
    }
  });
}

#define NO_GPU_MULTI(func)                                             \
  void func::eval_gpu(                                                 \
      const std::vector<array>& inputs, std::vector<array>& outputs) { \
    throw std::runtime_error(#func " has no CUDA implementation.");    \
  }

#define NO_GPU(func)                                                  \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) { \
    throw std::runtime_error(#func " has no CUDA implementation.");   \
  }

NO_GPU(ArgPartition)
NO_GPU(BlockMaskedMM)
NO_GPU_MULTI(Compiled)
NO_GPU(Convolution)
NO_GPU_MULTI(DivMod)
NO_GPU(DynamicSlice)
NO_GPU(DynamicSliceUpdate)
NO_GPU(FFT)
NO_GPU(Gather)
NO_GPU(GatherAxis)
NO_GPU(GatherMM)
NO_GPU(GatherQMM)
NO_GPU(Hadamard)
NO_GPU(Load)
NO_GPU(LogSumExp)
NO_GPU_MULTI(LUF)
NO_GPU(Partition)
NO_GPU_MULTI(QRF)
NO_GPU(QuantizedMatmul)
NO_GPU(Scan)
NO_GPU(Scatter)
NO_GPU(ScatterAxis)
NO_GPU(Select)
NO_GPU(SliceUpdate)
NO_GPU(Softmax)
NO_GPU_MULTI(SVD)
NO_GPU(Inverse)
NO_GPU(Cholesky)
NO_GPU_MULTI(Eig)
NO_GPU_MULTI(Eigh)

namespace fast {
NO_GPU_MULTI(LayerNorm)
NO_GPU_MULTI(LayerNormVJP)
NO_GPU_MULTI(RMSNorm)
NO_GPU_MULTI(RMSNormVJP)
NO_GPU_MULTI(RoPE)
NO_GPU(ScaledDotProductAttention)
NO_GPU_MULTI(AffineQuantize)
NO_GPU_MULTI(CustomKernel)
} // namespace fast

namespace distributed {
NO_GPU_MULTI(AllReduce)
NO_GPU_MULTI(AllGather)
NO_GPU_MULTI(Send)
NO_GPU_MULTI(Recv)
} // namespace distributed

} // namespace mlx::core
