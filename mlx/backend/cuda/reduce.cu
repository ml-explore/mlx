// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/reduce/reduce.cuh"
#include "mlx/backend/gpu/copy.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include <cassert>

namespace mlx::core {

void Reduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Reduce::eval_gpu");
  assert(inputs.size() == 1);
  array in = inputs[0];

  // Make sure no identity reductions trickle down here.
  assert(!axes_.empty());
  assert(out.size() != in.size());

  out.set_data(allocator::malloc(out.nbytes()));

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);

  // Fill out with init value.
  if (in.size() == 0) {
    encoder.launch_kernel([&](hipStream_t stream) {
      MLX_SWITCH_ALL_TYPES(in.dtype(), CTYPE, {
        MLX_SWITCH_REDUCE_OPS(reduce_type_, OP, {
          using InType = cuda_type_t<CTYPE>;
          using OutType = cu::ReduceResult<OP, InType>::type;
          thrust::fill_n(
              cu::thrust_policy(stream),
              thrust::device_pointer_cast(out.data<OutType>()),
              out.data_size(),
              cu::ReduceInit<OP, InType>::value());
        });
      });
    });
    return;
  }

  // Reduce.
  ReductionPlan plan = get_reduction_plan(in, axes_);

  // If it is a general reduce then copy the input to a contiguous array and
  // recompute the plan.
  if (plan.type == GeneralReduce) {
    array in_copy(in.shape(), in.dtype(), nullptr, {});
    copy_gpu(in, in_copy, CopyType::General, s);
    encoder.add_temporary(in_copy);
    in = in_copy;
    plan = get_reduction_plan(in, axes_);
  }

  if ((plan.type == ContiguousAllReduce) ||
      (plan.type == ContiguousReduce && plan.shape.size() == 1)) {
    segmented_reduce(encoder, in, out, reduce_type_, axes_, plan);
    return;
  }

  if (plan.type == ContiguousReduce || plan.type == GeneralContiguousReduce) {
    row_reduce(encoder, in, out, reduce_type_, axes_, plan);
    return;
  }

  if (plan.type == ContiguousStridedReduce ||
      plan.type == GeneralStridedReduce) {
    col_reduce(encoder, in, out, reduce_type_, axes_, plan);
    return;
  }

  throw std::runtime_error("No plan reached in reduce.");
}

} // namespace mlx::core
