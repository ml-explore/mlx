#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/reduce/reduce.cuh"
#include "mlx/backend/gpu/copy.h"

#include <nvtx3/nvtx3.hpp>

#include <cassert>

namespace mlx::core {

void Reduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Reduce::eval_gpu");
  assert(inputs.size() == 1);
  array in = inputs[0];

  // Make sure no identity reductions trickle down here.
  assert(!axes_.empty());
  assert(out.size() != in.size());

  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);

  if (in.size() == 0) {
    init_reduce(encoder, in, out, reduce_type_);
    return;
  }

  // Reduce.
  ReductionPlan plan = get_reduction_plan(in, axes_);

  // If it is a general reduce then copy the input to a contiguous array and
  // recompute the plan.
  //
  // TODO: Instead of copying we can use elem-to-loc to deal with broadcasting
  //       like we do in Metal. When it comes to broadcasted reduction axes
  //       some can be ignored eg for min/max.
  bool broadcasted = false;
  for (int i = 0, j = 0; i < in.ndim() && !broadcasted; i++) {
    if (j < axes_.size() && axes_[j] == i) {
      j++;
    } else {
      broadcasted = in.strides(i) == 0;
    }
  }
  if (plan.type == GeneralReduce || broadcasted || !in.flags().contiguous) {
    array in_copy = contiguous_copy_gpu(in, s);
    encoder.add_temporary(in_copy);
    in = in_copy;
    plan = get_reduction_plan(in, axes_);
  }

  if (plan.type == ContiguousAllReduce) {
    all_reduce(encoder, in, out, reduce_type_);
    return;
  }

  if (plan.type == ContiguousReduce || plan.type == GeneralContiguousReduce) {
    row_reduce(encoder, in, out, reduce_type_, axes_, plan);
    return;
  }

  if (plan.type == ContiguousStridedReduce ||
      plan.type == GeneralStridedReduce) {
    col_reduce(encoder, in, out, reduce_type_, axes_, plan);
    return;
  }

  throw std::runtime_error("No plan reached in reduce.");
}

} // namespace mlx::core
