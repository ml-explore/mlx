#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/unary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/unary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename In, typename Out, typename IdxT, int N_READS>
__global__ void unary_v(const In* in, Out* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();

  if ((index + 1) * N_READS > size) {
    for (IdxT i = index * N_READS; i < size; ++i) {
      out[i] = Op{}(in[i]);
    }
  } else {
    auto in_vec = load_vector<N_READS>(in, index);

    AlignedVector<Out, N_READS> out_vec;
#pragma unroll
    for (int i = 0; i < N_READS; ++i) {
      out_vec[i] = Op{}(in_vec[i]);
    }

    store_vector<N_READS>(out, index, out_vec);
  }
}

template <typename Op, typename In, typename Out, typename IdxT>
__global__ void unary_g(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto idx = elem_to_loc(index, shape.data(), strides.data(), ndim);
    out[index] = Op{}(in[idx]);
  }
}

template <typename Op, typename In, typename Out>
constexpr bool supports_unary_op() {
  if (std::is_same_v<Op, Abs> || std::is_same_v<Op, Negative> ||
      std::is_same_v<Op, Sign> || std::is_same_v<Op, Square>) {
    return std::is_same_v<In, Out>;
  }
  if (std::is_same_v<Op, ArcCosh> || std::is_same_v<Op, ArcSinh> ||
      std::is_same_v<Op, ArcTanh> || std::is_same_v<Op, Erf> ||
      std::is_same_v<Op, ErfInv> || std::is_same_v<Op, Expm1> ||
      std::is_same_v<Op, Sigmoid>) {
    return std::is_same_v<In, Out> && is_floating_v<In>;
  }
  if (std::is_same_v<Op, BitwiseInvert>) {
    return std::is_same_v<In, Out> && std::is_integral_v<In> &&
        !std::is_same_v<In, bool>;
  }
  if (std::is_same_v<Op, Ceil> || std::is_same_v<Op, Floor>) {
    return std::is_same_v<In, Out> && !mlx::core::is_complex_v<In>;
  }
  if (std::is_same_v<Op, Conjugate>) {
    return std::is_same_v<In, Out> && mlx::core::is_complex_v<In>;
  }
  if (std::is_same_v<Op, ArcCos> || std::is_same_v<Op, ArcSin> ||
      std::is_same_v<Op, ArcTan> || std::is_same_v<Op, Cos> ||
      std::is_same_v<Op, Cosh> || std::is_same_v<Op, Exp> ||
      std::is_same_v<Op, Log> || std::is_same_v<Op, Log2> ||
      std::is_same_v<Op, Log10> || std::is_same_v<Op, Log1p> ||
      std::is_same_v<Op, Round> || std::is_same_v<Op, Rsqrt> ||
      std::is_same_v<Op, Sqrt> || std::is_same_v<Op, Sin> ||
      std::is_same_v<Op, Sinh> || std::is_same_v<Op, Tan> ||
      std::is_same_v<Op, Tanh>) {
    return std::is_same_v<In, Out> && is_inexact_v<In>;
  }
  if (std::is_same_v<Op, Imag> || std::is_same_v<Op, Real>) {
    return mlx::core::is_complex_v<In> && std::is_same_v<Out, float>;
  }
  if (std::is_same_v<Op, LogicalNot>) {
    return std::is_same_v<In, Out> && std::is_same_v<In, bool>;
  }
  return false;
}

} // namespace cu

template <typename Op>
void unary_op_gpu_inplace(
    const std::vector<array>& inputs,
    array& out,
    const char* op,
    const Stream& s) {
  auto& in = inputs[0];
  if (in.size() == 0) {
    return;
  }
  bool contig = in.flags().contiguous;
  bool large;
  if (!contig) {
    large = in.data_size() > INT32_MAX || out.size() > INT32_MAX;
  } else {
    large = in.data_size() > UINT32_MAX;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_all_types(in.dtype(), [&](auto in_type_tag) {
    dispatch_all_types(out.dtype(), [&](auto out_type_tag) {
      using CTYPE_IN = MLX_GET_TYPE(in_type_tag);
      using CTYPE_OUT = MLX_GET_TYPE(out_type_tag);
      if constexpr (cu::supports_unary_op<Op, CTYPE_IN, CTYPE_OUT>()) {
        dispatch_bool(large, [&](auto large) {
          using InType = cuda_type_t<CTYPE_IN>;
          using OutType = cuda_type_t<CTYPE_OUT>;
          if (contig) {
            using IdxT = std::conditional_t<large(), int64_t, uint32_t>;
            // TODO: Choose optimized value based on type size.
            constexpr int N_READS = 4;
            auto [num_blocks, block_dims] = get_launch_args(
                out.data_size(), out.shape(), out.strides(), large, N_READS);
            encoder.add_kernel_node(
                cu::unary_v<Op, InType, OutType, IdxT, N_READS>,
                num_blocks,
                block_dims,
                0,
                in.data<InType>(),
                out.data<OutType>(),
                out.data_size());
          } else {
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            auto [shape, strides] = collapse_contiguous_dims(in);
            auto [num_blocks, block_dims] = get_launch_args(out, large);
            encoder.add_kernel_node(
                cu::unary_g<Op, InType, OutType, IdxT>,
                num_blocks,
                block_dims,
                0,
                in.data<InType>(),
                out.data<OutType>(),
                out.data_size(),
                const_param(shape),
                const_param(strides),
                shape.size());
          }
        });
      } else {
        throw std::runtime_error(fmt::format(
            "Can not do unary op {} on input of {} with output of {}.",
            op,
            dtype_to_string(in.dtype()),
            dtype_to_string(out.dtype())));
      }
    });
  });
}

template <typename Op>
void unary_op_gpu(
    const std::vector<array>& inputs,
    array& out,
    const char* op,
    const Stream& s) {
  set_unary_output_data(inputs[0], out);
  unary_op_gpu_inplace<Op>(inputs, out, op, s);
}

#define UNARY_GPU(func)                                               \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) { \
    nvtx3::scoped_range r(#func "::eval_gpu");                        \
    auto& s = out.primitive().stream();                               \
    unary_op_gpu<cu::func>(inputs, out, name(), s);                   \
  }

UNARY_GPU(Abs)
UNARY_GPU(ArcCos)
UNARY_GPU(ArcCosh)
UNARY_GPU(ArcSin)
UNARY_GPU(ArcSinh)
UNARY_GPU(ArcTan)
UNARY_GPU(ArcTanh)
UNARY_GPU(BitwiseInvert)
UNARY_GPU(Ceil)
UNARY_GPU(Conjugate)
UNARY_GPU(Cos)
UNARY_GPU(Cosh)
UNARY_GPU(Erf)
UNARY_GPU(ErfInv)
UNARY_GPU(Exp)
UNARY_GPU(Expm1)
UNARY_GPU(Floor)
UNARY_GPU(Imag)
UNARY_GPU(Log1p)
UNARY_GPU(LogicalNot)
UNARY_GPU(Negative)
UNARY_GPU(Real)
UNARY_GPU(Sigmoid)
UNARY_GPU(Sign)
UNARY_GPU(Sin)
UNARY_GPU(Sinh)
UNARY_GPU(Square)
UNARY_GPU(Tan)
UNARY_GPU(Tanh)

void Log::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Log::eval_gpu");
  auto& s = out.primitive().stream();
  switch (base_) {
    case Base::e:
      unary_op_gpu<cu::Log>(inputs, out, name(), s);
      break;
    case Base::two:
      unary_op_gpu<cu::Log2>(inputs, out, name(), s);
      break;
    case Base::ten:
      unary_op_gpu<cu::Log10>(inputs, out, name(), s);
      break;
  }
}

void Round::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Round::eval_gpu");
  assert(inputs.size() == 1);
  const auto& in = inputs[0];
  auto& s = out.primitive().stream();
  if (issubdtype(in.dtype(), inexact)) {
    unary_op_gpu<cu::Round>(inputs, out, name(), s);
  } else {
    // No-op integer types
    out.copy_shared_buffer(in);
  }
}

void Sqrt::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Sort::eval_gpu");
  auto& s = out.primitive().stream();
  if (recip_) {
    unary_op_gpu<cu::Rsqrt>(inputs, out, "Rsqrt", s);
  } else {
    unary_op_gpu<cu::Sqrt>(inputs, out, "Sqrt", s);
  }
}

} // namespace mlx::core
