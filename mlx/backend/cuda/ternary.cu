#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.
#include "mlx/backend/common/ternary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/ternary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename T, typename IdxT, int N_READS>
__global__ void
ternary_v(const bool* a, const T* b, const T* c, T* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();

  if ((index + 1) * N_READS > size) {
    for (IdxT i = index * N_READS; i < size; ++i) {
      out[i] = Op{}(a[i], b[i], c[i]);
    }
  } else {
    auto a_vec = load_vector<N_READS>(a, index);
    auto b_vec = load_vector<N_READS>(b, index);
    auto c_vec = load_vector<N_READS>(c, index);

    AlignedVector<T, N_READS> out_vec;
#pragma unroll
    for (int i = 0; i < N_READS; ++i) {
      out_vec[i] = Op{}(a_vec[i], b_vec[i], c_vec[i]);
    }

    store_vector<N_READS>(out, index, out_vec);
  }
}

template <typename Op, typename T, typename IdxT, int NDIM, int N_READS>
__global__ void ternary_g_nd(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size_rest,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> a_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> b_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> c_strides) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[NDIM - 1];
  auto a_stride_x = a_strides[NDIM - 1];
  auto b_stride_x = b_strides[NDIM - 1];
  auto c_stride_x = c_strides[NDIM - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto [a_idx, b_idx, c_idx] = elem_to_loc_nd<NDIM>(
      index_rest * shape_x,
      shape.data(),
      a_strides.data(),
      b_strides.data(),
      c_strides.data());
  auto a_vec =
      load_vector<N_READS>(a + a_idx, index_x, shape_x, a_stride_x, false);
  auto b_vec =
      load_vector<N_READS>(b + b_idx, index_x, shape_x, b_stride_x, T(0));
  auto c_vec =
      load_vector<N_READS>(c + c_idx, index_x, shape_x, c_stride_x, T(0));

  AlignedVector<T, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = Op{}(a_vec[i], b_vec[i], c_vec[i]);
  }
  store_vector(out + shape_x * index_rest, index_x, out_vec, shape_x);
}

template <typename Op, typename T, typename IdxT, int N_READS>
__global__ void ternary_g(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size_rest,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides a_strides,
    const __grid_constant__ Strides b_strides,
    const __grid_constant__ Strides c_strides,
    int ndim) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[ndim - 1];
  auto a_stride_x = a_strides[ndim - 1];
  auto b_stride_x = b_strides[ndim - 1];
  auto c_stride_x = c_strides[ndim - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto [a_idx, b_idx, c_idx] = elem_to_loc(
      index_rest * shape_x,
      shape.data(),
      a_strides.data(),
      b_strides.data(),
      c_strides.data(),
      ndim);
  auto a_vec =
      load_vector<N_READS>(a + a_idx, index_x, shape_x, a_stride_x, false);
  auto b_vec =
      load_vector<N_READS>(b + b_idx, index_x, shape_x, b_stride_x, T(0));
  auto c_vec =
      load_vector<N_READS>(c + c_idx, index_x, shape_x, c_stride_x, T(0));

  AlignedVector<T, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = Op{}(a_vec[i], b_vec[i], c_vec[i]);
  }
  store_vector(out + shape_x * index_rest, index_x, out_vec, shape_x);
}

} // namespace cu

template <typename Op>
void ternary_op_gpu_inplace(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  const auto& a = inputs[0];
  const auto& b = inputs[1];
  const auto& c = inputs[2];
  if (out.size() == 0) {
    return;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_input_array(c);
  encoder.set_output_array(out);
  dispatch_all_types(out.dtype(), [&](auto type_tag) {
    using DType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

    auto topt = get_ternary_op_type(a, b, c);
    if (topt == TernaryOpType::General) {
      dispatch_bool(
          a.data_size() > INT32_MAX || b.data_size() > INT32_MAX ||
              c.data_size() > INT32_MAX || out.data_size() > INT32_MAX,
          [&](auto large) {
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            Shape shape;
            std::vector<Strides> strides;
            std::tie(shape, strides) = collapse_contiguous_dims(a, b, c, out);
            auto& a_strides = strides[0];
            auto& b_strides = strides[1];
            auto& c_strides = strides[2];
            int ndim = shape.size();
            int work_per_thread = 1;
            auto dim0 = ndim > 0 ? shape.back() : 1;
            auto rest = out.size() / dim0;
            if (dim0 >= 4) {
              work_per_thread = 4;
            }
            dim0 = (dim0 + work_per_thread - 1) / work_per_thread;
            auto block_dims = get_block_dims(dim0, rest, 1);
            uint32_t num_blocks_x = cuda::ceil_div(dim0, block_dims.x);
            uint32_t num_blocks_y = cuda::ceil_div(rest, block_dims.y);

            if (ndim <= 3) {
              dispatch_1_2_3(ndim, [&](auto dims_constant) {
                auto kernel =
                    cu::ternary_g_nd<Op, DType, IdxT, dims_constant(), 1>;
                if (work_per_thread == 4) {
                  kernel =
                      cu::ternary_g_nd<Op, DType, IdxT, dims_constant(), 4>;
                }
                encoder.add_kernel_node(
                    kernel,
                    {num_blocks_x, num_blocks_y},
                    block_dims,
                    0,
                    a.data<bool>(),
                    b.data<DType>(),
                    c.data<DType>(),
                    out.data<DType>(),
                    rest,
                    const_param<dims_constant()>(shape),
                    const_param<dims_constant()>(a_strides),
                    const_param<dims_constant()>(b_strides),
                    const_param<dims_constant()>(c_strides));
              });
            } else {
              auto kernel = cu::ternary_g<Op, DType, IdxT, 1>;
              if (work_per_thread == 4) {
                kernel = cu::ternary_g<Op, DType, IdxT, 4>;
              }
              encoder.add_kernel_node(
                  kernel,
                  {num_blocks_x, num_blocks_y},
                  block_dims,
                  0,
                  a.data<bool>(),
                  b.data<DType>(),
                  c.data<DType>(),
                  out.data<DType>(),
                  rest,
                  const_param(shape),
                  const_param(a_strides),
                  const_param(b_strides),
                  const_param(c_strides),
                  ndim);
            }
          });
    } else {
      dispatch_bool(out.data_size() > UINT32_MAX, [&](auto large) {
        using IdxT = std::conditional_t<large(), int64_t, uint32_t>;
        constexpr int N_READS = 16 / sizeof(DType);
        auto [num_blocks, block_dims] = get_launch_args(
            out.data_size(), out.shape(), out.strides(), large(), N_READS);
        encoder.add_kernel_node(
            cu::ternary_v<Op, DType, IdxT, N_READS>,
            num_blocks,
            block_dims,
            0,
            a.data<bool>(),
            b.data<DType>(),
            c.data<DType>(),
            out.data<DType>(),
            out.data_size());
      });
    }
  });
}

template <typename Op>
void ternary_op_gpu(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto& c = inputs[2];
  auto topt = get_ternary_op_type(a, b, c);
  set_ternary_op_output_data(a, b, c, out, topt);
  ternary_op_gpu_inplace<Op>(inputs, out, s);
}

void Select::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Select::eval_gpu");
  auto& s = out.primitive().stream();
  ternary_op_gpu<cu::Select>(inputs, out, s);
}

} // namespace mlx::core
