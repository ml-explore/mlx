#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.
#include "mlx/backend/common/ternary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/ternary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename T, typename IdxT>
__global__ void
ternary_v(const bool* a, const T* b, const T* c, T* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    out[index] = Op{}(a[index], b[index], c[index]);
  }
}

template <typename Op, typename T, typename IdxT, int NDIM>
__global__ void ternary_g_nd(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> a_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> b_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> c_strides) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx, c_idx] = elem_to_loc_nd<NDIM>(
        index,
        shape.data(),
        a_strides.data(),
        b_strides.data(),
        c_strides.data());
    out[index] = Op{}(a[a_idx], b[b_idx], c[c_idx]);
  }
}

template <typename Op, typename T, typename IdxT>
__global__ void ternary_g(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides a_strides,
    const __grid_constant__ Strides b_strides,
    const __grid_constant__ Strides c_strides,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx, c_idx] = elem_to_loc_4d(
        index,
        shape.data(),
        a_strides.data(),
        b_strides.data(),
        c_strides.data(),
        ndim);
    out[index] = Op{}(a[a_idx], b[b_idx], c[c_idx]);
  }
}

} // namespace cu

template <typename Op>
void ternary_op_gpu_inplace(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  const auto& a = inputs[0];
  const auto& b = inputs[1];
  const auto& c = inputs[2];
  if (out.size() == 0) {
    return;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_input_array(c);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(out.dtype(), CTYPE, {
      using DType = cuda_type_t<CTYPE>;

      auto topt = get_ternary_op_type(a, b, c);
      if (topt == TernaryOpType::General) {
        auto [shape, strides] = collapse_contiguous_dims(a, b, c, out);
        auto& a_strides = strides[0];
        auto& b_strides = strides[1];
        auto& c_strides = strides[2];
        bool large = a.data_size() > UINT32_MAX || b.data_size() > UINT32_MAX ||
            c.data_size() > UINT32_MAX || out.data_size() > UINT32_MAX;
        MLX_SWITCH_BOOL(large, LARGE, {
          using IdxT = std::conditional_t<LARGE, int64_t, uint32_t>;
          int ndim = shape.size();
          if (ndim <= 3) {
            MLX_SWITCH_1_2_3(ndim, NDIM, {
              auto kernel = cu::ternary_g_nd<Op, DType, IdxT, NDIM>;
              auto [num_blocks, block_dims] =
                  get_launch_args(kernel, out, large);
              kernel<<<num_blocks, block_dims, 0, stream>>>(
                  a.data<bool>(),
                  b.data<DType>(),
                  c.data<DType>(),
                  out.data<DType>(),
                  out.data_size(),
                  const_param<NDIM>(shape),
                  const_param<NDIM>(a_strides),
                  const_param<NDIM>(b_strides),
                  const_param<NDIM>(c_strides));
            });
          } else {
            auto kernel = cu::ternary_g<Op, DType, IdxT>;
            auto [num_blocks, block_dims] = get_launch_args(kernel, out, large);
            kernel<<<num_blocks, block_dims, 0, stream>>>(
                a.data<bool>(),
                b.data<DType>(),
                c.data<DType>(),
                out.data<DType>(),
                out.data_size(),
                const_param(shape),
                const_param(a_strides),
                const_param(b_strides),
                const_param(c_strides),
                ndim);
          }
        });
      } else {
        MLX_SWITCH_BOOL(out.data_size() > UINT32_MAX, LARGE, {
          using IdxT = std::conditional_t<LARGE, int64_t, uint32_t>;
          auto kernel = cu::ternary_v<Op, DType, IdxT>;
          auto [num_blocks, block_dims] = get_launch_args(kernel, out, LARGE);
          kernel<<<num_blocks, block_dims, 0, stream>>>(
              a.data<bool>(),
              b.data<DType>(),
              c.data<DType>(),
              out.data<DType>(),
              out.data_size());
        });
      }
    });
  });
}

template <typename Op>
void ternary_op_gpu(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto& c = inputs[2];
  auto topt = get_ternary_op_type(a, b, c);
  set_ternary_op_output_data(a, b, c, out, topt);
  ternary_op_gpu_inplace<Op>(inputs, out, s);
}

void Select::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("select::eval_gpu");
  auto& s = out.primitive().stream();
  ternary_op_gpu<cu::Select>(inputs, out, s);
}

} // namespace mlx::core
