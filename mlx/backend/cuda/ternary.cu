#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.
#include "mlx/backend/common/ternary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/ternary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename T, typename IdxT, int N_READS>
__global__ void
ternary_v(const bool* a, const T* b, const T* c, T* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();

  if ((index + 1) * N_READS > size) {
    for (IdxT i = index * N_READS; i < size; ++i) {
      out[i] = Op{}(a[i], b[i], c[i]);
    }
  } else {
    auto a_vec = load_vector<N_READS>(a, index);
    auto b_vec = load_vector<N_READS>(b, index);
    auto c_vec = load_vector<N_READS>(c, index);

    AlignedVector<T, N_READS> out_vec;
#pragma unroll
    for (int i = 0; i < N_READS; ++i) {
      out_vec[i] = Op{}(a_vec[i], b_vec[i], c_vec[i]);
    }

    store_vector<N_READS>(out, index, out_vec);
  }
}

template <typename Op, typename T, typename IdxT, int NDIM>
__global__ void ternary_g_nd(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> a_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> b_strides,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> c_strides) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx, c_idx] = elem_to_loc_nd<NDIM>(
        index,
        shape.data(),
        a_strides.data(),
        b_strides.data(),
        c_strides.data());
    out[index] = Op{}(a[a_idx], b[b_idx], c[c_idx]);
  }
}

template <typename Op, typename T, typename IdxT>
__global__ void ternary_g(
    const bool* a,
    const T* b,
    const T* c,
    T* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides a_strides,
    const __grid_constant__ Strides b_strides,
    const __grid_constant__ Strides c_strides,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [a_idx, b_idx, c_idx] = elem_to_loc(
        index,
        shape.data(),
        a_strides.data(),
        b_strides.data(),
        c_strides.data(),
        ndim);
    out[index] = Op{}(a[a_idx], b[b_idx], c[c_idx]);
  }
}

} // namespace cu

template <typename Op>
void ternary_op_gpu_inplace(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  const auto& a = inputs[0];
  const auto& b = inputs[1];
  const auto& c = inputs[2];
  if (out.size() == 0) {
    return;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_input_array(c);
  encoder.set_output_array(out);
  dispatch_all_types(out.dtype(), [&](auto type_tag) {
    using DType = cuda_type_t<MLX_GET_TYPE(type_tag)>;

    auto topt = get_ternary_op_type(a, b, c);
    if (topt == TernaryOpType::General) {
      dispatch_bool(
          a.data_size() > INT32_MAX || b.data_size() > INT32_MAX ||
              c.data_size() > INT32_MAX || out.data_size() > INT32_MAX,
          [&](auto large) {
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            Shape shape;
            std::vector<Strides> strides;
            std::tie(shape, strides) = collapse_contiguous_dims(a, b, c, out);
            auto& a_strides = strides[0];
            auto& b_strides = strides[1];
            auto& c_strides = strides[2];
            int ndim = shape.size();
            if (ndim <= 3) {
              dispatch_1_2_3(ndim, [&](auto dims_constant) {
                auto [num_blocks, block_dims] = get_launch_args(out, large());
                encoder.add_kernel_node(
                    cu::ternary_g_nd<Op, DType, IdxT, dims_constant()>,
                    num_blocks,
                    block_dims,
                    0,
                    a.data<bool>(),
                    b.data<DType>(),
                    c.data<DType>(),
                    out.data<DType>(),
                    out.size(),
                    const_param<dims_constant()>(shape),
                    const_param<dims_constant()>(a_strides),
                    const_param<dims_constant()>(b_strides),
                    const_param<dims_constant()>(c_strides));
              });
            } else {
              auto [num_blocks, block_dims] = get_launch_args(out, large());
              encoder.add_kernel_node(
                  cu::ternary_g<Op, DType, IdxT>,
                  num_blocks,
                  block_dims,
                  0,
                  a.data<bool>(),
                  b.data<DType>(),
                  c.data<DType>(),
                  out.data<DType>(),
                  out.data_size(),
                  const_param(shape),
                  const_param(a_strides),
                  const_param(b_strides),
                  const_param(c_strides),
                  ndim);
            }
          });
    } else {
      dispatch_bool(out.data_size() > UINT32_MAX, [&](auto large) {
        using IdxT = std::conditional_t<large(), int64_t, uint32_t>;
        constexpr int N_READS = 16 / sizeof(DType);
        auto [num_blocks, block_dims] = get_launch_args(
            out.data_size(), out.shape(), out.strides(), large(), N_READS);
        encoder.add_kernel_node(
            cu::ternary_v<Op, DType, IdxT, N_READS>,
            num_blocks,
            block_dims,
            0,
            a.data<bool>(),
            b.data<DType>(),
            c.data<DType>(),
            out.data<DType>(),
            out.data_size());
      });
    }
  });
}

template <typename Op>
void ternary_op_gpu(
    const std::vector<array>& inputs,
    array& out,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto& c = inputs[2];
  auto topt = get_ternary_op_type(a, b, c);
  set_ternary_op_output_data(a, b, c, out, topt);
  ternary_op_gpu_inplace<Op>(inputs, out, s);
}

void Select::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Select::eval_gpu");
  auto& s = out.primitive().stream();
  ternary_op_gpu<cu::Select>(inputs, out, s);
}

} // namespace mlx::core
