#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/quantized/quantized.h"
#include "mlx/dtype_utils.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_fp4.h>
#include <hip/hip_fp8.h>

namespace mlx::core {
namespace cu {

template <int bits>
struct Quantize {
  __device__ uint8_t operator()(float x) {
    if constexpr (bits == 8) {
      return __hip_fp8_e4m3_fnuz(x).__x;
    } else {
      return __hip_fp4_e2m1(x).__x;
    }
  }
};

template <int bits>
struct Dequantize {
  __device__ float operator()(uint8_t x) {
    if constexpr (bits == 8) {
      return float(*(__hip_fp8_e4m3_fnuz*)(&x));
    } else {
      return float(*(__hip_fp4_e2m1*)(&x));
    }
  }
};

namespace cg = cooperative_groups;

template <typename T, int group_size, int bits, bool use_mx_scale>
__global__ void
fp_quantize(const T* w, uint8_t* out, uint8_t* scales, size_t size) {
  auto block_size = cg::this_thread_block().dim_threads();
  auto block_idx = cg::this_thread_block().group_index();
  auto idx_in_block = cg::this_thread_block().thread_index();

  auto tidx = block_idx.x * block_size.x + idx_in_block.x;
  auto tidy = block_idx.y * block_size.y + idx_in_block.y;

  auto grid_dim_x =
      cg::this_grid().dim_blocks().x * cg::this_grid().block_index().x;
  size_t out_index = tidx + grid_dim_x * size_t(tidy);
  size_t in_index = out_index;
  if (in_index >= size) {
    return;
  }

  float w_thread = w[in_index];

  cg::greater<float> max_op;
  auto warp = cg::tiled_partition<group_size>(cg::this_thread_block());

  float scale = cg::reduce(warp, abs(w_thread), max_op);
  scale /= bits == 4 ? 6.0f : 448.0f;
  // Convert to mx scale or nv scale
  using ScaleType =
      std::conditional_t<use_mx_scale, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz>;
  auto s = ScaleType(scale);
  uint8_t q_scale = s.__x;
  scale = float(s);

  // Write out the scales
  size_t gindex = in_index / group_size;
  if (in_index % group_size == 0) {
    scales[gindex] = q_scale;
  }

  uint8_t output = 0;
  uint8_t val = Quantize<bits>{}(scale == 0 ? 0.0f : w_thread / scale);
  output = val;
  if (bits == 4) {
    uint8_t sval = warp.shfl_down(val, 1);
    output |= sval << bits;
  }
  constexpr int pack_factor = bits == 8 ? 1 : 2;
  if (out_index % pack_factor == 0) {
    out[out_index / pack_factor] = output;
  }
}

template <typename T, int group_size, int bits, bool use_mx_scale>
__global__ void
fp_dequantize(const uint8_t* w, const uint8_t* scales, T* out, size_t size) {
  auto block_size = cg::this_thread_block().dim_threads();
  auto block_idx = cg::this_thread_block().group_index();
  auto idx_in_block = cg::this_thread_block().thread_index();

  auto tidx = block_idx.x * block_size.x + idx_in_block.x;
  auto tidy = block_idx.y * block_size.y + idx_in_block.y;

  auto grid_dim_x =
      cg::this_grid().dim_blocks().x * cg::this_grid().block_index().x;

  constexpr int pack_factor = bits == 8 ? 1 : 2;
  size_t offset = tidx + grid_dim_x * size_t(tidy);
  size_t oindex = offset * pack_factor;

  if (oindex >= size) {
    return;
  }

  size_t gindex = oindex / group_size;
  using ScaleType =
      std::conditional_t<use_mx_scale, __nv_fp8_e8m0, __hip_fp8_e4m3_fnuz>;
  auto scale = float(((ScaleType*)(scales))[gindex]);

  out += oindex;

  uint val = w[offset];
#pragma clang loop unroll(full)
  for (int i = 0; i < pack_factor; i++) {
    uint8_t d;
    if (bits == 4) {
      d = (val >> (bits * i)) & 0x0f;
    } else if (bits == 8) {
      d = val;
    }
    out[i] = static_cast<T>(scale * Dequantize<bits>{}(d));
  }
}

} // namespace cu

void fp_quantize(
    const array& w,
    array& wq,
    array& scales,
    int group_size,
    int bits,
    cu::CommandEncoder& enc,
    const Stream& s) {
  enc.set_input_array(w);
  enc.set_output_array(wq);
  enc.set_output_array(scales);
  dispatch_float_types(w.dtype(), "fp_quantize", [&](auto type_tag) {
    using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    if constexpr (!std::is_same_v<T, double>) {
      auto kernel = cu::fp_quantize<T, 32, 4, true>;
      if (bits == 8) {
        kernel = cu::fp_quantize<T, 32, 8, true>;
      } else if (group_size == 16) {
        kernel = cu::fp_quantize<T, 16, 4, false>;
      }
      bool large = w.size() > UINT_MAX;
      auto [num_blocks, block_dims] =
          get_launch_args(w.size(), w.shape(), w.strides(), large);
      enc.add_kernel_node(
          kernel,
          num_blocks,
          block_dims,
          0,
          w.data<T>(),
          wq.data<uint8_t>(),
          scales.data<uint8_t>(),
          w.size());
    } else {
      throw std::runtime_error(
          "[Quantize::eval_gpu] Can not quantize input with type float64.");
    }
  });
}

void fp_dequantize(
    const array& wq,
    const array& scales,
    array& w,
    int group_size,
    int bits,
    cu::CommandEncoder& enc,
    const Stream& s) {
  constexpr int uint8_per_uint32 = 4;
  int packs_per_int = 8 / bits;

  size_t size = w.size() / packs_per_int;
  bool large = size > UINT_MAX;
  auto grid_shape = w.shape();
  grid_shape.back() *= uint8_per_uint32;

  enc.set_input_array(wq);
  enc.set_input_array(scales);
  enc.set_output_array(w);
  dispatch_float_types(w.dtype(), "fp_dequantize", [&](auto type_tag) {
    using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    if constexpr (!std::is_same_v<T, double>) {
      auto kernel = cu::fp_dequantize<T, 32, 4, true>;
      if (bits == 8) {
        kernel = cu::fp_dequantize<T, 32, 8, true>;
      } else if (group_size == 16) {
        kernel = cu::fp_dequantize<T, 16, 4, false>;
      }
      auto [num_blocks, block_dims] =
          get_launch_args(size, grid_shape, w.strides(), large);
      enc.add_kernel_node(
          kernel,
          num_blocks,
          block_dims,
          0,
          wq.data<uint8_t>(),
          scales.data<T>(),
          w.data<T>(),
          w.size());
    } else {
      throw std::runtime_error(
          "[Quantize::eval_gpu] Can not dequantize to output with type float64.");
    }
  });
}

} // namespace mlx::core
