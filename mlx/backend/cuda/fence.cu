#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/fence.h"
#include "mlx/scheduler.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace {

__host__ __device__ void busy_wait(cuda::atomic<uint64_t>* ac, uint64_t value) {
  while (true) {
    // In theory the atomic_thread_fence is not needed, but for CUDA 11 without
    // it the load() may never return new value.
    cuda::atomic_thread_fence(cuda::memory_order_seq_cst);
    uint64_t current = ac->load();
    if (current >= value) {
      break;
    }
  }
}

__global__ void busy_wait_kernel(cuda::atomic<uint64_t>* ac, uint64_t value) {
  busy_wait(ac, value);
}

} // namespace

struct FenceImpl {
  uint32_t count;
  cu::SharedEvent event;
};

Fence::Fence(Stream s) {
  fence_ = std::shared_ptr<void>(
      new FenceImpl{0}, [](void* ptr) { delete static_cast<FenceImpl*>(ptr); });
}

void Fence::wait(Stream s, const array&) {
  auto* fence = static_cast<FenceImpl*>(fence_.get());
  // We can't use SharedEvent::wait because it could hang in CUDA 11, see also:
  // https://github.com/ml-explore/mlx/issues/2137
  const auto& ac = fence->event.atomic();
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [ac, count = fence->count]() {
      nvtx3::scoped_range r("Fence::wait()");
      busy_wait(ac.get(), count);
    });
  } else {
    nvtx3::scoped_range r("Fence::wait(s)");
    auto& encoder = cu::get_command_encoder(s);
    encoder.launch_kernel(
        encoder.stream().last_cuda_stream(), [&](hipStream_t stream) {
          busy_wait_kernel<<<1, 1, 0>>>(ac.get(), fence->count);
        });
    encoder.add_completed_handler([ac]() {});
    encoder.end_encoding();
  }
}

void Fence::update(Stream s, const array&) {
  auto* fence = static_cast<FenceImpl*>(fence_.get());
  fence->count++;
  fence->event.signal(s, fence->count);
}

} // namespace mlx::core
