#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT, int NDIM>
__global__ void copy_gg_nd(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_in,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_out) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [idx_in, idx_out] = elem_to_loc_nd<NDIM>(
        index, shape.data(), strides_in.data(), strides_out.data());
    out[idx_out] = CastOp<In, Out>{}(in[idx_in]);
  }
}

template <typename In, typename Out, typename IdxT>
__global__ void copy_gg(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides_in,
    const __grid_constant__ Strides strides_out,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [idx_in, idx_out] = elem_to_loc(
        index, shape.data(), strides_in.data(), strides_out.data(), ndim);
    out[idx_out] = CastOp<In, Out>{}(in[idx_in]);
  }
}

} // namespace cu

void copy_general(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t offset_in,
    int64_t offset_out,
    const Shape& shape,
    const Strides& strides_in,
    const Strides& strides_out) {
  dispatch_all_types(in.dtype(), [&](auto in_type_tag) {
    dispatch_all_types(out.dtype(), [&](auto out_type_tag) {
      dispatch_bool(
          in.data_size() > INT32_MAX || out.data_size() > INT32_MAX,
          [&](auto large) {
            using InType = cuda_type_t<MLX_GET_TYPE(in_type_tag)>;
            using OutType = cuda_type_t<MLX_GET_TYPE(out_type_tag)>;
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            const InType* in_ptr = in.data<InType>() + offset_in;
            OutType* out_ptr = out.data<OutType>() + offset_out;
            int ndim = shape.size();
            size_t data_size = 1;
            for (auto& s : shape)
              data_size *= s;
            if (ndim <= 3) {
              dispatch_1_2_3(ndim, [&](auto ndim_constant) {
                auto [num_blocks, block_dims] =
                    get_launch_args(data_size, shape, out.strides(), large());
                encoder.add_kernel_node(
                    cu::copy_gg_nd<InType, OutType, IdxT, ndim_constant()>,
                    num_blocks,
                    block_dims,
                    0,
                    in_ptr,
                    out_ptr,
                    data_size,
                    const_param<ndim_constant()>(shape),
                    const_param<ndim_constant()>(strides_in),
                    const_param<ndim_constant()>(strides_out));
              });
            } else { // ndim >= 4
              auto [num_blocks, block_dims] =
                  get_launch_args(data_size, shape, out.strides(), large());
              encoder.add_kernel_node(
                  cu::copy_gg<InType, OutType, IdxT>,
                  num_blocks,
                  block_dims,
                  0,
                  in_ptr,
                  out_ptr,
                  data_size,
                  const_param(shape),
                  const_param(strides_in),
                  const_param(strides_out),
                  ndim);
            }
          });
    });
  });
}

} // namespace mlx::core
