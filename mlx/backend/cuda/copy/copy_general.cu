#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT, int NDIM, int N_READS>
__global__ void copy_gg_nd(
    const In* in,
    Out* out,
    IdxT size_rest,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_in,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_out) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[NDIM - 1];
  auto in_stride_x = strides_in[NDIM - 1];
  auto out_stride_x = strides_out[NDIM - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto [idx_in, idx_out] = elem_to_loc_nd<NDIM>(
      index_rest * shape_x,
      shape.data(),
      strides_in.data(),
      strides_out.data());

  auto in_vec =
      load_vector<N_READS>(in + idx_in, index_x, shape_x, in_stride_x, In(0));
  AlignedVector<Out, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = CastOp<In, Out>{}(in_vec[i]);
  }
  store_vector(out + idx_out, index_x, out_vec, shape_x, out_stride_x);
}

template <typename In, typename Out, typename IdxT, int N_READS>
__global__ void copy_gg(
    const In* in,
    Out* out,
    IdxT size_rest,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides_in,
    const __grid_constant__ Strides strides_out,
    int ndim) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[ndim - 1];
  auto in_stride_x = strides_in[ndim - 1];
  auto out_stride_x = strides_out[ndim - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto [idx_in, idx_out] = elem_to_loc(
      index_rest * shape_x,
      shape.data(),
      strides_in.data(),
      strides_out.data(),
      ndim);

  auto in_vec =
      load_vector<N_READS>(in + idx_in, index_x, shape_x, in_stride_x, In(0));
  AlignedVector<Out, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = CastOp<In, Out>{}(in_vec[i]);
  }
  store_vector(out + idx_out, index_x, out_vec, shape_x, out_stride_x);
}

} // namespace cu

void copy_general(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t offset_in,
    int64_t offset_out,
    const Shape& shape,
    const Strides& strides_in,
    const Strides& strides_out) {
  dispatch_all_types(in.dtype(), [&](auto in_type_tag) {
    dispatch_all_types(out.dtype(), [&](auto out_type_tag) {
      dispatch_bool(
          in.data_size() > INT32_MAX || out.data_size() > INT32_MAX,
          [&](auto large) {
            using InType = cuda_type_t<MLX_GET_TYPE(in_type_tag)>;
            using OutType = cuda_type_t<MLX_GET_TYPE(out_type_tag)>;
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            const InType* in_ptr = in.data<InType>() + offset_in;
            OutType* out_ptr = out.data<OutType>() + offset_out;
            int ndim = shape.size();
            size_t data_size = 1;
            for (auto& s : shape)
              data_size *= s;

            int work_per_thread = 1;
            auto dim0 = ndim > 0 ? shape.back() : 1;
            auto rest = data_size / dim0;
            if (dim0 >= 4) {
              work_per_thread = 4;
            }

            dim0 = (dim0 + work_per_thread - 1) / work_per_thread;
            auto block_dims = get_block_dims(dim0, rest, 1);
            uint32_t num_blocks_x = cuda::ceil_div(dim0, block_dims.x);
            uint32_t num_blocks_y = cuda::ceil_div(rest, block_dims.y);

            if (ndim <= 3) {
              dispatch_1_2_3(ndim, [&](auto ndim_constant) {
                auto kernel =
                    cu::copy_gg_nd<InType, OutType, IdxT, ndim_constant(), 1>;
                if (work_per_thread == 4) {
                  kernel =
                      cu::copy_gg_nd<InType, OutType, IdxT, ndim_constant(), 4>;
                }
                encoder.add_kernel_node(
                    kernel,
                    {num_blocks_x, num_blocks_y},
                    block_dims,
                    0,
                    in_ptr,
                    out_ptr,
                    rest,
                    const_param<ndim_constant()>(shape),
                    const_param<ndim_constant()>(strides_in),
                    const_param<ndim_constant()>(strides_out));
              });
            } else { // ndim >= 4
              auto kernel = cu::copy_gg<InType, OutType, IdxT, 1>;
              if (work_per_thread == 4) {
                kernel = cu::copy_gg<InType, OutType, IdxT, 4>;
              }
              encoder.add_kernel_node(
                  kernel,
                  {num_blocks_x, num_blocks_y},
                  block_dims,
                  0,
                  in_ptr,
                  out_ptr,
                  rest,
                  const_param(shape),
                  const_param(strides_in),
                  const_param(strides_out),
                  ndim);
            }
          });
    });
  });
}

} // namespace mlx::core
