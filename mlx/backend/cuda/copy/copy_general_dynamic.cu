#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT, int NDIM>
__global__ void copy_gg_dynamic_nd(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_in,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_out,
    const int64_t* offset_in,
    const int64_t* offset_out) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [idx_in, idx_out] = elem_to_loc_nd<NDIM>(
        index, shape.data(), strides_in.data(), strides_out.data());
    out[idx_out + *offset_out] = CastOp<In, Out>{}(in[idx_in + *offset_in]);
  }
}

template <typename In, typename Out, typename IdxT>
__global__ void copy_gg_dynamic(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides_in,
    const __grid_constant__ Strides strides_out,
    int ndim,
    const int64_t* offset_in,
    const int64_t* offset_out) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    auto [idx_in, idx_out] = elem_to_loc_4d(
        index, shape.data(), strides_in.data(), strides_out.data(), ndim);
    out[idx_out + *offset_out] = CastOp<In, Out>{}(in[idx_in + *offset_in]);
  }
}

} // namespace cu

void copy_general_dynamic(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t offset_in,
    int64_t offset_out,
    const Shape& shape,
    const Strides& strides_in,
    const Strides& strides_out,
    const array& dynamic_offset_in,
    const array& dynamic_offset_out) {
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_COPY_TYPES(in, out, InType, OutType, {
      const InType* in_ptr = in.data<InType>() + offset_in;
      OutType* out_ptr = out.data<OutType>() + offset_out;
      bool large = in.data_size() > UINT32_MAX || out.data_size() > UINT32_MAX;
      MLX_SWITCH_BOOL(large, LARGE, {
        using IdxT = std::conditional_t<LARGE, int64_t, uint32_t>;
        int ndim = shape.size();
        if (ndim <= 3) {
          MLX_SWITCH_1_2_3(ndim, NDIM, {
            auto kernel = cu::copy_gg_dynamic_nd<InType, OutType, IdxT, NDIM>;
            auto [num_blocks, block_dims] = get_launch_args(kernel, out, large);
            kernel<<<num_blocks, block_dims, 0, stream>>>(
                in_ptr,
                out_ptr,
                out.data_size(),
                const_param<NDIM>(shape),
                const_param<NDIM>(strides_in),
                const_param<NDIM>(strides_out),
                dynamic_offset_in.data<int64_t>(),
                dynamic_offset_out.data<int64_t>());
          });
        } else { // ndim >= 4
          auto kernel = cu::copy_gg_dynamic<InType, OutType, IdxT>;
          auto [num_blocks, block_dims] = get_launch_args(kernel, out, large);
          kernel<<<num_blocks, block_dims, 0, stream>>>(
              in_ptr,
              out_ptr,
              out.data_size(),
              const_param(shape),
              const_param(strides_in),
              const_param(strides_out),
              ndim,
              dynamic_offset_in.data<int64_t>(),
              dynamic_offset_out.data<int64_t>());
        }
      });
    });
  });
}

} // namespace mlx::core
