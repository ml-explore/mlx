#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT>
__global__ void copy_s(const In* in, Out* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    out[index] = CastOp<In, Out>{}(in[0]);
  }
}

template <typename In, typename Out, typename IdxT>
__global__ void copy_v(const In* in, Out* out, IdxT size) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    out[index] = CastOp<In, Out>{}(in[index]);
  }
}

} // namespace cu

void copy_contiguous(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t in_offset,
    int64_t out_offset) {
  encoder.launch_kernel([&](hipStream_t stream) {
    dispatch_all_types(in.dtype(), [&](auto in_type_tag) {
      dispatch_all_types(out.dtype(), [&](auto out_type_tag) {
        dispatch_bool(out.data_size() > INT32_MAX, [&](auto large) {
          using InType = cuda_type_t<MLX_GET_TYPE(in_type_tag)>;
          using OutType = cuda_type_t<MLX_GET_TYPE(out_type_tag)>;
          using IdxT = std::conditional_t<large(), int64_t, uint32_t>;
          auto kernel = cu::copy_s<InType, OutType, IdxT>;
          if (ctype == CopyType::Vector) {
            kernel = cu::copy_v<InType, OutType, IdxT>;
          }
          auto [num_blocks, block_dims] = get_launch_args(
              kernel, out.data_size(), out.shape(), out.strides(), large());
          kernel<<<num_blocks, block_dims, 0, stream>>>(
              in.data<InType>() + in_offset,
              out.data<OutType>() + out_offset,
              out.data_size());
        });
      });
    });
  });
}

} // namespace mlx::core
