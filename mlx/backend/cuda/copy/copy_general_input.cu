#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT, int NDIM>
__global__ void copy_g_nd(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides_in) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    IdxT idx_in = elem_to_loc_nd<NDIM>(index, shape.data(), strides_in.data());
    out[index] = CastOp<In, Out>{}(in[idx_in]);
  }
}

template <typename In, typename Out, typename IdxT>
__global__ void copy_g(
    const In* in,
    Out* out,
    IdxT size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides_in,
    int ndim) {
  IdxT index = cg::this_grid().thread_rank();
  if (index < size) {
    IdxT idx_in = elem_to_loc_4d(index, shape.data(), strides_in.data(), ndim);
    out[index] = CastOp<In, Out>{}(in[idx_in]);
  }
}

} // namespace cu

void copy_general_input(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t offset_in,
    int64_t offset_out,
    const Shape& shape,
    const Strides& strides_in) {
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_COPY_TYPES(in, out, InType, OutType, {
      const InType* in_ptr = in.data<InType>() + offset_in;
      OutType* out_ptr = out.data<OutType>() + offset_out;
      bool large = in.data_size() > UINT32_MAX || out.data_size() > UINT32_MAX;
      MLX_SWITCH_BOOL(large, LARGE, {
        using IdxT = std::conditional_t<LARGE, int64_t, uint32_t>;
        int ndim = shape.size();
        if (ndim <= 3) {
          MLX_SWITCH_1_2_3(ndim, NDIM, {
            auto kernel = cu::copy_g_nd<InType, OutType, IdxT, NDIM>;
            auto [num_blocks, block_dims] = get_launch_args(kernel, out, large);
            kernel<<<num_blocks, block_dims, 0, stream>>>(
                in_ptr,
                out_ptr,
                out.data_size(),
                const_param<NDIM>(shape),
                const_param<NDIM>(strides_in));
          });
        } else { // ndim >= 4
          auto kernel = cu::copy_g<InType, OutType, IdxT>;
          auto [num_blocks, block_dims] = get_launch_args(kernel, out, large);
          kernel<<<num_blocks, block_dims, 0, stream>>>(
              in_ptr,
              out_ptr,
              out.data_size(),
              const_param(shape),
              const_param(strides_in),
              ndim);
        }
      });
    });
  });
}

} // namespace mlx::core
