#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/copy/copy.cuh"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename In, typename Out, typename IdxT, int NDIM, int N_READS>
__global__ void copy_g_nd(
    const In* in,
    Out* out,
    IdxT size_rest,
    const __grid_constant__ cuda::std::array<int32_t, NDIM> shape,
    const __grid_constant__ cuda::std::array<int64_t, NDIM> strides) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[NDIM - 1];
  auto stride_x = strides[NDIM - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto idx =
      elem_to_loc_nd<NDIM>(index_rest * shape_x, shape.data(), strides.data());
  auto in_vec =
      load_vector<N_READS>(in + idx, index_x, shape_x, stride_x, In(0));
  AlignedVector<Out, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = CastOp<In, Out>{}(in_vec[i]);
  }
  store_vector(out + shape_x * index_rest, index_x, out_vec, shape_x);
}

template <typename In, typename Out, typename IdxT, int N_READS>
__global__ void copy_g(
    const In* in,
    Out* out,
    IdxT size_rest,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides strides,
    int ndim) {
  auto block = cg::this_thread_block();
  auto grid = cg::this_grid();
  IdxT index_rest =
      grid.block_index().y * block.dim_threads().y + block.thread_index().y;
  if (index_rest >= size_rest) {
    return;
  }

  auto shape_x = shape[ndim - 1];
  auto stride_x = strides[ndim - 1];
  IdxT index_x =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  auto idx =
      elem_to_loc(index_rest * shape_x, shape.data(), strides.data(), ndim);
  auto in_vec =
      load_vector<N_READS>(in + idx, index_x, shape_x, stride_x, In(0));
  AlignedVector<Out, N_READS> out_vec;
#pragma unroll
  for (int i = 0; i < N_READS; ++i) {
    out_vec[i] = CastOp<In, Out>{}(in_vec[i]);
  }
  store_vector(out + shape_x * index_rest, index_x, out_vec, shape_x);
}

} // namespace cu

void copy_general_input(
    cu::CommandEncoder& encoder,
    CopyType ctype,
    const array& in,
    array& out,
    int64_t offset_in,
    int64_t offset_out,
    const Shape& shape,
    const Strides& strides_in) {
  dispatch_all_types(in.dtype(), [&](auto in_type_tag) {
    dispatch_all_types(out.dtype(), [&](auto out_type_tag) {
      dispatch_bool(
          in.data_size() > INT32_MAX || out.data_size() > INT32_MAX,
          [&](auto large) {
            using InType = cuda_type_t<MLX_GET_TYPE(in_type_tag)>;
            using OutType = cuda_type_t<MLX_GET_TYPE(out_type_tag)>;
            using IdxT = std::conditional_t<large(), int64_t, int32_t>;
            const InType* in_ptr = in.data<InType>() + offset_in;
            OutType* out_ptr = out.data<OutType>() + offset_out;
            int ndim = shape.size();
            int work_per_thread = 1;
            auto dim0 = ndim > 0 ? shape.back() : 1;
            auto rest = out.size() / dim0;
            if (dim0 >= 4) {
              work_per_thread = 4;
            }
            dim0 = (dim0 + work_per_thread - 1) / work_per_thread;
            auto block_dims = get_block_dims(dim0, rest, 1);
            uint32_t num_blocks_x = cuda::ceil_div(dim0, block_dims.x);
            uint32_t num_blocks_y = cuda::ceil_div(rest, block_dims.y);

            if (ndim <= 3) {
              dispatch_1_2_3(ndim, [&](auto dims_constant) {
                auto kernel =
                    cu::copy_g_nd<InType, OutType, IdxT, dims_constant(), 1>;
                if (work_per_thread == 4) {
                  kernel =
                      cu::copy_g_nd<InType, OutType, IdxT, dims_constant(), 4>;
                }
                encoder.add_kernel_node(
                    kernel,
                    {num_blocks_x, num_blocks_y},
                    block_dims,
                    0,
                    in_ptr,
                    out_ptr,
                    rest,
                    const_param<dims_constant()>(shape),
                    const_param<dims_constant()>(strides_in));
              });
            } else { // ndim >= 4
              auto kernel = cu::copy_g<InType, OutType, IdxT, 1>;
              if (work_per_thread == 4) {
                kernel = cu::copy_g<InType, OutType, IdxT, 4>;
              }
              encoder.add_kernel_node(
                  kernel,
                  {num_blocks_x, num_blocks_y},
                  block_dims,
                  0,
                  in_ptr,
                  out_ptr,
                  rest,
                  const_param(shape),
                  const_param(strides_in),
                  ndim);
            }
          });
    });
  });
}

} // namespace mlx::core
