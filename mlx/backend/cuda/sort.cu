// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/utils.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <cub/device/device_segmented_sort.cuh>

#include <cassert>
#include <numeric>

namespace mlx::core {

namespace {

template <typename T>
struct ModOp {
  T divisor;
  __device__ T operator()(T x) {
    return x % divisor;
  }
};

// We can not use any op in eval, make an utility.
array swapaxes_in_eval(const array& in, int axis1, int axis2) {
  std::vector<int> axes(in.ndim());
  std::iota(axes.begin(), axes.end(), 0);
  std::swap(axes[axis1], axes[axis2]);
  // TODO: Share the code with Transpose::eval.
  Shape shape(axes.size());
  Strides strides(in.ndim());
  for (size_t ax = 0; ax < axes.size(); ++ax) {
    shape[ax] = in.shape()[axes[ax]];
    strides[ax] = in.strides()[axes[ax]];
  }
  auto flags = in.flags();
  if (flags.contiguous) {
    auto [_, row_contiguous, col_contiguous] = check_contiguity(shape, strides);
    flags.row_contiguous = row_contiguous;
    flags.col_contiguous = col_contiguous;
  }
  array out(shape, in.dtype(), nullptr, {});
  out.copy_shared_buffer(in, strides, flags, in.data_size());
  return out;
}

template <typename... Args>
void segmented_sort_pairs(cu::CommandEncoder& encoder, Args&&... args) {
  // Allocate temporary storage.
  size_t size;
  CHECK_CUDA_ERROR(
      hipcub::DeviceSegmentedSort::StableSortPairs(nullptr, size, args...));
  array temp(allocator::malloc(size), {static_cast<int>(size)}, uint8);
  encoder.add_temporary(temp);
  // Run op.
  CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortPairs(
      temp.data<void>(), size, args...));
}

template <typename... Args>
void segmented_sort(cu::CommandEncoder& encoder, Args&&... args) {
  // Allocate temporary storage.
  size_t size;
  CHECK_CUDA_ERROR(
      hipcub::DeviceSegmentedSort::StableSortKeys(nullptr, size, args...));
  array temp(allocator::malloc(size), {static_cast<int>(size)}, uint8);
  encoder.add_temporary(temp);
  // Run op.
  CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortKeys(
      temp.data<void>(), size, args...));
}

void gpu_sort(const Stream& s, array in, array& out_, int axis, bool argsort) {
  array out = out_;
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);

  if (axis < 0) {
    axis += in.ndim();
  }
  int nsort = in.shape(axis);
  int nsegments = in.data_size() / nsort;
  int last_dim = in.ndim() - 1;

  // If we are not sorting the innermost dimension of a contiguous array,
  // transpose and make a copy.
  bool is_segmented_sort = in.flags().contiguous && in.strides()[axis] == 1;
  if (!is_segmented_sort) {
    array trans = swapaxes_in_eval(in, axis, last_dim);
    in = array(trans.shape(), trans.dtype(), nullptr, {});
    copy_gpu(trans, in, CopyType::General, s);
    encoder.add_temporary(in);
    out = array(allocator::malloc(out.nbytes()), in.shape(), out.dtype());
    encoder.add_temporary(out);
  } else {
    out.set_data(allocator::malloc(out.nbytes()));
  }

  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(in.dtype(), CTYPE, {
      if constexpr (!std::is_same_v<CTYPE, complex64_t>) {
        using Type = cuda_type_t<CTYPE>;
        auto offsets = thrust::make_transform_iterator(
            thrust::make_counting_iterator(0),
            [nsort] __device__(int i) { return i * nsort; });
        if (argsort) {
          // Indices in the sorted dimension.
          array indices(
              allocator::malloc(out.nbytes()), in.shape(), out.dtype());
          encoder.add_temporary(indices);
          thrust::transform(
              cu::thrust_policy(stream),
              thrust::counting_iterator<uint32_t>(0),
              thrust::counting_iterator<uint32_t>(indices.data_size()),
              thrust::device_pointer_cast(indices.data<uint32_t>()),
              ModOp<uint32_t>{static_cast<uint32_t>(nsort)});

          // In argsort though we don't need the result of sorted values, the
          // API requires us to provide an array to store it.
          array discard(allocator::malloc(in.nbytes()), in.shape(), in.dtype());
          encoder.add_temporary(discard);

          segmented_sort_pairs(
              encoder,
              in.data<Type>(),
              discard.data<Type>(),
              indices.data<uint32_t>(),
              out.data<uint32_t>(),
              in.data_size(),
              nsegments,
              offsets,
              offsets + 1,
              stream);
        } else {
          segmented_sort(
              encoder,
              in.data<Type>(),
              out.data<Type>(),
              in.data_size(),
              nsegments,
              offsets,
              offsets + 1,
              stream);
        }
      } else {
        throw std::runtime_error(
            "CUDA backend does not support sorting complex numbers");
      }
    });
  });

  if (!is_segmented_sort) {
    // Swap the sorted axis back.
    // TODO: Do in-place transpose instead of using a temporary out array.
    copy_gpu(swapaxes_in_eval(out, axis, last_dim), out_, CopyType::General, s);
  }
}

} // namespace

void ArgSort::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgSort::eval_gpu");
  assert(inputs.size() == 1);
  gpu_sort(stream(), inputs[0], out, axis_, true);
}

void Sort::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Sort::eval_gpu");
  assert(inputs.size() == 1);
  gpu_sort(stream(), inputs[0], out, axis_, false);
}

} // namespace mlx::core
