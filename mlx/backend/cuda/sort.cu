// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/utils.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <cub/device/device_segmented_sort.cuh>

#include <cassert>

namespace mlx::core {

namespace {

template <typename T>
struct ModOp {
  T divisor;
  __device__ T operator()(T x) {
    return x % divisor;
  }
};

struct OffsetTransform {
  int nsort;

  int __device__ operator()(int i) {
    return i * nsort;
  }
};

void gpu_sort(const Stream& s, array in, array& out_, int axis, bool argsort) {
  array out = out_;
  auto& encoder = cu::get_command_encoder(s);
  if (axis < 0) {
    axis += in.ndim();
  }
  int nsort = in.shape(axis);
  int last_dim = in.ndim() - 1;

  // If we are not sorting the innermost dimension of a contiguous array,
  // transpose and make a copy.
  bool is_segmented_sort = in.flags().contiguous && in.strides()[axis] == 1;
  if (!is_segmented_sort) {
    array trans = swapaxes_in_eval(in, axis, last_dim);
    in = contiguous_copy_gpu(trans, s);
    encoder.add_temporary(in);
    out = array(allocator::malloc(out.nbytes()), in.shape(), out.dtype());
    encoder.add_temporary(out);
  } else {
    out.set_data(
        allocator::malloc(in.data_size() * out.itemsize()),
        in.data_size(),
        in.strides(),
        in.flags());
  }

  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    using CTYPE = MLX_GET_TYPE(type_tag);
    auto& stream = encoder.stream();
    if constexpr (!std::is_same_v<CTYPE, complex64_t>) {
      using Type = cuda_type_t<CTYPE>;
      auto offsets = thrust::make_transform_iterator(
          thrust::make_counting_iterator(0), OffsetTransform{nsort});
      if (argsort) {
        // Indices in the sorted dimension.
        array indices(allocator::malloc(out.nbytes()), in.shape(), out.dtype());
        encoder.add_temporary(indices);

        // In argsort though we don't need the result of sorted values, the
        // API requires us to provide an array to store it.
        array discard(allocator::malloc(in.nbytes()), in.shape(), in.dtype());
        encoder.add_temporary(discard);

        size_t size;
        CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortPairs(
            nullptr,
            size,
            in.data<Type>(),
            discard.data<Type>(),
            indices.data<uint32_t>(),
            out.data<uint32_t>(),
            in.data_size(),
            in.data_size() / nsort,
            offsets,
            offsets + 1,
            stream));

        array temp(allocator::malloc(size), {static_cast<int>(size)}, uint8);
        encoder.add_temporary(temp);

        // Start capturing after allocations
        auto capture = encoder.capture_context();
        thrust::transform(
            cu::thrust_policy(stream),
            thrust::counting_iterator<uint32_t>(0),
            thrust::counting_iterator<uint32_t>(indices.data_size()),
            thrust::device_pointer_cast(indices.data<uint32_t>()),
            ModOp<uint32_t>{static_cast<uint32_t>(nsort)});

        CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortPairs(
            temp.data<void>(),
            size,
            in.data<Type>(),
            discard.data<Type>(),
            indices.data<uint32_t>(),
            out.data<uint32_t>(),
            in.data_size(),
            in.data_size() / nsort,
            offsets,
            offsets + 1,
            stream));
      } else {
        size_t size;
        CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortKeys(
            nullptr,
            size,
            in.data<Type>(),
            out.data<Type>(),
            in.data_size(),
            in.data_size() / nsort,
            offsets,
            offsets + 1,
            stream));

        array temp(allocator::malloc(size), {static_cast<int>(size)}, uint8);
        encoder.add_temporary(temp);

        // Start capturing after allocations
        auto capture = encoder.capture_context();
        CHECK_CUDA_ERROR(hipcub::DeviceSegmentedSort::StableSortKeys(
            temp.data<void>(),
            size,
            in.data<Type>(),
            out.data<Type>(),
            in.data_size(),
            in.data_size() / nsort,
            offsets,
            offsets + 1,
            stream));
      }
    } else {
      throw std::runtime_error(
          "CUDA backend does not support sorting complex numbers");
    }
  });

  if (!is_segmented_sort) {
    // Swap the sorted axis back.
    // TODO: Do in-place transpose instead of using a temporary out array.
    copy_gpu(swapaxes_in_eval(out, axis, last_dim), out_, CopyType::General, s);
  }
}

} // namespace

void ArgSort::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgSort::eval_gpu");
  assert(inputs.size() == 1);
  gpu_sort(stream(), inputs[0], out, axis_, true);
}

void Sort::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Sort::eval_gpu");
  assert(inputs.size() == 1);
  gpu_sort(stream(), inputs[0], out, axis_, false);
}

void ArgPartition::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgPartition::eval_gpu");
  gpu_sort(stream(), inputs[0], out, axis_, true);
}

void Partition::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Partition::eval_gpu");
  gpu_sort(stream(), inputs[0], out, axis_, false);
}

} // namespace mlx::core
