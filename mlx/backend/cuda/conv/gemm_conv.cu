#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/conv/conv.h"
#include "mlx/backend/cuda/gemms/cublas_gemm.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"

#include <hip/hip_cooperative_groups.h>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename T, int NDIM>
__global__ void naive_unfold_nd(
    const T* in,
    T* out,
    int filter_size,
    int out_pixels,
    const __grid_constant__ ConvParams<NDIM> params) {
  auto block = cg::this_thread_block();
  auto tid = block.group_index();
  auto lid = block.thread_index();

  int index_batch = tid.z / out_pixels; // [0, N)
  int index_out_spatial = tid.z % out_pixels; // [0, H_out * W_out)
  int index_wt_spatial =
      tid.x * block.dim_threads().x + lid.x; // [0, H_wt * W_wt)

  if (index_wt_spatial >= filter_size / params.C) {
    return;
  }

  in += tid.y; // [0, C)
  out += tid.z * filter_size + index_wt_spatial * params.C + tid.y;

  bool valid = index_batch < params.N;

  // Get the coordinates in input.
  int index_in[NDIM] = {};
#pragma unroll
  for (int i = NDIM - 1; i >= 0; --i) {
    int index_out = index_out_spatial % params.out_spatial_dims[i];
    int index_wt = index_wt_spatial % params.wt_spatial_dims[i];

    if (params.flip) {
      index_wt = params.wt_spatial_dims[i] - index_wt - 1;
    }

    int index = index_out * params.strides[i] - params.padding[i] +
        index_wt * params.kernel_dilation[i];
    int index_max =
        1 + params.input_dilation[i] * (params.in_spatial_dims[i] - 1);

    valid &= (index >= 0) && (index < index_max) &&
        (index % params.input_dilation[i] == 0);

    index_in[i] = index / params.input_dilation[i];

    index_out_spatial /= params.out_spatial_dims[i];
    index_wt_spatial /= params.wt_spatial_dims[i];
  }

  if (valid) {
    int in_offset = index_batch * params.in_strides[0];
#pragma unroll
    for (int i = 0; i < NDIM; ++i) {
      in_offset += index_in[i] * params.in_strides[i + 1];
    }
    *out = in[in_offset];
  } else {
    *out = T{0};
  }
}

} // namespace cu

template <int NDIM>
array unfold_inputs_nd(
    cu::CommandEncoder& encoder,
    const array& in,
    int mat_M,
    int mat_K,
    int mat_N,
    ConvParams<NDIM>& params) {
  array unfolded({mat_M, mat_K}, in.dtype(), nullptr, {});
  unfolded.set_data(allocator::malloc(unfolded.nbytes()));
  encoder.add_temporary(unfolded);

  int filter_size = params.C;
#pragma unroll
  for (int i = 0; i < NDIM; ++i) {
    filter_size *= params.wt_spatial_dims[i];
  }

  int out_pixels = 1;
#pragma unroll
  for (int i = 0; i < NDIM; ++i) {
    out_pixels *= params.out_spatial_dims[i];
  }

  int wt_spatial_size = mat_K / params.C;
  dim3 block_dims;
  block_dims.x = std::min(std::max(wt_spatial_size, 32), 1024);
  dim3 num_blocks;
  num_blocks.x = cuda::ceil_div(wt_spatial_size, block_dims.x);
  num_blocks.y = params.C;
  num_blocks.z = mat_M;

  encoder.set_input_array(in);
  encoder.set_output_array(unfolded);
  dispatch_float_types(in.dtype(), "unfold", [&](auto type_tag) {
    using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    encoder.add_kernel_node(
        cu::naive_unfold_nd<DataType, NDIM>,
        num_blocks,
        block_dims,
        0,
        in.data<DataType>(),
        unfolded.data<DataType>(),
        filter_size,
        out_pixels,
        params);
  });

  return unfolded;
}

template <int NDIM>
void gemm_conv_nd(
    cu::CommandEncoder& encoder,
    const array& in,
    const array& wt,
    array& out,
    ConvParams<NDIM>& params,
    Stream s) {
  // Get gemm shapes.
  int mat_M = out.size() / params.O; // N * H_out * W_out
  int mat_K = wt.size() / params.O; // C * H_wt * W_wt
  int mat_N = params.O; // O

  // Unfold input to (N * H_out * W_out, C * H_wt * W_wt) for gemm.
  array in_unfolded =
      unfold_inputs_nd<NDIM>(encoder, in, mat_M, mat_K, mat_N, params);

  // Reshape weight to (C * H_wt * W_wt, O) for gemm.
  array wt_reshaped({mat_K, mat_N}, wt.dtype(), nullptr, {});
  wt_reshaped.copy_shared_buffer(
      wt,
      {1, mat_K},
      {false, false, /* col_contiguous */ true},
      wt.data_size());

  // Single batch.
  Shape batch_shape{1};
  Strides a_batch_strides{0};
  Strides b_batch_strides{0};

  // Run matmul.
  CublasGemm gemm(
      encoder.device(),
      in.dtype(),
      false, // a_transposed
      mat_M, // a_rows
      mat_K, // a_cols
      mat_K, // lda
      true, // b_transposed
      mat_K, // b_rows
      mat_N, // b_cols
      mat_K, // ldb
      batch_shape.back(),
      a_batch_strides.back(),
      b_batch_strides.back());
  gemm.run(
      encoder,
      out,
      in_unfolded,
      wt_reshaped,
      batch_shape,
      a_batch_strides,
      b_batch_strides);
}

void gemm_conv(
    cu::CommandEncoder& encoder,
    const array& in,
    const array& wt,
    array& out,
    const std::vector<int>& strides,
    const std::vector<int>& padding,
    const std::vector<int>& kernel_dilation,
    const std::vector<int>& input_dilation,
    bool flip,
    Stream s) {
  int conv_ndim = in.ndim() - 2;
  if (conv_ndim < 1 || conv_ndim > 3) {
    throw std::runtime_error(
        fmt::format("[conv] Unsupported gemm_conv for {}D conv.", conv_ndim));
  }
  dispatch_1_2_3(conv_ndim, [&](auto ndim_constant) {
    ConvParams<ndim_constant()> params(
        in,
        wt,
        out,
        strides,
        padding,
        kernel_dilation,
        input_dilation,
        1, // groups
        flip);
    gemm_conv_nd<ndim_constant()>(encoder, in, wt, out, params, s);
  });
}

} // namespace mlx::core
