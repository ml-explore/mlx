#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/binary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/reduce/reduce_ops.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include <nvtx3/nvtx3.hpp>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename T>
struct ScanResult {
  using type = T;
};

template <>
struct ScanResult<Sum, bool> {
  using type = int32_t;
};

template <typename T>
struct ReduceInit<LogAddExp, T> {
  static constexpr __host__ __device__ T value() {
    return Limits<T>::min();
  }
};

template <bool reverse, typename T, typename U, int N_READS>
inline __device__ void
load_vals(int index, const T* in, U (&vals)[N_READS], int size, U init) {
  int remaining = size - index * N_READS;
  if constexpr (reverse) {
    in += remaining - N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        vals[N_READS - i - 1] =
            (N_READS - i - 1 < remaining) ? cast_to<U>(in[i]) : init;
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        vals[N_READS - i - 1] = cast_to<U>(in[i]);
      }
    }
  } else {
    in += index * N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        vals[i] = (i < remaining) ? cast_to<U>(in[i]) : init;
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        vals[i] = cast_to<U>(in[i]);
      }
    }
  }
}

template <bool reverse, typename T, int N_READS>
inline __device__ void
store_vals(int index, T* out, T (&vals)[N_READS], int size, int offset = 0) {
  int start = index * N_READS + offset;
  int remaining = size - start;
  if constexpr (reverse) {
    out += remaining - N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        if (N_READS - i - 1 < remaining) {
          out[i] = vals[N_READS - i - 1];
        }
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        out[i] = vals[N_READS - i - 1];
      }
    }
  } else {
    out += start;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        if (i < remaining) {
          out[i] = vals[i];
        }
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        out[i] = vals[i];
      }
    }
  }
}

template <
    typename T,
    typename U,
    typename Op,
    int N_READS,
    bool inclusive,
    bool reverse>
__global__ void contiguous_scan(const T* in, U* out, int32_t axis_size) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  in += grid.block_rank() * axis_size;
  out += grid.block_rank() * axis_size;

  __shared__ U warp_sums[WARP_SIZE];

  Op op;
  U init = ReduceInit<Op, T>::value();
  U prefix = init;

  // Scan per block.
  for (int r = 0; r < cuda::ceil_div(axis_size, block.size() * N_READS); ++r) {
    int32_t index = r * block.size() + block.thread_rank();
    U vals[N_READS];
    load_vals<reverse>(index, in, vals, axis_size, init);

    // Compute an inclusive scan per thread.
    for (int i = 1; i < N_READS; i++) {
      vals[i] = op(vals[i], vals[i - 1]);
    }

    // Compute exclusive scan of thread sums.
    U prev_thread_sum = cg::exclusive_scan(warp, vals[N_READS - 1], op);
    if (warp.thread_rank() == 0) {
      prev_thread_sum = init;
    }

    // Write wrap's sum to shared memory.
    if (warp.thread_rank() == warp.size() - 1) {
      warp_sums[warp.meta_group_rank()] =
          op(prev_thread_sum, vals[N_READS - 1]);
    }
    block.sync();

    // Compute exclusive scan of warp sums.
    if (warp.meta_group_rank() == 0) {
      U prev_warp_sum =
          cg::exclusive_scan(warp, warp_sums[warp.thread_rank()], op);
      if (warp.thread_rank() == 0) {
        prev_warp_sum = init;
      }
      warp_sums[warp.thread_rank()] = prev_warp_sum;
    }
    block.sync();

    // Compute the output.
    for (int i = 0; i < N_READS; ++i) {
      vals[i] = op(vals[i], prefix);
      vals[i] = op(vals[i], warp_sums[warp.meta_group_rank()]);
      vals[i] = op(vals[i], prev_thread_sum);
    }

    // Write the values.
    if (inclusive) {
      store_vals<reverse>(index, out, vals, axis_size);
    } else {
      store_vals<reverse>(index, out, vals, axis_size, 1);
      if (reverse) {
        if (block.thread_rank() == 0 && index == 0) {
          out[axis_size - 1] = init;
        }
      } else {
        if (block.thread_rank() == 0 && index == 0) {
          out[0] = init;
        }
      }
    }
    block.sync();

    // Share the prefix.
    if ((warp.meta_group_rank() == warp.meta_group_size() - 1) &&
        (warp.thread_rank() == warp.size() - 1)) {
      warp_sums[0] = vals[N_READS - 1];
    }
    block.sync();
    prefix = warp_sums[0];
  }
}

} // namespace cu

template <typename F>
void dispatch_scan_ops(Scan::ReduceType scan_op, F&& f) {
  if (scan_op == Scan::ReduceType::Max) {
    f(type_identity<cu::Max>{});
  } else if (scan_op == Scan::ReduceType::Min) {
    f(type_identity<cu::Min>{});
  } else if (scan_op == Scan::ReduceType::Sum) {
    f(type_identity<cu::Sum>{});
  } else if (scan_op == Scan::ReduceType::Prod) {
    f(type_identity<cu::Prod>{});
  } else if (scan_op == Scan::ReduceType::LogAddExp) {
    f(type_identity<cu::LogAddExp>{});
  } else {
    throw std::invalid_argument("Unknown reduce type.");
  }
}

template <typename Op>
const char* op_to_string() {
  if (cuda::std::is_same_v<Op, cu::Max>) {
    return "Max";
  } else if (cuda::std::is_same_v<Op, cu::Min>) {
    return "Min";
  } else if (cuda::std::is_same_v<Op, cu::Sum>) {
    return "Sum";
  } else if (cuda::std::is_same_v<Op, cu::Prod>) {
    return "Prod";
  } else if (cuda::std::is_same_v<Op, cu::LogAddExp>) {
    return "LogAddExp";
  } else {
    throw std::invalid_argument("Unknown op.");
  }
}

template <typename Op, typename T>
constexpr bool supports_scan_op() {
  if constexpr (cuda::std::is_same_v<Op, LogAddExp>) {
    return is_inexact_v<T>;
  } else {
    return true;
  }
}

void Scan::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Scan::eval_gpu");
  assert(inputs.size() == 1);
  auto in = inputs[0];
  auto& s = stream();

  if (in.flags().contiguous && in.strides()[axis_] != 0) {
    if (in.is_donatable() && in.itemsize() == out.itemsize()) {
      out.copy_shared_buffer(in);
    } else {
      out.set_data(
          allocator::malloc(in.data_size() * out.itemsize()),
          in.data_size(),
          in.strides(),
          in.flags());
    }
  } else {
    array arr_copy(in.shape(), in.dtype(), nullptr, {});
    copy_gpu(in, arr_copy, CopyType::General, s);
    in = std::move(arr_copy);
    out.copy_shared_buffer(in);
  }

  bool contiguous = in.strides()[axis_] == 1;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);

  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    dispatch_scan_ops(reduce_type_, [&](auto scan_op_tag) {
      using Op = MLX_GET_TYPE(scan_op_tag);
      if constexpr (supports_scan_op<Op, T>) {
        using U = typename cu::ScanResult<Op, T>::type;
        dispatch_bool(inclusive_, [&](auto inclusive) {
          dispatch_bool(reverse_, [&](auto reverse) {
            if (contiguous) {
              constexpr int N_READS = 4;
              auto kernel = cu::contiguous_scan<
                  T,
                  U,
                  Op,
                  N_READS,
                  inclusive.value,
                  reverse.value>;
              int32_t axis_size = in.shape(axis_);
              int block_dim = cuda::ceil_div(axis_size, N_READS);
              block_dim = cuda::ceil_div(block_dim, WARP_SIZE) * WARP_SIZE;
              block_dim = std::min(block_dim, WARP_SIZE * WARP_SIZE);
              encoder.add_kernel_node(
                  kernel,
                  in.data_size() / axis_size,
                  block_dim,
                  in.data<T>(),
                  out.data<U>(),
                  axis_size);
            } else {
              throw std::runtime_error("Strided Scan NYI");
            }
          });
        });
      } else {
        throw std::runtime_error(fmt::format(
            "Can not do scan op {} on inputs of {} with result of {}.",
            op_to_string<Op>(),
            dtype_to_string(in.dtype()),
            dtype_to_string(out.dtype())));
      }
    });
  });
}

} // namespace mlx::core
