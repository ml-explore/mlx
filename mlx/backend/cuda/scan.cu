#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/binary_ops.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/reduce/reduce_ops.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/scan.h>
#include <nvtx3/nvtx3.hpp>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename Op, typename T>
struct ScanResult {
  using type = T;
};

template <>
struct ScanResult<Sum, bool> {
  using type = int32_t;
};

template <typename T>
struct ReduceInit<LogAddExp, T> {
  static constexpr __host__ __device__ T value() {
    return Limits<T>::min();
  }
};

template <bool reverse, typename T, typename U, int N_READS>
inline __device__ void
load_values(int index, const T* in, U (&values)[N_READS], int size, U init) {
  int remaining = size - index * N_READS;
  if constexpr (reverse) {
    in += remaining - N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        values[N_READS - i - 1] =
            (N_READS - i - 1 < remaining) ? cast_to<U>(in[i]) : init;
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        values[N_READS - i - 1] = cast_to<U>(in[i]);
      }
    }
  } else {
    in += index * N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        values[i] = (i < remaining) ? cast_to<U>(in[i]) : init;
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        values[i] = cast_to<U>(in[i]);
      }
    }
  }
}

template <bool reverse, int offset, typename T, int N_READS>
inline __device__ void
store_values(int index, T* out, T (&values)[N_READS], int size) {
  int start = index * N_READS + offset;
  int remaining = size - start;
  if constexpr (reverse) {
    out += remaining - N_READS;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        if (N_READS - i - 1 < remaining) {
          out[i] = values[N_READS - i - 1];
        }
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        out[i] = values[N_READS - i - 1];
      }
    }
  } else {
    out += start;
    if (remaining < N_READS) {
      for (int i = 0; i < N_READS; ++i) {
        if (i < remaining) {
          out[i] = values[i];
        }
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        out[i] = values[i];
      }
    }
  }
}

template <
    typename T,
    typename U,
    typename Op,
    int N_READS,
    bool inclusive,
    bool reverse>
__global__ void contiguous_scan(const T* in, U* out, int32_t axis_size) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  in += grid.block_rank() * axis_size;
  out += grid.block_rank() * axis_size;

  __shared__ U warp_sums[WARP_SIZE];

  Op op;
  U init = ReduceInit<Op, T>::value();
  U prefix = init;

  // Scan per block.
  for (int r = 0; r < cuda::ceil_div(axis_size, block.size() * N_READS); ++r) {
    int32_t index = r * block.size() + block.thread_rank();
    U values[N_READS];
    load_values<reverse>(index, in, values, axis_size, init);

    // Compute an inclusive scan per thread.
    for (int i = 1; i < N_READS; ++i) {
      values[i] = op(values[i], values[i - 1]);
    }

    // Compute exclusive scan of thread sums.
    U prev_thread_sum = cg::exclusive_scan(warp, values[N_READS - 1], op);
    if (warp.thread_rank() == 0) {
      prev_thread_sum = init;
    }

    // Write wrap's sum to shared memory.
    if (warp.thread_rank() == WARP_SIZE - 1) {
      warp_sums[warp.meta_group_rank()] =
          op(prev_thread_sum, values[N_READS - 1]);
    }
    block.sync();

    // Compute exclusive scan of warp sums.
    if (warp.meta_group_rank() == 0) {
      U prev_warp_sum =
          cg::exclusive_scan(warp, warp_sums[warp.thread_rank()], op);
      if (warp.thread_rank() == 0) {
        prev_warp_sum = init;
      }
      warp_sums[warp.thread_rank()] = prev_warp_sum;
    }
    block.sync();

    // Compute the output.
    for (int i = 0; i < N_READS; ++i) {
      values[i] = op(values[i], prefix);
      values[i] = op(values[i], warp_sums[warp.meta_group_rank()]);
      values[i] = op(values[i], prev_thread_sum);
    }

    // Write the values.
    if (inclusive) {
      store_values<reverse, 0>(index, out, values, axis_size);
    } else {
      store_values<reverse, 1>(index, out, values, axis_size);
      if (reverse) {
        if (block.thread_rank() == 0 && index == 0) {
          out[axis_size - 1] = init;
        }
      } else {
        if (block.thread_rank() == 0 && index == 0) {
          out[0] = init;
        }
      }
    }
    block.sync();

    // Share the prefix.
    if ((warp.meta_group_rank() == warp.meta_group_size() - 1) &&
        (warp.thread_rank() == WARP_SIZE - 1)) {
      warp_sums[0] = values[N_READS - 1];
    }
    block.sync();
    prefix = warp_sums[0];
  }
}

template <
    typename T,
    typename U,
    typename Op,
    int N_READS,
    int BM,
    int BN,
    bool inclusive,
    bool reverse>
__global__ void strided_scan(
    const T* in,
    U* out,
    int32_t axis_size,
    int64_t stride,
    int64_t stride_blocks) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  constexpr int BN_pad = WARP_SIZE + 16 / sizeof(U);
  constexpr int n_warps = BN / N_READS;
  constexpr int n_scans = BN / n_warps;

  __shared__ U read_buffer[BM * BN_pad];

  Op op;
  U init = ReduceInit<Op, T>::value();
  U values[n_scans];
  U prefix[n_scans];
  for (int i = 0; i < n_scans; ++i) {
    prefix[i] = init;
  }

  // Compute offsets.
  int64_t offset = (grid.block_rank() / stride_blocks) * axis_size * stride;
  int64_t global_index_x = (grid.block_rank() % stride_blocks) * BN;
  uint read_offset_y = (block.thread_rank() * N_READS) / BN;
  uint read_offset_x = (block.thread_rank() * N_READS) % BN;
  uint scan_offset_y = warp.thread_rank();
  uint scan_offset_x = warp.meta_group_rank() * n_scans;

  uint stride_limit = stride - global_index_x;
  in += offset + global_index_x + read_offset_x;
  out += offset + global_index_x + read_offset_x;
  U* read_into = read_buffer + read_offset_y * BN_pad + read_offset_x;
  U* read_from = read_buffer + scan_offset_y * BN_pad + scan_offset_x;

  for (uint j = 0; j < axis_size; j += BM) {
    // Calculate the indices for the current thread.
    uint index_y = j + read_offset_y;
    uint check_index_y = index_y;
    if (reverse) {
      index_y = axis_size - 1 - index_y;
    }

    // Read in SM.
    if (check_index_y < axis_size && (read_offset_x + N_READS) < stride_limit) {
      for (int i = 0; i < N_READS; ++i) {
        read_into[i] = in[index_y * stride + i];
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        if (check_index_y < axis_size && (read_offset_x + i) < stride_limit) {
          read_into[i] = in[index_y * stride + i];
        } else {
          read_into[i] = init;
        }
      }
    }
    block.sync();

    // Read strided into registers.
    for (int i = 0; i < n_scans; ++i) {
      values[i] = read_from[i];
    }

    // Perform the scan.
    for (int i = 0; i < n_scans; ++i) {
      values[i] = cg::inclusive_scan(warp, values[i], op);
      values[i] = op(values[i], prefix[i]);
      prefix[i] = warp.shfl(values[i], WARP_SIZE - 1);
    }

    // Write to SM.
    for (int i = 0; i < n_scans; ++i) {
      read_from[i] = values[i];
    }
    block.sync();

    // Write to device memory.
    if (!inclusive) {
      if (check_index_y == 0) {
        if ((read_offset_x + N_READS) < stride_limit) {
          for (int i = 0; i < N_READS; ++i) {
            out[index_y * stride + i] = init;
          }
        } else {
          for (int i = 0; i < N_READS; ++i) {
            if ((read_offset_x + i) < stride_limit) {
              out[index_y * stride + i] = init;
            }
          }
        }
      }
      if (reverse) {
        index_y -= 1;
        check_index_y += 1;
      } else {
        index_y += 1;
        check_index_y += 1;
      }
    }
    if (check_index_y < axis_size && (read_offset_x + N_READS) < stride_limit) {
      for (int i = 0; i < N_READS; ++i) {
        out[index_y * stride + i] = read_into[i];
      }
    } else {
      for (int i = 0; i < N_READS; ++i) {
        if (check_index_y < axis_size && (read_offset_x + i) < stride_limit) {
          out[index_y * stride + i] = read_into[i];
        }
      }
    }
  }
}

} // namespace cu

template <typename F>
void dispatch_scan_ops(Scan::ReduceType scan_op, F&& f) {
  if (scan_op == Scan::ReduceType::Max) {
    f(type_identity<cu::Max>{});
  } else if (scan_op == Scan::ReduceType::Min) {
    f(type_identity<cu::Min>{});
  } else if (scan_op == Scan::ReduceType::Sum) {
    f(type_identity<cu::Sum>{});
  } else if (scan_op == Scan::ReduceType::Prod) {
    f(type_identity<cu::Prod>{});
  } else if (scan_op == Scan::ReduceType::LogAddExp) {
    f(type_identity<cu::LogAddExp>{});
  } else {
    throw std::invalid_argument("Unknown reduce type.");
  }
}

template <typename Op>
const char* op_to_string() {
  if (cuda::std::is_same_v<Op, cu::Max>) {
    return "Max";
  } else if (cuda::std::is_same_v<Op, cu::Min>) {
    return "Min";
  } else if (cuda::std::is_same_v<Op, cu::Sum>) {
    return "Sum";
  } else if (cuda::std::is_same_v<Op, cu::Prod>) {
    return "Prod";
  } else if (cuda::std::is_same_v<Op, cu::LogAddExp>) {
    return "LogAddExp";
  } else {
    throw std::invalid_argument("Unknown op.");
  }
}

template <typename Op, typename T>
constexpr bool supports_scan_op() {
  if constexpr (cuda::std::is_same_v<Op, LogAddExp>) {
    return is_inexact_v<T>;
  } else {
    return true;
  }
}

void Scan::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Scan::eval_gpu");
  assert(inputs.size() == 1);
  auto in = inputs[0];
  auto& s = stream();

  if (in.flags().contiguous && in.strides()[axis_] != 0) {
    if (in.is_donatable() && in.itemsize() == out.itemsize()) {
      out.copy_shared_buffer(in);
    } else {
      out.set_data(
          allocator::malloc(in.data_size() * out.itemsize()),
          in.data_size(),
          in.strides(),
          in.flags());
    }
  } else {
    in = contiguous_copy_gpu(in, s);
    out.copy_shared_buffer(in);
  }

  constexpr int N_READS = 4;
  int32_t axis_size = in.shape(axis_);
  bool contiguous = in.strides()[axis_] == 1;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);

  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    dispatch_scan_ops(reduce_type_, [&](auto scan_op_tag) {
      using Op = MLX_GET_TYPE(scan_op_tag);
      if constexpr (supports_scan_op<Op, T>) {
        using U = typename cu::ScanResult<Op, T>::type;
        dispatch_bool(inclusive_, [&](auto inclusive) {
          dispatch_bool(reverse_, [&](auto reverse) {
            if (contiguous) {
              auto kernel = cu::contiguous_scan<
                  T,
                  U,
                  Op,
                  N_READS,
                  inclusive.value,
                  reverse.value>;
              int block_dim = cuda::ceil_div(axis_size, N_READS);
              block_dim = cuda::ceil_div(block_dim, WARP_SIZE) * WARP_SIZE;
              block_dim = std::min(block_dim, WARP_SIZE * WARP_SIZE);
              encoder.add_kernel_node(
                  kernel,
                  in.data_size() / axis_size,
                  block_dim,
                  0,
                  in.data<T>(),
                  out.data<U>(),
                  axis_size);
            } else {
              constexpr int BM = WARP_SIZE;
              constexpr int BN = WARP_SIZE;
              auto kernel = cu::strided_scan<
                  T,
                  U,
                  Op,
                  N_READS,
                  BM,
                  BN,
                  inclusive.value,
                  reverse.value>;
              int64_t stride = in.strides()[axis_];
              int64_t stride_blocks = cuda::ceil_div(stride, BN);
              dim3 num_blocks = get_2d_grid_dims(
                  in.shape(), in.strides(), axis_size * stride);
              if (num_blocks.x * stride_blocks <= UINT32_MAX) {
                num_blocks.x *= stride_blocks;
              } else {
                num_blocks.y *= stride_blocks;
              }
              int block_dim = (BN / N_READS) * WARP_SIZE;
              encoder.add_kernel_node(
                  kernel,
                  num_blocks,
                  block_dim,
                  0,
                  in.data<T>(),
                  out.data<U>(),
                  axis_size,
                  stride,
                  stride_blocks);
            }
          });
        });
      } else {
        throw std::runtime_error(fmt::format(
            "Can not do scan op {} on inputs of {} with result of {}.",
            op_to_string<Op>(),
            dtype_to_string(in.dtype()),
            dtype_to_string(out.dtype())));
      }
    });
  });
}

} // namespace mlx::core
