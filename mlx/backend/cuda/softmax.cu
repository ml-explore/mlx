#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/cast_op.cuh"
#include "mlx/backend/cuda/device/fp16_math.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvtx3.hpp>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename T>
inline __device__ T softmax_exp(T x) {
  // Softmax doesn't need high precision exponential cause x is gonna be in
  // (-oo, 0] anyway and subsequently it will be divided by sum(exp(x_i)).
  return __expf(x);
}

template <typename T, typename AccT, int BLOCK_DIM, int N_READS = 4>
__global__ void softmax(const T* in, T* out, int axis_size) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  in += grid.block_rank() * axis_size;
  out += grid.block_rank() * axis_size;

  cg::greater<AccT> max_op;
  cg::plus<AccT> plus_op;

  // Thread reduce.
  AccT prevmax;
  AccT maxval = Limits<AccT>::finite_min();
  AccT normalizer = cast_to<AccT>(0);
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    auto vals = load_vector<N_READS>(in, index, axis_size, Limits<T>::min());
    prevmax = maxval;
#pragma unroll
    for (int i = 0; i < N_READS; ++i) {
      maxval = max_op(maxval, static_cast<AccT>(vals[i]));
    }

    // Online normalizer calculation for softmax:
    // https://github.com/NVIDIA/online-softmax
    normalizer = normalizer * softmax_exp(prevmax - maxval);
#pragma unroll
    for (int i = 0; i < N_READS; i++) {
      normalizer =
          normalizer + softmax_exp(static_cast<AccT>(vals[i]) - maxval);
    }
  }

  // First warp reduce.
  prevmax = maxval;
  maxval = cg::reduce(warp, maxval, max_op);
  normalizer = normalizer * softmax_exp(prevmax - maxval);
  normalizer = cg::reduce(warp, normalizer, plus_op);

  __shared__ AccT local_max[WARP_SIZE];
  __shared__ AccT local_normalizer[WARP_SIZE];

  // Write to shared memory and do second warp reduce.
  prevmax = maxval;
  if (warp.thread_rank() == 0) {
    local_max[warp.meta_group_rank()] = maxval;
  }
  block.sync();
  maxval = warp.thread_rank() < warp.meta_group_size()
      ? local_max[warp.thread_rank()]
      : Limits<AccT>::min();
  maxval = cg::reduce(warp, maxval, max_op);
  normalizer = normalizer * softmax_exp(prevmax - maxval);
  if (warp.thread_rank() == 0) {
    local_normalizer[warp.meta_group_rank()] = normalizer;
  }
  block.sync();
  normalizer = warp.thread_rank() < warp.meta_group_size()
      ? local_normalizer[warp.thread_rank()]
      : AccT{};
  normalizer = cg::reduce(warp, normalizer, plus_op);
  normalizer = 1 / normalizer;

  // Write output.
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    auto vals = load_vector<N_READS>(in, index, axis_size, T(0));
    for (int i = 0; i < N_READS; i++) {
      vals[i] = softmax_exp(static_cast<AccT>(vals[i]) - maxval) * normalizer;
    }
    store_vector<N_READS>(out, index, vals, axis_size);
  }
}

} // namespace cu

void Softmax::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Softmax::eval_gpu");
  assert(inputs.size() == 1);
  auto& s = stream();

  // Make sure that the last dimension is contiguous.
  auto set_output = [&s, &out](const array& x) {
    if (x.flags().contiguous && x.strides()[x.ndim() - 1] == 1) {
      if (x.is_donatable()) {
        out.copy_shared_buffer(x);
      } else {
        out.set_data(
            allocator::malloc(x.data_size() * x.itemsize()),
            x.data_size(),
            x.strides(),
            x.flags());
      }
      return x;
    } else {
      array x_copy = contiguous_copy_gpu(x, s);
      out.copy_shared_buffer(x_copy);
      return x_copy;
    }
  };

  array in = set_output(inputs[0]);
  bool precise = in.dtype() != float32 && precise_;

  int axis_size = in.shape().back();
  int n_rows = in.data_size() / axis_size;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_float_types(out.dtype(), "softmax", [&](auto type_tag) {
    using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    constexpr int N_READS = 16 / sizeof(DataType);
    dispatch_block_dim(cuda::ceil_div(axis_size, N_READS), [&](auto block_dim) {
      auto kernel = cu::softmax<DataType, DataType, block_dim(), N_READS>;
      if (precise) {
        kernel = cu::softmax<DataType, float, block_dim(), N_READS>;
      }
      encoder.add_kernel_node(
          kernel,
          n_rows,
          block_dim(),
          0,
          in.data<DataType>(),
          out.data<DataType>(),
          axis_size);
    });
  });
}

} // namespace mlx::core
