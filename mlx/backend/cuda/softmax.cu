#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/cast_op.cuh"
#include "mlx/backend/cuda/device/fp16_math.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvtx3.hpp>
#include <cub/block/block_load.cuh>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename T>
inline __device__ T softmax_exp(T x) {
  // Softmax doesn't need high precision exponential cause x is gonna be in
  // (-oo, 0] anyway and subsequently it will be divided by sum(exp(x_i)).
  return __expf(x);
}

template <typename T, typename AccT, int BLOCK_DIM, int N_READS = 4>
__global__ void softmax(const T* in, T* out, int axis_size) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  in += grid.block_rank() * axis_size;
  out += grid.block_rank() * axis_size;

  cg::greater<AccT> max_op;
  cg::plus<AccT> plus_op;

  // Thread reduce.
  AccT prevmax;
  AccT maxval = Limits<AccT>::finite_min();
  AccT normalizer = 0;
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    AccT vals[N_READS];
    hipcub::LoadDirectBlocked(
        r * BLOCK_DIM + block.thread_rank(),
        make_cast_iterator<AccT>(in),
        vals,
        axis_size,
        Limits<AccT>::finite_min());
    prevmax = maxval;
    maxval = max_op(maxval, hipcub::ThreadReduce(vals, max_op));
    // Online normalizer calculation for softmax:
    // https://github.com/NVIDIA/online-softmax
    normalizer = normalizer * softmax_exp(prevmax - maxval);
    for (int i = 0; i < N_READS; i++) {
      normalizer = normalizer + softmax_exp(vals[i] - maxval);
    }
  }

  // First warp reduce.
  prevmax = maxval;
  maxval = cg::reduce(warp, maxval, max_op);
  normalizer = normalizer * softmax_exp(prevmax - maxval);
  normalizer = cg::reduce(warp, normalizer, plus_op);

  __shared__ AccT local_max[WARP_SIZE];
  __shared__ AccT local_normalizer[WARP_SIZE];

  // Write to shared memory and do second warp reduce.
  prevmax = maxval;
  if (warp.thread_rank() == 0) {
    local_max[warp.meta_group_rank()] = maxval;
  }
  block.sync();
  maxval = warp.thread_rank() < warp.meta_group_size()
      ? local_max[warp.thread_rank()]
      : Limits<AccT>::finite_min();
  maxval = cg::reduce(warp, maxval, max_op);
  normalizer = normalizer * softmax_exp(prevmax - maxval);
  if (warp.thread_rank() == 0) {
    local_normalizer[warp.meta_group_rank()] = normalizer;
  }
  block.sync();
  normalizer = warp.thread_rank() < warp.meta_group_size()
      ? local_normalizer[warp.thread_rank()]
      : AccT{};
  normalizer = cg::reduce(warp, normalizer, plus_op);
  normalizer = 1 / normalizer;

  // Write output.
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    T vals[N_READS];
    hipcub::LoadDirectBlocked(index, in, vals, axis_size);
    for (int i = 0; i < N_READS; i++) {
      vals[i] = softmax_exp(static_cast<AccT>(vals[i]) - maxval) * normalizer;
    }
    hipcub::StoreDirectBlocked(index, out, vals, axis_size);
  }
}

} // namespace cu

void Softmax::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("Softmax::eval_gpu");
  assert(inputs.size() == 1);
  auto& s = stream();

  // Make sure that the last dimension is contiguous.
  auto set_output = [&s, &out](const array& x) {
    if (x.flags().contiguous && x.strides()[x.ndim() - 1] == 1) {
      if (x.is_donatable()) {
        out.copy_shared_buffer(x);
      } else {
        out.set_data(
            allocator::malloc(x.data_size() * x.itemsize()),
            x.data_size(),
            x.strides(),
            x.flags());
      }
      return x;
    } else {
      auto x_copy = array(x.shape(), x.dtype(), nullptr, {});
      copy_gpu(x, x_copy, CopyType::General, s);
      out.copy_shared_buffer(x_copy);
      return x_copy;
    }
  };

  array in = set_output(inputs[0]);
  bool precise = in.dtype() != float32 && precise_;

  int axis_size = in.shape().back();
  int n_rows = in.data_size() / axis_size;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_FLOAT_TYPES_CHECKED(out.dtype(), "softmax", CTYPE, {
      using DataType = cuda_type_t<CTYPE>;
      constexpr int N_READS = 4;
      MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
        auto kernel = cu::softmax<DataType, DataType, BLOCK_DIM, N_READS>;
        if (precise) {
          kernel = cu::softmax<DataType, float, BLOCK_DIM, N_READS>;
        }
        kernel<<<n_rows, BLOCK_DIM, 0, stream>>>(
            in.data<DataType>(), out.data<DataType>(), axis_size);
      });
    });
  });
}

} // namespace mlx::core
