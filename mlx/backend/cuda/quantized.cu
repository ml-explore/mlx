#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvtx3/nvtx3.hpp>

namespace mlx::core {
namespace cu {

namespace cg = cooperative_groups;

template <int bits, int wsize = 8>
inline constexpr __device__ short get_pack_factor() {
  return (bits == 3 || bits == 5) ? 8 : (bits == 6 ? 4 : wsize / bits);
}

template <int bits, int wsize = 8>
inline constexpr __device__ short get_bytes_per_pack() {
  constexpr int power_of_2_bits = (bits & (bits - 1)) == 0;
  return power_of_2_bits ? (wsize / 8) : (bits == 5 ? 5 : 3);
}

template <typename T, int group_size, int bits>
__global__ void
affine_quantize(const T* w, uint8_t* out, T* scales, T* biases, size_t size) {
  auto block_size = cg::this_thread_block().dim_threads();
  auto block_idx = cg::this_thread_block().group_index();
  auto idx_in_block = cg::this_thread_block().thread_index();

  auto tidx = block_idx.x * block_size.x + idx_in_block.x;
  auto tidy = block_idx.y * block_size.y + idx_in_block.y;

  auto grid_dim_x =
      cg::this_grid().dim_blocks().x * cg::this_grid().block_index().x;
  constexpr float eps = 1e-7;
  constexpr int simd_size = WARP_SIZE;
  constexpr float n_bins = (1 << bits) - 1;
  constexpr int pack_factor = get_pack_factor<bits, 8>();
  constexpr int bytes_per_pack = get_bytes_per_pack<bits>();
  constexpr int values_per_reduce = group_size / simd_size;
  constexpr int writes_per_reduce = pack_factor / values_per_reduce;
  constexpr int writes_per_pack =
      writes_per_reduce > 1 ? 1 : values_per_reduce / pack_factor;
  constexpr int power_of_2_bits = (bits & (bits - 1)) == 0;

  size_t offset = tidx + grid_dim_x * size_t(tidy);
  size_t in_index = offset * values_per_reduce;
  if (in_index >= size) {
    return;
  }
  size_t out_index = power_of_2_bits
      ? offset * writes_per_pack
      : offset * bytes_per_pack / writes_per_reduce;

  float w_thread[values_per_reduce];
  float w_min = Limits<float>::max();
  float w_max = 0;

#pragma clang loop unroll(full)
  for (int i = 0; i < values_per_reduce; i++) {
    float val = w[in_index + i];
    w_thread[i] = val;
    w_min = min(w_min, val);
    w_max = max(w_max, val);
  }

  cg::greater<float> max_op;
  cg::less<float> min_op;
  auto warp = cg::tiled_partition<WARP_SIZE>(cg::this_thread_block());

  w_min = cg::reduce(warp, w_min, min_op);
  w_max = cg::reduce(warp, w_max, max_op);

  float scale = max((w_max - w_min) / n_bins, eps);
  bool side = abs(w_min) > abs(w_max);
  scale = side ? scale : -scale;
  float edge = side ? w_min : w_max;
  float q0 = round(edge / scale);
  bool at_zero = q0 == 0.0f;
  scale = at_zero ? scale : edge / q0;
  float bias = at_zero ? 0 : edge;

  // Write out the scales and biases
  size_t gindex = in_index / group_size;
  if (in_index % group_size == 0) {
    scales[gindex] = static_cast<T>(scale);
    biases[gindex] = static_cast<T>(bias);
  }

  using OutType = std::conditional_t<bits == 5, uint64_t, uint32_t>;
  OutType output = 0;

#pragma clang loop unroll(full)
  for (int i = 0; i < values_per_reduce; i++) {
    uint8_t val = min(round((w_thread[i] - bias) / scale), n_bins);
    if (bits == 8) {
      output = val;
    } else {
      output |= val << (bits * (i % pack_factor));
    }

    if (pack_factor < values_per_reduce && i % pack_factor == pack_factor - 1) {
      out[out_index + i / pack_factor] = output;
      output = 0;
    } else {
#pragma clang loop unroll(full)
      for (int j = 1; j < writes_per_reduce; j++) {
        uint8_t sval = warp.shfl_down(val, j);
        output |= static_cast<OutType>(sval)
            << (bits * (j * values_per_reduce + i));
      }
    }
  }
  if constexpr (bits == 3 || bits == 6) {
    if (in_index % pack_factor == 0 && out_index % bytes_per_pack == 0) {
      out[out_index] = output & 0xff;
      out[out_index + 1] = (output & 0xff00) >> 8;
      out[out_index + 2] = (output & 0xff0000) >> 16;
    }
  } else if constexpr (bits == 5) {
    if (in_index % pack_factor == 0 && out_index % bytes_per_pack == 0) {
      out[out_index] = output & 0xff;
      out[out_index + 1] = (output & 0xff00) >> 8;
      out[out_index + 2] = (output & 0xff0000) >> 16;
      out[out_index + 3] = (output & 0xff000000) >> 24;
      out[out_index + 4] = (output & 0xff00000000) >> 32;
    }
  } else {
    if constexpr (writes_per_reduce > 0) {
      if (out_index % writes_per_reduce == 0) {
        out[out_index / writes_per_reduce] = output;
      }
    }
  }
}

template <typename T, int group_size, int bits>
__global__ void affine_dequantize(
    const uint8_t* w,
    const T* scales,
    const T* biases,
    T* out,
    size_t size) {
  auto block_size = cg::this_thread_block().dim_threads();
  auto block_idx = cg::this_thread_block().group_index();
  auto idx_in_block = cg::this_thread_block().thread_index();

  auto tidx = block_idx.x * block_size.x + idx_in_block.x;
  auto tidy = block_idx.y * block_size.y + idx_in_block.y;

  auto grid_dim_x =
      cg::this_grid().dim_blocks().x * cg::this_grid().block_index().x;

  constexpr int pack_factor = get_pack_factor<bits, 8>();
  constexpr int bytes_per_pack = get_bytes_per_pack<bits>();

  size_t offset = tidx + grid_dim_x * size_t(tidy);
  size_t oindex = offset * pack_factor;

  if (oindex >= size) {
    return;
  }

  size_t gindex = oindex / group_size;
  T scale = scales[gindex];
  T bias = biases[gindex];
  out += oindex;

  if constexpr (bits == 3) {
    w += offset * bytes_per_pack;
    out[0] = static_cast<T>(w[0] & 0x7) * scale + bias;
    out[1] = static_cast<T>((w[0] & 0x38) >> 3) * scale + bias;
    out[2] = (static_cast<T>((w[0] & 0xc0) >> 6) +
              static_cast<T>((w[1] & 0x1) << 2)) *
            scale +
        bias;
    out[3] = static_cast<T>((w[1] & 0xe) >> 1) * scale + bias;
    out[4] = static_cast<T>((w[1] & 0x70) >> 4) * scale + bias;
    out[5] = (static_cast<T>((w[1] & 0x80) >> 7) +
              static_cast<T>((w[2] & 0x3) << 1)) *
            scale +
        bias;
    out[6] = static_cast<T>((w[2] & 0x1c) >> 2) * scale + bias;
    out[7] = static_cast<T>((w[2] & 0xe0) >> 5) * scale + bias;
  } else if constexpr (bits == 5) {
    w += offset * bytes_per_pack;
    out[0] = static_cast<T>(w[0] & 0x1f) * scale + bias;
    out[1] = (static_cast<T>((w[0] & 0xe0) >> 5) +
              static_cast<T>((w[1] & 0x3) << 3)) *
            scale +
        bias;
    out[2] = static_cast<T>((w[1] & 0x7c) >> 2) * scale + bias;
    out[3] = (static_cast<T>((w[1] & 0x80) >> 7) +
              static_cast<T>((w[2] & 0xf) << 1)) *
            scale +
        bias;
    out[4] = (static_cast<T>((w[2] & 0xf0) >> 4) +
              static_cast<T>((w[3] & 0x1) << 4)) *
            scale +
        bias;
    out[5] = static_cast<T>((w[3] & 0x3e) >> 1) * scale + bias;
    out[6] = (static_cast<T>((w[3] & 0xc0) >> 6) +
              static_cast<T>((w[4] & 0x7) << 2)) *
            scale +
        bias;
    out[7] = static_cast<T>((w[4] & 0xf8) >> 3) * scale + bias;
  } else if constexpr (bits == 6) {
    w += offset * bytes_per_pack;
    out[0] = static_cast<T>(w[0] & 0x3f) * scale + bias;
    out[1] = (static_cast<T>((w[0] >> 6) & 0x03) +
              static_cast<T>((w[1] & 0x0f) << 2)) *
            scale +
        bias;
    out[2] = (static_cast<T>((w[1] >> 4) & 0x0f) +
              static_cast<T>((w[2] & 0x03) << 4)) *
            scale +
        bias;
    out[3] = static_cast<T>((w[2] >> 2) & 0x3f) * scale + bias;
  } else {
    uint val = w[offset];
#pragma clang loop unroll(full)
    for (int i = 0; i < pack_factor; i++) {
      uint8_t d;
      if (bits == 2) {
        d = (val >> (bits * i)) & 0x03;
      } else if (bits == 4) {
        d = (val >> (bits * i)) & 0x0f;
      } else if (bits == 8) {
        d = val;
      }
      out[i] = scale * static_cast<T>(d) + bias;
    }
  }
}

} // namespace cu
namespace {

inline array ensure_row_contiguous(
    const array& x,
    cu::CommandEncoder& enc,
    const Stream& s) {
  if (!x.flags().row_contiguous) {
    array x_copy = contiguous_copy_gpu(x, s);
    enc.add_temporary(x_copy);
    return x_copy;
  } else {
    return x;
  }
}

} // namespace

template <typename F>
void dispatch_groups(int group_size, F&& f) {
  switch (group_size) {
    case 32:
      f(std::integral_constant<int, 32>{});
      break;
    case 64:
      f(std::integral_constant<int, 64>{});
      break;
    case 128:
      f(std::integral_constant<int, 128>{});
      break;
  }
}

template <typename F>
void dispatch_bits(int bits, F&& f) {
  switch (bits) {
    case 2:
      f(std::integral_constant<int, 2>{});
      break;
    case 3:
      f(std::integral_constant<int, 3>{});
      break;
    case 4:
      f(std::integral_constant<int, 4>{});
      break;
    case 5:
      f(std::integral_constant<int, 5>{});
      break;
    case 6:
      f(std::integral_constant<int, 6>{});
      break;
    case 8:
      f(std::integral_constant<int, 8>{});
      break;
  }
}

void fast::AffineQuantize::eval_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs) {
  auto& w_pre = inputs[0];
  auto& out = outputs[0];
  out.set_data(allocator::malloc(out.nbytes()));

  auto& s = stream();
  auto& d = cu::device(s.device);
  auto& enc = d.get_command_encoder(s);

  auto w = ensure_row_contiguous(w_pre, enc, s);
  enc.set_input_array(w);
  if (dequantize_) {
    auto scales = ensure_row_contiguous(inputs[1], enc, s);
    auto biases = ensure_row_contiguous(inputs[2], enc, s);
    enc.set_input_array(scales);
    enc.set_input_array(biases);
    enc.set_output_array(out);
  } else {
    auto& scales = outputs[1];
    auto& biases = outputs[2];
    scales.set_data(allocator::malloc(scales.nbytes()));
    biases.set_data(allocator::malloc(biases.nbytes()));
    enc.set_output_array(out);
    enc.set_output_array(scales);
    enc.set_output_array(biases);
  }

  auto dtype = dequantize_ ? outputs[0].dtype() : inputs[0].dtype();

  // Treat uint32 as uint8 in kernel
  int uint8_per_uint32 = 4;
  int packs_per_int = (bits_ == 3 || bits_ == 5) ? 8
      : bits_ == 6                               ? 4
                                                 : 8 / bits_;
  int per_thread = dequantize_ ? packs_per_int : group_size_ / WARP_SIZE;
  size_t size =
      dequantize_ ? out.size() / packs_per_int : w.size() / per_thread;

  bool large = size > UINT_MAX;
  auto grid_shape = w.shape();

  if (dequantize_) {
    grid_shape.back() *= uint8_per_uint32;
  } else {
    grid_shape.back() /= per_thread;
  }

  dispatch_float_types(dtype, "affine_quantize", [&](auto type_tag) {
    dispatch_groups(group_size_, [&](auto group_size) {
      dispatch_bits(bits_, [&](auto bits) {
        using DataType = cuda_type_t<MLX_GET_TYPE(type_tag)>;
        if (dequantize_) {
          auto [num_blocks, block_dims] =
              get_launch_args(size, grid_shape, w.strides(), large);
          enc.add_kernel_node(
              cu::affine_dequantize<DataType, group_size.value, bits.value>,
              num_blocks,
              block_dims,
              w.data<uint8_t>(),
              inputs[1].data<DataType>(),
              inputs[2].data<DataType>(),
              out.data<DataType>(),
              out.size());
        } else {
          auto [num_blocks, block_dims] =
              get_launch_args(size, grid_shape, w.strides(), large);
          enc.add_kernel_node(
              cu::affine_quantize<DataType, group_size.value, bits.value>,
              num_blocks,
              block_dims,
              w.data<DataType>(),
              out.data<uint8_t>(),
              outputs[1].data<DataType>(),
              outputs[2].data<DataType>(),
              w.size());
        }
      });
    });
  });
}

} // namespace mlx::core
