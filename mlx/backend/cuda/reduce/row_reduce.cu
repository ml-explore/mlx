#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include <numeric>

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/reduce/reduce.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

struct RowReduceArgs {
  // The size of the row being reduced, i.e. the size of last dimension.
  int row_size;

  // Input shape and strides excluding the reduction axes.
  Shape shape;
  Strides strides;
  int ndim;

  // Input shape and strides of the reduction axes excluding last dimension.
  Shape reduce_shape;
  Strides reduce_strides;
  int reduce_ndim;

  // The number of rows we are reducing. Namely prod(reduce_shape).
  size_t non_row_reductions;

  RowReduceArgs(
      const array& in,
      const ReductionPlan& plan,
      const std::vector<int>& axes) {
    assert(!plan.shape.empty());
    row_size = plan.shape.back();

    auto [shape_vec, strides_vec] = shapes_without_reduction_axes(in, axes);
    std::tie(shape_vec, strides_vec) =
        collapse_contiguous_dims(shape_vec, strides_vec);
    shape = const_param(shape_vec);
    strides = const_param(strides_vec);
    ndim = shape_vec.size();

    reduce_shape = const_param(plan.shape);
    reduce_strides = const_param(plan.strides);
    reduce_ndim = plan.shape.size() - 1;

    non_row_reductions = 1;
    for (int i = 0; i < reduce_ndim; i++) {
      non_row_reductions *= reduce_shape[i];
    }
  }

  // Convert shape and strides as if in was contiguous
  void sort_access_pattern(const array& in, const std::vector<int>& axes) {
    auto shape_vec = in.shape();
    auto strides_vec = in.strides();
    std::tie(shape_vec, strides_vec) =
        shapes_without_reduction_axes(shape_vec, strides_vec, axes);
    std::vector<int> indices(shape_vec.size());
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(), indices.end(), [&](int left, int right) {
      return strides_vec[left] > strides_vec[right];
    });
    decltype(shape_vec) sorted_shape;
    decltype(strides_vec) sorted_strides;
    for (auto idx : indices) {
      sorted_shape.push_back(shape_vec[idx]);
      sorted_strides.push_back(strides_vec[idx]);
    }
    std::tie(shape_vec, strides_vec) =
        collapse_contiguous_dims(sorted_shape, sorted_strides);
    shape = const_param(shape_vec);
    strides = const_param(strides_vec);
    ndim = shape_vec.size();
  }
};

template <typename T, typename U, typename ReduceOp, int N = 4, int M = 1>
__global__ void row_reduce_simple(T* in, U* out, size_t n_rows, int size) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  const U init = cu::ReduceInit<ReduceOp, T>::value();
  ReduceOp op;

  T vals[M][N];
  U accs[M];
  for (int i = 0; i < M; i++) {
    accs[i] = init;
  }

  const size_t start_row =
      min(n_rows - M, static_cast<size_t>(grid.block_rank() * M));
  const size_t full_blocks = size / (block.size() * N);
  const size_t final_offset = full_blocks * (block.size() * N);
  in += start_row * size;
  out += start_row;

  if (size % N == 0) {
    for (size_t r = 0; r < full_blocks; r++) {
      for (int k = 0; k < M; k++) {
        hipcub::LoadDirectBlockedVectorized<T, N>(
            block.thread_rank(),
            in + k * size + r * (block.size() * N),
            vals[k]);
        for (int j = 0; j < N; j++) {
          accs[k] = op(accs[k], cast_to<U>(vals[k][j]));
        }
      }
    }
  } else {
    for (size_t r = 0; r < full_blocks; r++) {
      for (int k = 0; k < M; k++) {
        hipcub::LoadDirectBlocked(
            block.thread_rank(),
            in + k * size + r * (block.size() * N),
            vals[k]);
        for (int j = 0; j < N; j++) {
          accs[k] = op(accs[k], cast_to<U>(vals[k][j]));
        }
      }
    }
  }

  if (final_offset < size) {
    for (int k = 0; k < M; k++) {
      hipcub::LoadDirectBlocked(
          block.thread_rank(),
          in + k * size + final_offset,
          vals[k],
          size,
          cast_to<T>(init));
      for (int j = 0; j < N; j++) {
        accs[k] = op(accs[k], cast_to<U>(vals[k][j]));
      }
    }
  }

  __shared__ U shared_accumulators[32 * M];
  block_reduce(block, warp, accs, shared_accumulators, op, init);

  if (block.thread_rank() == 0) {
    if (grid.block_rank() * M + M <= n_rows) {
      for (int i = 0; i < M; i++) {
        out[i] = accs[i];
      }
    } else {
      short offset = grid.block_rank() * M + M - n_rows;
      for (int i = offset; i < M; i++) {
        out[i] = accs[i];
      }
    }
  }
}

template <
    typename T,
    typename U,
    typename Op,
    int NDIM,
    int BLOCK_DIM,
    int N_READS = 4>
__global__ void row_reduce_looped(
    T* in,
    U* out,
    size_t out_size,
    const __grid_constant__ RowReduceArgs args) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  size_t out_idx = grid.block_rank();

  Op op;

  U total[1];
  U init = ReduceInit<Op, T>::value();
  total[0] = init;
  LoopedElemToLoc<NDIM, (NDIM > 2)> loop(args.reduce_ndim);
  size_t full_blocks = args.row_size / (BLOCK_DIM * N_READS);
  size_t final_offset = full_blocks * BLOCK_DIM * N_READS;

  in += elem_to_loc(out_idx, args.shape.data(), args.strides.data(), args.ndim);

  for (size_t n = 0; n < args.non_row_reductions; n++) {
    for (size_t r = 0; r < full_blocks; r++) {
      T vals[N_READS];
      hipcub::LoadDirectBlockedVectorized<T, N_READS>(
          block.thread_rank(),
          in + loop.location() + r * BLOCK_DIM * N_READS,
          vals);
      for (int i = 0; i < N_READS; i++) {
        total[0] = op(total[0], cast_to<U>(vals[i]));
      }
    }
    if (final_offset < args.row_size) {
      T vals[N_READS];
      hipcub::LoadDirectBlocked(
          block.thread_rank(),
          in + loop.location() + final_offset,
          vals,
          args.row_size - final_offset,
          cast_to<T>(init));
      for (int i = 0; i < N_READS; i++) {
        total[0] = op(total[0], cast_to<U>(vals[i]));
      }
    }
    // TODO: Maybe block.sync() here?
    loop.next(args.reduce_shape.data(), args.reduce_strides.data());
  }

  __shared__ U shared_accumulators[32];
  block_reduce(block, warp, total, shared_accumulators, op, init);

  if (block.thread_rank() == 0) {
    out[out_idx] = total[0];
  }
}

} // namespace cu

void row_reduce_simple(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan) {
  constexpr int N_READS = 8;

  // Allocate data for the output using in's layout to avoid elem_to_loc in the
  // kernel.
  allocate_same_layout(out, in, axes);

  // TODO: If out.size() < 1024 which will be a common case then write this in
  //       2 passes. Something like 32 * out.size() and then do a warp reduce.
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    dispatch_reduce_ops(reduce_type, [&](auto reduce_type_tag) {
      using OP = MLX_GET_TYPE(reduce_type_tag);
      using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
      using U = typename cu::ReduceResult<OP, T>::type;

      // Cub doesn't like const pointers for vectorized loads. (sigh)
      T* indata = const_cast<T*>(in.data<T>());

      // Calculate the grid and block dims
      size_t reductions = (plan.shape.back() + N_READS - 1) / N_READS;
      dim3 grid = get_2d_grid_dims(out.shape(), out.strides());
      int threads = std::min(1024UL, reductions);
      threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
      dim3 block(threads, 1, 1);

      // Pick the kernel
      auto kernel = cu::row_reduce_simple<T, U, OP, N_READS>;
      if (grid.x >= 1024) {
        grid.x = (grid.x + 1) / 2;
        kernel = cu::row_reduce_simple<T, U, OP, N_READS, 2>;
      }

      int size = plan.shape.back();
      encoder.add_kernel_node(
          kernel, grid, block, 0, indata, out.data<U>(), out.size(), size);
    });
  });
}

void row_reduce_looped(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan,
    cu::RowReduceArgs args) {
  constexpr int N_READS = 8;

  // Allocate data for the output using in's layout to access them as
  // contiguously as possible.
  allocate_same_layout(out, in, axes);

  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    dispatch_reduce_ops(reduce_type, [&](auto reduce_type_tag) {
      using OP = MLX_GET_TYPE(reduce_type_tag);
      using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
      using U = typename cu::ReduceResult<OP, T>::type;
      // Cub doesn't like const pointers for vectorized loads. (sigh)
      T* indata = const_cast<T*>(in.data<T>());

      // Calculate the grid and block dims
      args.sort_access_pattern(in, axes);
      dim3 grid = get_2d_grid_dims(out.shape(), out.strides());
      size_t reductions = (args.row_size + N_READS - 1) / N_READS;
      int threads = std::min(1024UL, reductions);
      threads = ((threads + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
      dim3 block(threads, 1, 1);

      // Pick the kernel
      auto kernel = cu::row_reduce_looped<T, U, OP, 1, 32, N_READS>;
      dispatch_reduce_ndim(args.reduce_ndim, [&](auto reduce_ndim) {
        dispatch_block_dim(threads, [&](auto threads_constant) {
          kernel = cu::row_reduce_looped<
              T,
              U,
              OP,
              reduce_ndim.value,
              threads_constant.value,
              N_READS>;
          block.x = threads_constant.value;
        });
      });

      encoder.add_kernel_node(
          kernel, grid, block, 0, indata, out.data<U>(), out.size(), args);
    });
  });
}

void row_reduce(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan) {
  // Current row reduction options
  //
  // - row_reduce_simple
  //
  //   That means that we are simply reducing across the fastest moving axis.
  //   We are reducing 1 or 2 rows per threadblock depending on the size of
  //   output.
  //
  // - row_reduce_looped
  //
  //   It is a general row reduction. We are computing 1 output per
  //   threadblock. We read the fastest moving axis vectorized and loop over
  //   the rest of the axes.
  //
  // Notes: We opt to read as much in order as possible and leave
  //        transpositions as they are (contrary to our Metal backend).

  // Simple row reduce means that we have 1 axis that we are reducing over and
  // it has stride 1.
  if (plan.shape.size() == 1) {
    row_reduce_simple(encoder, in, out, reduce_type, axes, plan);
    return;
  }

  // Make the args struct to help route to the best kernel
  cu::RowReduceArgs args(in, plan, axes);

  // Fallback row reduce
  row_reduce_looped(encoder, in, out, reduce_type, axes, plan, std::move(args));
}

} // namespace mlx::core
