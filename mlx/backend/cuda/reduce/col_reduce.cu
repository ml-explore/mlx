#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/cast_op.cuh"
#include "mlx/backend/cuda/reduce/reduce.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cub/block/block_load.cuh>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

struct ColReduceArgs {
  // The size of the contiguous column reduction.
  size_t reduction_size;
  int64_t reduction_stride;

  // Input shape and strides excluding the reduction axes.
  Shape shape;
  Strides strides;
  int ndim;

  // Input shape and strides of the reduction axes (including last dimension).
  Shape reduce_shape;
  Strides reduce_strides;
  int reduce_ndim;

  // The number of column we are reducing. Namely prod(reduce_shape).
  size_t non_col_reductions;

  ColReduceArgs(
      const array& in,
      const ReductionPlan& plan,
      const std::vector<int>& axes) {
    assert(!plan.shape.empty());
    reduction_size = plan.shape.back();
    reduction_stride = plan.strides.back();

    int64_t stride_back = 1;
    auto [shape_vec, strides_vec] = shapes_without_reduction_axes(in, axes);
    while (!shape_vec.empty() && stride_back < reduction_stride) {
      stride_back *= shape_vec.back();
      shape_vec.pop_back();
      strides_vec.pop_back();
    }
    std::tie(shape_vec, strides_vec) =
        collapse_contiguous_dims(shape_vec, strides_vec);
    shape = const_param(shape_vec);
    strides = const_param(strides_vec);
    ndim = shape_vec.size();

    reduce_shape = const_param(plan.shape);
    reduce_strides = const_param(plan.strides);
    reduce_ndim = plan.shape.size();

    non_col_reductions = 1;
    for (int i = 0; i < reduce_ndim - 1; i++) {
      non_col_reductions *= reduce_shape[i];
    }
  }
};

template <
    typename T,
    typename U,
    typename Op,
    int NDIM,
    int BM,
    int BN,
    int N_READS = 4>
__global__ void
col_reduce_looped(T* in, U* out, const __grid_constant__ ColReduceArgs args) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  constexpr int threads_per_row = BN / N_READS;

  // Compute the indices for the tile
  size_t tile_idx = grid.block_rank();
  size_t tile_x = tile_idx % ((args.reduction_stride + BN - 1) / BN);
  size_t tile_y = tile_idx / ((args.reduction_stride + BN - 1) / BN);

  // Compute the indices for the thread within the tile
  short thread_x = block.thread_rank() % threads_per_row;
  short thread_y = block.thread_rank() / threads_per_row;

  // Move the input pointer
  in += elem_to_loc(tile_y, args.shape.data(), args.strides.data(), args.ndim) +
      tile_x * BN;

  // Initialize the running totals
  Op op;
  U totals[N_READS];
  for (int i = 0; i < N_READS; i++) {
    totals[i] = ReduceInit<Op, T>::value();
  }

  LoopedElemToLoc<NDIM, (NDIM > 2)> loop(args.reduce_ndim);
  loop.next(thread_y, args.reduce_shape.data(), args.reduce_strides.data());
  size_t total = args.non_col_reductions * args.reduction_size;
  if (tile_x * BN + BN <= args.reduction_stride) {
    for (size_t r = thread_y; r < total; r += BM) {
      T vals[N_READS];
      hipcub::LoadDirectBlockedVectorized(thread_x, in + loop.location(), vals);
      for (int i = 0; i < N_READS; i++) {
        totals[i] = op(totals[i], __cast<U, T>(vals[i]));
      }
      loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
    }
  } else {
    for (size_t r = thread_y; r < total; r += BM) {
      T vals[N_READS];
      hipcub::LoadDirectBlocked(
          thread_x,
          in + loop.location(),
          vals,
          args.reduction_stride - tile_x * BN,
          __cast<T, U>(ReduceInit<Op, T>::value()));
      for (int i = 0; i < N_READS; i++) {
        totals[i] = op(totals[i], __cast<U, T>(vals[i]));
      }
      loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
    }
  }

  // Do warp reduce for each output.
  constexpr int n_outputs = BN / threads_per_row;
  static_assert(BM == 32 && n_outputs == N_READS);
  __shared__ U shared_vals[BM * BN];
  short s_idx = thread_y * BN + thread_x * N_READS;
  for (int i = 0; i < N_READS; i++) {
    shared_vals[s_idx + i] = totals[i];
  }
  block.sync();
  s_idx = warp.thread_rank() * BN + warp.meta_group_rank() * n_outputs;
  for (int i = 0; i < n_outputs; i++) {
    totals[i] = cg::reduce(warp, shared_vals[s_idx + i], op);
  }

  // Write result.
  if (warp.thread_rank() == 0) {
    hipcub::StoreDirectBlocked(
        warp.meta_group_rank(),
        out + tile_y * args.reduction_stride + tile_x * BN,
        totals,
        args.reduction_stride - tile_x * BN);
  }
}

} // namespace cu

inline auto output_grid_for_col_reduce(
    const array& out,
    const cu::ColReduceArgs& args) {
  auto out_shape = out.shape();
  auto out_strides = out.strides();
  while (!out_shape.empty() && out_strides.back() < args.reduction_stride) {
    out_shape.pop_back();
    out_strides.pop_back();
  }
  return get_2d_grid_dims(out_shape, out_strides);
}

void col_reduce_looped(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan,
    cu::ColReduceArgs args) {
  // Allocate data for the output using in's layout to access them as
  // contiguously as possible.
  allocate_same_layout(out, in, axes);

  // Just a way to get out of the constness because cub doesn't like it ...
  // (sigh)
  array x = in;

  encoder.set_input_array(x);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(x.dtype(), CTYPE, {
      MLX_SWITCH_REDUCE_OPS(reduce_type, OP, {
        MLX_SWITCH_REDUCE_NDIM(args.reduce_ndim, NDIM, {
          using T = cuda_type_t<CTYPE>;
          using U = cu::ReduceResult<OP, T>::type;

          constexpr int N_READS = 4;
          constexpr int BM = 32;
          constexpr int BN = 32;
          dim3 grid = output_grid_for_col_reduce(out, args);
          size_t extra_blocks = cuda::ceil_div(args.reduction_stride, BN);
          if (grid.x * extra_blocks < INT32_MAX) {
            grid.x *= extra_blocks;
          } else if (grid.y * extra_blocks < 65536) {
            grid.y *= extra_blocks;
          } else {
            throw std::runtime_error(
                "[col_reduce_looped] Need to factorize reduction_stride");
          }
          int blocks = BM * BN / N_READS;
          auto kernel = cu::col_reduce_looped<T, U, OP, NDIM, BM, BN, N_READS>;
          kernel<<<grid, blocks, 0, stream>>>(x.data<T>(), out.data<U>(), args);
        });
      });
    });
  });
}

void col_reduce(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan) {
  // Current col reduce options
  //
  // - col_reduce_looped
  //
  //   It is a general strided reduce. Each threadblock computes the output for
  //   a subrow of the fast moving axis. For instance 32 elements.
  //
  // Notes: As in row reduce we opt to read as much in order as possible and
  // leave
  //        transpositions as they are (contrary to our Metal backend).
  //
  //        Moreover we need different kernels for short rows and tuning

  // Make the args struct to help route to the best kernel
  cu::ColReduceArgs args(in, plan, axes);

  // Fallback col reduce
  col_reduce_looped(encoder, in, out, reduce_type, axes, plan, args);
}

} // namespace mlx::core
