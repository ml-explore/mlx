#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/cast_op.cuh"
#include "mlx/backend/cuda/reduce/reduce.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cub/block/block_load.cuh>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

struct ColReduceArgs {
  // The size of the contiguous column reduction.
  size_t reduction_size;
  int64_t reduction_stride;

  // Input shape and strides excluding the reduction axes.
  Shape shape;
  Strides strides;
  int ndim;

  // Input shape and strides of the reduction axes (including last dimension).
  Shape reduce_shape;
  Strides reduce_strides;
  int reduce_ndim;

  // The number of column we are reducing. Namely prod(reduce_shape).
  size_t non_col_reductions;

  ColReduceArgs(
      const array& in,
      const ReductionPlan& plan,
      const std::vector<int>& axes) {
    assert(!plan.shape.empty());
    reduction_size = plan.shape.back();
    reduction_stride = plan.strides.back();

    int64_t stride_back = 1;
    auto [shape_vec, strides_vec] = shapes_without_reduction_axes(in, axes);
    while (!shape_vec.empty() && stride_back < reduction_stride) {
      stride_back *= shape_vec.back();
      shape_vec.pop_back();
      strides_vec.pop_back();
    }
    std::tie(shape_vec, strides_vec) =
        collapse_contiguous_dims(shape_vec, strides_vec);
    shape = const_param(shape_vec);
    strides = const_param(strides_vec);
    ndim = shape_vec.size();

    reduce_shape = const_param(plan.shape);
    reduce_strides = const_param(plan.strides);
    reduce_ndim = plan.shape.size();

    non_col_reductions = 1;
    for (int i = 0; i < reduce_ndim - 1; i++) {
      non_col_reductions *= reduce_shape[i];
    }
  }
};

template <typename T, typename U, typename Op, int NDIM, int N_READS = 4>
__global__ void col_reduce_small(
    const T* in,
    U* out,
    const __grid_constant__ ColReduceArgs args) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();

  int column =
      grid.block_index().x * block.dim_threads().x + block.thread_index().x;
  if (column * N_READS >= args.reduction_stride) {
    return;
  }

  int out_idx = grid.block_rank() / grid.dim_blocks().x;
  in += elem_to_loc(out_idx, args.shape.data(), args.strides.data(), args.ndim);

  Op op;
  U totals[N_READS];
  for (int i = 0; i < N_READS; i++) {
    totals[i] = ReduceInit<Op, T>::value();
  }

  // Read input to local.
  LoopedElemToLoc<NDIM, (NDIM > 2)> loop(args.reduce_ndim);
  loop.next(
      block.thread_index().y,
      args.reduce_shape.data(),
      args.reduce_strides.data());
  for (size_t r = block.thread_index().y;
       r < args.non_col_reductions * args.reduction_size;
       r += block.dim_threads().y) {
    U vals[N_READS];
    hipcub::LoadDirectBlocked(
        column,
        make_cast_iterator<U>(in + loop.location()),
        vals,
        args.reduction_stride,
        ReduceInit<Op, T>::value());
    for (int i = 0; i < N_READS; i++) {
      totals[i] = op(vals[i], totals[i]);
    }
    loop.next(
        block.dim_threads().y,
        args.reduce_shape.data(),
        args.reduce_strides.data());
  }

  // Do block reduce when each column has more than 1 element to reduce.
  if (block.dim_threads().y > 1) {
    __shared__ U shared_vals[32 * 8 * N_READS];
    size_t col =
        block.thread_index().y * block.dim_threads().x + block.thread_index().x;
    for (int i = 0; i < N_READS; i++) {
      shared_vals[col * N_READS + i] = totals[i];
    }
    block.sync();
    if (block.thread_index().y == 0) {
      for (int i = 0; i < N_READS; i++) {
        totals[i] = shared_vals[block.thread_index().x * N_READS + i];
      }
      for (int j = 1; j < block.dim_threads().y; j++) {
        col = j * block.dim_threads().x + block.thread_index().x;
        for (int i = 0; i < N_READS; i++) {
          totals[i] = op(shared_vals[col * N_READS + i], totals[i]);
        }
      }
    }
  }

  // Write result.
  if (block.thread_index().y == 0) {
    hipcub::StoreDirectBlocked(
        column,
        out + out_idx * args.reduction_stride,
        totals,
        args.reduction_stride);
  }
}

template <
    typename T,
    typename U,
    typename Op,
    int NDIM,
    int BM,
    int BN,
    int N_READS = 4>
__global__ void col_reduce_looped(
    const T* in,
    U* out,
    const __grid_constant__ ColReduceArgs args) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  constexpr int n_warps = BN / N_READS;

  int out_idx = grid.block_rank() / grid.dim_blocks().x;
  in += elem_to_loc(out_idx, args.shape.data(), args.strides.data(), args.ndim);

  Op op;
  U totals[N_READS];
  for (int i = 0; i < N_READS; i++) {
    totals[i] = ReduceInit<Op, T>::value();
  }

  // Read input to local.
  int r = block.thread_rank() / n_warps;
  int column = block.thread_rank() % n_warps;
  int in_offset = grid.block_index().x * BN;
  LoopedElemToLoc<NDIM, (NDIM > 2)> loop(args.reduce_ndim);
  loop.next(r, args.reduce_shape.data(), args.reduce_strides.data());
  for (; r < args.non_col_reductions * args.reduction_size; r += BM) {
    U vals[N_READS];
    hipcub::LoadDirectBlocked(
        column,
        make_cast_iterator<U>(in + loop.location() + in_offset),
        vals,
        args.reduction_stride - in_offset,
        ReduceInit<Op, T>::value());
    for (int i = 0; i < N_READS; i++) {
      totals[i] = op(vals[i], totals[i]);
    }
    loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
  }

  // Do warp reduce for each output.
  constexpr int n_outputs = BN / n_warps;
  static_assert(BM == 32 && n_outputs == N_READS);
  __shared__ U shared_vals[BM * BN];
  size_t col = block.thread_index().y * BN + block.thread_index().x * N_READS;
  for (int i = 0; i < N_READS; i++) {
    shared_vals[col + i] = totals[i];
  }
  block.sync();
  col = warp.thread_rank() * BN + warp.meta_group_rank() * n_outputs;
  for (int i = 0; i < n_outputs; i++) {
    totals[i] = cg::reduce(warp, shared_vals[col + i], op);
  }

  // Write result.
  if (warp.thread_rank() == 0) {
    size_t out_offset = grid.block_index().x * BN;
    hipcub::StoreDirectBlocked(
        warp.meta_group_rank(),
        out + out_idx * args.reduction_stride + out_offset,
        totals,
        args.reduction_stride - out_offset);
  }
}

} // namespace cu

inline auto output_grid_for_col_reduce(
    const array& out,
    const cu::ColReduceArgs& args) {
  auto out_shape = out.shape();
  auto out_strides = out.strides();
  while (!out_shape.empty() && out_strides.back() < args.reduction_stride) {
    out_shape.pop_back();
    out_strides.pop_back();
  }
  return get_2d_grid_dims(out_shape, out_strides);
}

void col_reduce(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan) {
  cu::ColReduceArgs args(in, plan, axes);

  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(in.dtype(), CTYPE, {
      using InType = cuda_type_t<CTYPE>;
      MLX_SWITCH_REDUCE_OPS(reduce_type, OP, {
        using OutType = cu::ReduceResult<OP, InType>::type;
        MLX_SWITCH_REDUCE_NDIM(args.reduce_ndim, NDIM, {
          constexpr int N_READS = 4;
          dim3 block_dims;
          dim3 num_blocks = output_grid_for_col_reduce(out, args);
          num_blocks.z = num_blocks.y;
          num_blocks.y = num_blocks.x;
          auto kernel =
              cu::col_reduce_small<InType, OutType, OP, NDIM, N_READS>;
          size_t total = args.non_col_reductions * args.reduction_size;
          if (total < 32) {
            size_t stride_blocks =
                cuda::ceil_div(args.reduction_stride, N_READS);
            block_dims.x = std::min(stride_blocks, 32ul);
            block_dims.y = std::min(total, 8ul);
            num_blocks.x = cuda::ceil_div(stride_blocks, block_dims.x);
          } else {
            constexpr int BM = 32;
            constexpr int BN = 32;
            block_dims.x = BM * BN / N_READS;
            num_blocks.x = cuda::ceil_div(args.reduction_stride, BN);
            kernel = cu::
                col_reduce_looped<InType, OutType, OP, NDIM, BM, BN, N_READS>;
          }
          kernel<<<num_blocks, block_dims, 0, stream>>>(
              in.data<InType>(), out.data<OutType>(), args);
        });
      });
    });
  });
}

} // namespace mlx::core
