#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include <numeric>

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/reduce/reduce.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cub/block/block_load.cuh>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

struct ColReduceArgs {
  // The size of the contiguous column reduction.
  size_t reduction_size;
  int64_t reduction_stride;

  // Input shape and strides excluding the reduction axes.
  Shape shape;
  Strides strides;
  int ndim;

  // Input shape and strides of the reduction axes (including last dimension).
  Shape reduce_shape;
  Strides reduce_strides;
  int reduce_ndim;

  // The number of column we are reducing. Namely prod(reduce_shape).
  size_t non_col_reductions;

  ColReduceArgs(
      const array& in,
      const ReductionPlan& plan,
      const std::vector<int>& axes) {
    using ShapeVector = decltype(plan.shape);
    using StridesVector = decltype(plan.strides);

    ShapeVector shape_vec;
    StridesVector strides_vec;

    assert(!plan.shape.empty());
    reduction_size = plan.shape.back();
    reduction_stride = plan.strides.back();

    int64_t stride_back = 1;
    std::tie(shape_vec, strides_vec) = shapes_without_reduction_axes(in, axes);
    while (!shape_vec.empty() && stride_back < reduction_stride) {
      stride_back *= shape_vec.back();
      shape_vec.pop_back();
      strides_vec.pop_back();
    }
    std::vector<int> indices(shape_vec.size());
    std::iota(indices.begin(), indices.end(), 0);
    std::sort(indices.begin(), indices.end(), [&](int left, int right) {
      return strides_vec[left] > strides_vec[right];
    });
    ShapeVector sorted_shape;
    StridesVector sorted_strides;
    for (auto idx : indices) {
      sorted_shape.push_back(shape_vec[idx]);
      sorted_strides.push_back(strides_vec[idx]);
    }
    std::tie(shape_vec, strides_vec) =
        collapse_contiguous_dims(sorted_shape, sorted_strides);
    shape = const_param(shape_vec);
    strides = const_param(strides_vec);
    ndim = shape_vec.size();

    reduce_shape = const_param(plan.shape);
    reduce_strides = const_param(plan.strides);
    reduce_ndim = plan.shape.size();

    non_col_reductions = 1;
    for (int i = 0; i < reduce_ndim - 1; i++) {
      non_col_reductions *= reduce_shape[i];
    }
  }
};

template <
    typename T,
    typename U,
    typename Op,
    int NDIM,
    int BM,
    int BN,
    int N_READS = 4>
__global__ void
col_reduce_looped(T* in, U* out, const __grid_constant__ ColReduceArgs args) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();
  auto warp = cg::tiled_partition<WARP_SIZE>(block);

  constexpr int threads_per_row = BN / N_READS;

  // Compute the indices for the tile
  size_t tile_idx = grid.block_rank();
  size_t tile_x = tile_idx % ((args.reduction_stride + BN - 1) / BN);
  size_t tile_y = tile_idx / ((args.reduction_stride + BN - 1) / BN);

  // Compute the indices for the thread within the tile
  short thread_x = block.thread_rank() % threads_per_row;
  short thread_y = block.thread_rank() / threads_per_row;

  // Move the input pointer
  in += elem_to_loc(tile_y, args.shape.data(), args.strides.data(), args.ndim) +
      tile_x * BN;

  // Initialize the running totals
  Op op;
  U totals[N_READS];
  for (int i = 0; i < N_READS; i++) {
    totals[i] = ReduceInit<Op, T>::value();
  }

  LoopedElemToLoc<NDIM, (NDIM > 2)> loop(args.reduce_ndim);
  loop.next(thread_y, args.reduce_shape.data(), args.reduce_strides.data());
  size_t total = args.non_col_reductions * args.reduction_size;
  if (tile_x * BN + BN <= args.reduction_stride) {
    if (args.reduction_stride % N_READS == 0) {
      for (size_t r = thread_y; r < total; r += BM) {
        T vals[N_READS];
        hipcub::LoadDirectBlockedVectorized(thread_x, in + loop.location(), vals);
        for (int i = 0; i < N_READS; i++) {
          totals[i] = op(totals[i], cast_to<U>(vals[i]));
        }
        loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
      }
    } else {
      for (size_t r = thread_y; r < total; r += BM) {
        T vals[N_READS];
        hipcub::LoadDirectBlocked(thread_x, in + loop.location(), vals);
        for (int i = 0; i < N_READS; i++) {
          totals[i] = op(totals[i], cast_to<U>(vals[i]));
        }
        loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
      }
    }
  } else {
    for (size_t r = thread_y; r < total; r += BM) {
      T vals[N_READS];
      hipcub::LoadDirectBlocked(
          thread_x,
          in + loop.location(),
          vals,
          args.reduction_stride - tile_x * BN,
          cast_to<T>(ReduceInit<Op, T>::value()));
      for (int i = 0; i < N_READS; i++) {
        totals[i] = op(totals[i], cast_to<U>(vals[i]));
      }
      loop.next(BM, args.reduce_shape.data(), args.reduce_strides.data());
    }
  }

  // Do warp reduce for each output.
  constexpr int n_outputs = BN / threads_per_row;
  static_assert(BM == 32 && n_outputs == N_READS);
  __shared__ U shared_vals[BM * BN];
  short s_idx = thread_y * BN + thread_x * N_READS;
  for (int i = 0; i < N_READS; i++) {
    shared_vals[s_idx + i] = totals[i];
  }
  block.sync();
  s_idx = warp.thread_rank() * BN + warp.meta_group_rank() * n_outputs;
  for (int i = 0; i < n_outputs; i++) {
    totals[i] = cg::reduce(warp, shared_vals[s_idx + i], op);
  }

  // Write result.
  if (warp.thread_rank() == 0) {
    hipcub::StoreDirectBlocked(
        warp.meta_group_rank(),
        out + tile_y * args.reduction_stride + tile_x * BN,
        totals,
        args.reduction_stride - tile_x * BN);
  }
}

} // namespace cu

inline auto output_grid_for_col_reduce(
    const array& out,
    const cu::ColReduceArgs& args,
    int bn) {
  int gx, gy = 1;
  size_t n_inner_blocks = cuda::ceil_div(args.reduction_stride, bn);
  size_t n_outer_blocks = out.size() / args.reduction_stride;
  size_t n_blocks = n_outer_blocks * n_inner_blocks;
  while (n_blocks / gy > INT32_MAX) {
    gy *= 2;
  }
  gx = cuda::ceil_div(n_blocks, gy);

  return dim3(gx, gy, 1);
}

void col_reduce_looped(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan,
    cu::ColReduceArgs args) {
  // Allocate data for the output using in's layout to access them as
  // contiguously as possible.
  allocate_same_layout(out, in, axes);

  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_all_types(in.dtype(), [&](auto type_tag) {
    dispatch_reduce_ops(reduce_type, [&](auto reduce_type_tag) {
      dispatch_reduce_ndim(args.reduce_ndim, [&](auto reduce_ndim) {
        using OP = MLX_GET_TYPE(reduce_type_tag);
        using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
        using U = typename cu::ReduceResult<OP, T>::type;
        // Cub doesn't like const pointers for vectorized loads. (sigh)
        T* indata = const_cast<T*>(in.data<T>());

        constexpr int N_READS = 4;
        constexpr int BM = 32;
        constexpr int BN = 32;
        dim3 grid = output_grid_for_col_reduce(out, args, BN);
        int blocks = BM * BN / N_READS;
        auto kernel =
            cu::col_reduce_looped<T, U, OP, reduce_ndim(), BM, BN, N_READS>;
        encoder.add_kernel_node(
            kernel, grid, blocks, 0, indata, out.data<U>(), args);
      });
    });
  });
}

void col_reduce(
    cu::CommandEncoder& encoder,
    const array& in,
    array& out,
    Reduce::ReduceType reduce_type,
    const std::vector<int>& axes,
    const ReductionPlan& plan) {
  // Current col reduce options
  //
  // - col_reduce_looped
  //
  //   It is a general strided reduce. Each threadblock computes the output for
  //   a subrow of the fast moving axis. For instance 32 elements.
  //
  // Notes: As in row reduce we opt to read as much in order as possible and
  //        leave transpositions as they are (contrary to our Metal backend).
  //
  //        Moreover we need different kernels for short rows and tuning

  // Make the args struct to help route to the best kernel
  cu::ColReduceArgs args(in, plan, axes);

  // Fallback col reduce
  col_reduce_looped(encoder, in, out, reduce_type, axes, plan, args);
}

} // namespace mlx::core
