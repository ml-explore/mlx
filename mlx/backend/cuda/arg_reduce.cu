#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.
#include "mlx/backend/common/utils.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/iterators/strided_iterator.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename T>
struct IndexValPair {
  uint32_t index;
  T val;
};

template <typename T>
struct ArgMin {
  constexpr __device__ T init() {
    return Limits<T>::max();
  }

  __device__ IndexValPair<T> operator()(
      const IndexValPair<T>& best,
      const IndexValPair<T>& current) {
    if (best.val > current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<T>
  reduce_many(IndexValPair<T> best, T (&vals)[N], uint32_t offset) {
    for (int i = 0; i < N; i++) {
      if (vals[i] < best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename T>
struct ArgMax {
  constexpr __device__ T init() {
    return Limits<T>::min();
  }

  __device__ IndexValPair<T> operator()(
      const IndexValPair<T>& best,
      const IndexValPair<T>& current) {
    if (best.val < current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<T>
  reduce_many(IndexValPair<T> best, T (&vals)[N], uint32_t offset) {
    for (int i = 0; i < N; i++) {
      if (vals[i] > best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename T, typename Op, int BLOCK_DIM, int N_READS = 4>
__global__ void arg_reduce_general(
    const T* in,
    uint32_t* out,
    size_t size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides in_strides,
    const __grid_constant__ Strides out_strides,
    int32_t ndim,
    int64_t axis_stride,
    int32_t axis_size) {
  auto block = cg::this_thread_block();

  int64_t index = cg::this_grid().block_rank();
  if (index >= size) {
    return;
  }

  int64_t in_idx = elem_to_loc(index, shape.data(), in_strides.data(), ndim);
  int64_t out_idx = elem_to_loc(index, shape.data(), out_strides.data(), ndim);

  Op op;
  T init = op.init();
  IndexValPair<T> best{0, init};

  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); ++r) {
    T vals[N_READS];
    auto tid = r * BLOCK_DIM + block.thread_index().x;
    hipcub::LoadDirectBlocked(
        tid, strided_iterator(in + in_idx, axis_stride), vals, axis_size, init);
    best = op.reduce_many(best, vals, tid * N_READS);
  }

  typedef hipcub::BlockReduce<IndexValPair<T>, BLOCK_DIM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp;

  best = BlockReduceT(temp).Reduce(best, op);

  if (block.thread_rank() == 0) {
    out[out_idx] = best.index;
  }
}

} // namespace cu

void ArgReduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgReduce::eval_gpu");
  assert(inputs.size() == 1);
  auto& in = inputs[0];
  out.set_data(allocator::malloc(out.nbytes()));
  auto& s = stream();

  // Prepare the shapes, strides and axis arguments.
  Shape shape = remove_index(in.shape(), axis_);
  Strides in_strides = remove_index(in.strides(), axis_);
  Strides out_strides = out.ndim() == in.ndim()
      ? remove_index(out.strides(), axis_)
      : out.strides();
  int64_t axis_stride = in.strides()[axis_];
  int32_t axis_size = in.shape()[axis_];
  int32_t ndim = shape.size();

  // ArgReduce.
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_REAL_TYPES_CHECKED(in.dtype(), "ArgReduce", CTYPE, {
      using InType = cuda_type_t<CTYPE>;
      constexpr uint32_t N_READS = 4;
      MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
        dim3 num_blocks = get_2d_grid_dims(out.shape(), out.strides());
        dim3 block_dims{BLOCK_DIM, 1, 1};
        auto kernel = &cu::arg_reduce_general<
            InType,
            cu::ArgMax<InType>,
            BLOCK_DIM,
            N_READS>;
        if (reduce_type_ == ArgReduce::ArgMin) {
          kernel = &cu::arg_reduce_general<
              InType,
              cu::ArgMin<InType>,
              BLOCK_DIM,
              N_READS>;
        }
        kernel<<<num_blocks, block_dims, 0, stream>>>(
            in.data<InType>(),
            out.data<uint32_t>(),
            out.size(),
            const_param(shape),
            const_param(in_strides),
            const_param(out_strides),
            ndim,
            axis_stride,
            axis_size);
      });
    });
  });
}

} // namespace mlx::core
