#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/utils.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/device/fp16_math.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename T>
struct IndexValPair {
  uint32_t index;
  T val;
};

template <typename T>
struct ArgMin {
  constexpr __device__ T init() {
    return Limits<T>::max();
  }

  __device__ IndexValPair<T> operator()(
      const IndexValPair<T>& best,
      const IndexValPair<T>& current) {
    if (best.val > current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<T> reduce_many(
      IndexValPair<T> best,
      const AlignedVector<T, N>& vals,
      uint32_t offset) {
#pragma unroll
    for (int i = 0; i < N; i++) {
      if (vals[i] < best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename T>
struct ArgMax {
  constexpr __device__ T init() {
    return Limits<T>::min();
  }

  __device__ IndexValPair<T> operator()(
      const IndexValPair<T>& best,
      const IndexValPair<T>& current) {
    if (best.val < current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<T> reduce_many(
      IndexValPair<T> best,
      const AlignedVector<T, N>& vals,
      uint32_t offset) {
#pragma unroll
    for (int i = 0; i < N; i++) {
      if (vals[i] > best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename T, typename Op, int BLOCK_DIM, int N_READS = 4>
__global__ void arg_reduce_general(
    const T* in,
    uint32_t* out,
    size_t size,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides in_strides,
    const __grid_constant__ Strides out_strides,
    int32_t ndim,
    int64_t axis_stride,
    int32_t axis_size) {
  auto block = cg::this_thread_block();

  int64_t index = cg::this_grid().block_rank();
  if (index >= size) {
    return;
  }

  int64_t in_idx = elem_to_loc(index, shape.data(), in_strides.data(), ndim);
  int64_t out_idx = elem_to_loc(index, shape.data(), out_strides.data(), ndim);
  in += in_idx;

  Op op;
  T init = op.init();
  IndexValPair<T> best{0, init};

  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); ++r) {
    auto tid = r * BLOCK_DIM + block.thread_index().x;
    auto vals = load_vector<N_READS>(in, tid, axis_size, axis_stride, init);
    best = op.reduce_many(best, vals, tid * N_READS);
  }

  typedef hipcub::BlockReduce<IndexValPair<T>, BLOCK_DIM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp;

  best = BlockReduceT(temp).Reduce(best, op);

  if (block.thread_rank() == 0) {
    out[out_idx] = best.index;
  }
}

} // namespace cu

void ArgReduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgReduce::eval_gpu");
  assert(inputs.size() == 1);
  auto& in = inputs[0];
  out.set_data(allocator::malloc(out.nbytes()));
  auto& s = stream();

  // Prepare the shapes, strides and axis arguments.
  Shape shape = remove_index(in.shape(), axis_);
  Strides in_strides = remove_index(in.strides(), axis_);
  Strides out_strides = out.ndim() == in.ndim()
      ? remove_index(out.strides(), axis_)
      : out.strides();
  int64_t axis_stride = in.strides()[axis_];
  int32_t axis_size = in.shape()[axis_];
  int32_t ndim = shape.size();

  // ArgReduce.
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  dispatch_real_types(in.dtype(), "ArgReduce", [&](auto type_tag) {
    using T = cuda_type_t<MLX_GET_TYPE(type_tag)>;
    constexpr uint32_t N_READS = 4;
    dispatch_block_dim(cuda::ceil_div(axis_size, N_READS), [&](auto block_dim) {
      dim3 num_blocks = get_2d_grid_dims(out.shape(), out.strides());
      auto kernel =
          cu::arg_reduce_general<T, cu::ArgMax<T>, block_dim(), N_READS>;
      if (reduce_type_ == ArgReduce::ArgMin) {
        kernel = cu::arg_reduce_general<T, cu::ArgMin<T>, block_dim(), N_READS>;
      }
      encoder.add_kernel_node(
          kernel,
          num_blocks,
          block_dim(),
          0,
          in.data<T>(),
          out.data<uint32_t>(),
          out.size(),
          const_param(shape),
          const_param(in_strides),
          const_param(out_strides),
          ndim,
          axis_stride,
          axis_size);
    });
  });
}

} // namespace mlx::core
