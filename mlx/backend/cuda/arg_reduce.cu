#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/iterators/strided_iterator.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

#include <cassert>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

template <typename U>
struct IndexValPair {
  uint32_t index;
  U val;
};

template <typename U>
struct ArgMin {
  static constexpr U init = Limits<U>::max;

  __device__ IndexValPair<U> operator()(
      const IndexValPair<U>& best,
      const IndexValPair<U>& current) {
    if (best.val > current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<U>
  reduce_many(IndexValPair<U> best, U (&vals)[N], uint32_t offset) {
    for (int i = 0; i < N; i++) {
      if (vals[i] < best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename U>
struct ArgMax {
  static constexpr U init = Limits<U>::min;

  __device__ IndexValPair<U> operator()(
      const IndexValPair<U>& best,
      const IndexValPair<U>& current) {
    if (best.val < current.val ||
        (best.val == current.val && best.index > current.index)) {
      return current;
    } else {
      return best;
    }
  }

  template <int N>
  __device__ IndexValPair<U>
  reduce_many(IndexValPair<U> best, U (&vals)[N], uint32_t offset) {
    for (int i = 0; i < N; i++) {
      if (vals[i] > best.val) {
        best.val = vals[i];
        best.index = offset + i;
      }
    }
    return best;
  }
};

template <typename U>
inline __device__ IndexValPair<U> warp_shuffle_down(
    const cg::thread_block_tile<WARP_SIZE>& g,
    const IndexValPair<U>& data,
    int delta) {
  return {g.shfl_down(data.index, delta), g.shfl_down(data.val, delta)};
}

template <typename T, typename Op, int BLOCK_DIM, int N_READS = 4>
__global__ void arg_reduce_general(
    const T* in,
    uint32_t* out,
    const __grid_constant__ Shape shape,
    const __grid_constant__ Strides in_strides,
    const __grid_constant__ Strides out_strides,
    size_t ndim,
    int64_t axis_stride,
    size_t axis_size) {
  // Shapes and strides *do not* contain the reduction axis. The reduction size
  // and stride are provided in axis_stride and axis_size.
  //
  // Note: in shape == out shape with this convention.
  Op op;

  // Compute the input/output index. There is one beginning and one output for
  // the whole block.
  auto elem = cg::this_grid().block_rank();
  auto in_idx = elem_to_loc(elem, shape.data(), in_strides.data(), ndim);
  auto out_idx = elem_to_loc(elem, shape.data(), out_strides.data(), ndim);

  IndexValPair<T> best{0, Op::init};

  auto block = cg::this_thread_block();
  for (size_t r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    T vals[N_READS];
    auto index = r * BLOCK_DIM + block.thread_index().z;
    hipcub::LoadDirectBlocked(
        index,
        strided_iterator(in + in_idx, axis_stride),
        vals,
        axis_size,
        Op::init);
    best = op.reduce_many(best, vals, index * N_READS);
  }

  typedef hipcub::BlockReduce<IndexValPair<T>, BLOCK_DIM> BlockReduceT;
  __shared__ typename BlockReduceT::TempStorage temp;

  best = BlockReduceT(temp).Reduce(best, op);

  if (block.thread_rank() == 0) {
    out[out_idx] = best.index;
  }
}

} // namespace cu

void ArgReduce::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("ArgReduce::eval_gpu");
  assert(inputs.size() == 1);
  auto& in = inputs[0];
  out.set_data(allocator::malloc(out.nbytes()));
  auto& s = stream();

  // Prepare the shapes, strides and axis arguments.
  auto in_strides = in.strides();
  auto shape = in.shape();
  auto out_strides = out.strides();
  auto axis_stride = in_strides[axis_];
  size_t axis_size = shape[axis_];
  if (out_strides.size() == in_strides.size()) {
    out_strides.erase(out_strides.begin() + axis_);
  }
  in_strides.erase(in_strides.begin() + axis_);
  shape.erase(shape.begin() + axis_);
  size_t ndim = shape.size();

  // ArgReduce.
  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(in);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_REAL_TYPES_CHECKED(in.dtype(), "ArgReduce", CTYPE, {
      using InType = cuda_type_t<CTYPE>;
      constexpr uint32_t N_READS = 4;
      MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
        dim3 num_blocks = get_2d_grid_dims(out.shape(), out.strides());
        dim3 block_dims{1, 1, BLOCK_DIM};
        auto kernel = &cu::arg_reduce_general<
            InType,
            cu::ArgMax<InType>,
            BLOCK_DIM,
            N_READS>;
        if (reduce_type_ == ArgReduce::ArgMin) {
          kernel = &cu::arg_reduce_general<
              InType,
              cu::ArgMin<InType>,
              BLOCK_DIM,
              N_READS>;
        }
        kernel<<<num_blocks, block_dims, 0, stream>>>(
            in.data<InType>(),
            out.data<uint32_t>(),
            const_param(shape),
            const_param(in_strides),
            const_param(out_strides),
            ndim,
            axis_stride,
            axis_size);
      });
    });
  });
}

} // namespace mlx::core
