#include "hip/hip_runtime.h"
// Copyright © 2024 Apple Inc.

#include "mlx/backend/cuda/allocator.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/event.h"
#include "mlx/scheduler.h"

#include <map>
#include <vector>

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

///////////////////////////////////////////////////////////////////////////////
// CudaEvent implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

// Manage cached hipEvent_t objects.
struct CudaEventPool {
  static CudaEventHandle create(int flags) {
    auto& cache = cache_for(flags);
    if (cache.empty()) {
      return CudaEventHandle(flags);
    } else {
      CudaEventHandle ret = std::move(cache.back());
      cache.pop_back();
      return ret;
    }
  }

  static void release(CudaEventHandle event) {
    cache_for(event.flags).push_back(std::move(event));
  }

  static std::vector<CudaEventHandle>& cache_for(int flags) {
    static std::map<int, std::vector<CudaEventHandle>> cache;
    return cache[flags];
  }
};

} // namespace

CudaEventHandle::CudaEventHandle(int flags) : flags(flags) {
  CHECK_CUDA_ERROR(hipEventCreateWithFlags(&handle_, flags));
  assert(handle_ != nullptr);
}

CudaEvent::CudaEvent(int flags) : event_(CudaEventPool::create(flags)) {}

CudaEvent::~CudaEvent() {
  CudaEventPool::release(std::move(event_));
}

void CudaEvent::wait() {
  nvtx3::scoped_range r("cu::CudaEvent::wait");
  hipEventSynchronize(event_);
}

void CudaEvent::wait(hipStream_t stream) {
  hipStreamWaitEvent(stream, event_);
}

void CudaEvent::record(hipStream_t stream) {
  hipEventRecord(event_, stream);
}

bool CudaEvent::completed() const {
  return hipEventQuery(event_) == hipSuccess;
}

// Wraps CudaEvent with a few features:
// 1. The class can be copied.
// 2. Make wait/record work with CPU streams.
// 3. Add checks for waiting on un-recorded event.
class CopyableCudaEvent {
 public:
  CopyableCudaEvent()
      : event_(std::make_shared<CudaEvent>(
            hipEventDisableTiming | hipEventBlockingSync)) {}

  void wait() {
    event_->wait();
  }

  void wait(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      scheduler::enqueue(s, [*this]() mutable {
        check_recorded();
        event_->wait();
      });
    } else {
      check_recorded();
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->wait(encoder.stream());
    }
  }

  void record(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      throw std::runtime_error("CudaEvent can not wait on CPU stream.");
    } else {
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->record(encoder.stream());
      recorded_ = true;
    }
  }

  bool is_signaled() const {
    return recorded_ && event_->completed();
  }

 private:
  void check_recorded() const {
    if (!recorded_) {
      throw std::runtime_error(
          "Should not wait on a CudaEvent before recording.");
    }
  }

  std::shared_ptr<CudaEvent> event_;
  bool recorded_{false};
};

///////////////////////////////////////////////////////////////////////////////
// AtomicEvent implementations
///////////////////////////////////////////////////////////////////////////////

__host__ __device__ void event_wait(AtomicEvent::Atomic* ac, uint64_t value) {
  uint64_t current;
  while ((current = ac->load()) < value) {
    ac->wait(current);
  }
}

__host__ __device__ void event_signal(AtomicEvent::Atomic* ac, uint64_t value) {
  ac->store(value);
  ac->notify_all();
}

__global__ void event_wait_kernel(AtomicEvent::Atomic* ac, uint64_t value) {
  event_wait(ac, value);
}

__global__ void event_signal_kernel(AtomicEvent::Atomic* ac, uint64_t value) {
  event_signal(ac, value);
}

AtomicEvent::AtomicEvent() {
  buf_ = std::shared_ptr<Buffer>(
      new Buffer{allocator().malloc(sizeof(Atomic))}, [](Buffer* ptr) {
        allocator().free(*ptr);
        delete ptr;
      });
  *static_cast<uint64_t*>(buf_->raw_ptr()) = 0;
}

void AtomicEvent::wait(uint64_t value) {
  nvtx3::scoped_range r("cu::AtomicEvent::wait");
  event_wait(atomic(), value);
}

void AtomicEvent::wait(hipStream_t stream, uint64_t value) {
  event_wait_kernel<<<1, 1, 0, stream>>>(atomic(), value);
}

void AtomicEvent::wait(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::AtomicEvent::wait(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { wait(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    wait(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

void AtomicEvent::signal(uint64_t value) {
  nvtx3::scoped_range r("cu::AtomicEvent::signal");
  event_signal(atomic(), value);
}

void AtomicEvent::signal(hipStream_t stream, uint64_t value) {
  event_signal_kernel<<<1, 1, 0, stream>>>(atomic(), value);
}

void AtomicEvent::signal(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::AtomicEvent::signal(s)");
  if (s.device == mlx::core::Device::cpu) {
    // Signal through a GPU stream so the atomic is updated in GPU - updating
    // the atomic in CPU sometimes does not get GPU notified.
    static CudaStream stream(device(mlx::core::Device::gpu));
    scheduler::enqueue(s, [*this, value]() mutable { signal(stream, value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    signal(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

bool AtomicEvent::is_signaled(uint64_t value) const {
  nvtx3::scoped_range r("cu::AtomicEvent::is_signaled");
  return atomic()->load() >= value;
}

uint64_t AtomicEvent::value() const {
  nvtx3::scoped_range r("cu::AtomicEvent::value");
  return atomic()->load();
}

} // namespace cu

///////////////////////////////////////////////////////////////////////////////
// Event implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

struct EventImpl {
  // CudaEvent is preferred when possible because it is fast, however we have
  // to fallback to AtomicEvent in following cases:
  // 1. the event is used to wait/signal a cpu stream;
  // 2. signal value other than 1 has been specified.
  std::unique_ptr<cu::CopyableCudaEvent> cuda;
  std::unique_ptr<cu::AtomicEvent> atomic;

  bool is_created() const {
    return cuda || atomic;
  }

  void ensure_created(Stream s, uint64_t signal_value) {
    if (is_created()) {
      return;
    }
    if (s.device == mlx::core::Device::cpu || signal_value > 1) {
      nvtx3::mark("Using slow AtomicEvent");
      atomic = std::make_unique<cu::AtomicEvent>();
    } else {
      cuda = std::make_unique<cu::CopyableCudaEvent>();
    }
  }
};

} // namespace

Event::Event(Stream s) : stream_(s) {
  event_ = std::shared_ptr<void>(
      new EventImpl(), [](void* ptr) { delete static_cast<EventImpl*>(ptr); });
}

void Event::wait() {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait();
  } else {
    event->atomic->wait(value());
  }
}

void Event::wait(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait(s);
  } else {
    event->atomic->wait(s, value());
  }
}

void Event::signal(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  event->ensure_created(s, value());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->record(s);
  } else {
    event->atomic->signal(s, value());
  }
}

bool Event::is_signaled() const {
  auto* event = static_cast<EventImpl*>(event_.get());
  if (!event->is_created()) {
    return false;
  }
  if (event->cuda) {
    assert(value() == 1);
    return event->cuda->is_signaled();
  } else {
    return event->atomic->is_signaled(value());
  }
}

} // namespace mlx::core
