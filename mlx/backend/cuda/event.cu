#include "hip/hip_runtime.h"
// Copyright © 2024 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/backend/cuda/utils.h"
#include "mlx/event.h"
#include "mlx/scheduler.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

///////////////////////////////////////////////////////////////////////////////
// CudaEvent implementations
///////////////////////////////////////////////////////////////////////////////

// Cuda event managed with RAII.
class CudaEventHandle {
 public:
  CudaEventHandle() {
    CHECK_CUDA_ERROR(hipEventCreateWithFlags(
        &event_, hipEventDisableTiming | hipEventBlockingSync));
  }

  ~CudaEventHandle() {
    CHECK_CUDA_ERROR(hipEventDestroy(event_));
  }

  CudaEventHandle(const CudaEventHandle&) = delete;
  CudaEventHandle& operator=(const CudaEventHandle&) = delete;

  operator hipEvent_t() const {
    return event_;
  }

 private:
  hipEvent_t event_;
};

CudaEvent::CudaEvent() : event_(std::make_shared<CudaEventHandle>()) {}

void CudaEvent::wait() {
  nvtx3::scoped_range r("cu::CudaEvent::wait");
  if (!recorded_) {
    throw std::runtime_error("Should not wait on a CudaEvent before record.");
  }
  hipEventSynchronize(*event_);
}

void CudaEvent::wait(hipStream_t stream) {
  if (!recorded_) {
    throw std::runtime_error("Should not wait on a CudaEvent before record.");
  }
  hipStreamWaitEvent(stream, *event_);
}

void CudaEvent::wait(Stream s) {
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this]() mutable { wait(); });
  } else {
    wait(cu::get_stream(s).last_cuda_stream());
  }
}

void CudaEvent::record(hipStream_t stream) {
  hipEventRecord(*event_, stream);
  recorded_ = true;
}

void CudaEvent::record(Stream s) {
  if (s.device == mlx::core::Device::cpu) {
    throw std::runtime_error("CudaEvent can not wait on cpu stream.");
  } else {
    record(cu::get_stream(s).last_cuda_stream());
  }
}

bool CudaEvent::completed() const {
  return hipEventQuery(*event_) == hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////
// SharedEvent implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

__host__ __device__ void event_wait(SharedEvent::Atomic* ac, uint64_t value) {
  uint64_t current;
  while ((current = ac->load()) < value) {
    ac->wait(current);
  }
}

__host__ __device__ void event_signal(SharedEvent::Atomic* ac, uint64_t value) {
  ac->store(value);
  ac->notify_all();
}

__global__ void event_wait_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_wait(ac, value);
}

__global__ void event_signal_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_signal(ac, value);
}

} // namespace

SharedEvent::SharedEvent() {
  // Allocate cuda::atomic on managed memory.
  allocator::Buffer buffer = allocator::malloc(sizeof(Atomic));
  Atomic* ac = static_cast<Atomic*>(buffer.raw_ptr());
  new (ac) Atomic(0);
  ac_ = std::shared_ptr<Atomic>(ac, [buffer](Atomic* ptr) {
    ptr->~Atomic();
    allocator::free(buffer);
  });
}

void SharedEvent::wait(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait");
  event_wait(ac_.get(), value);
}

void SharedEvent::wait(hipStream_t stream, uint64_t value) {
  event_wait_kernel<<<1, 1, 0, stream>>>(ac_.get(), value);
}

void SharedEvent::wait(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { wait(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.launch_kernel(
        encoder.stream().last_cuda_stream(),
        [this, value](hipStream_t stream) { wait(stream, value); });
    encoder.add_completed_handler([ac = ac_]() {});
    encoder.end_encoding();
  }
}

void SharedEvent::signal(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal");
  event_signal(ac_.get(), value);
}

void SharedEvent::signal(hipStream_t stream, uint64_t value) {
  event_signal_kernel<<<1, 1, 0, stream>>>(ac_.get(), value);
}

void SharedEvent::signal(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { signal(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.launch_kernel(
        encoder.stream().last_cuda_stream(),
        [this, value](hipStream_t stream) { signal(stream, value); });
    encoder.add_completed_handler([ac = ac_]() {});
    encoder.end_encoding();
  }
}

bool SharedEvent::is_signaled(uint64_t value) const {
  nvtx3::scoped_range r("cu::SharedEvent::is_signaled");
  return ac_->load() >= value;
}

uint64_t SharedEvent::value() const {
  nvtx3::scoped_range r("cu::SharedEvent::value");
  return ac_->load();
}

} // namespace cu

///////////////////////////////////////////////////////////////////////////////
// Event implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

struct EventImpl {
  // CudaEvent is preferred when possible because it is fast, however we have
  // to fallback to SharedEvent in following cases:
  // 1. the event is used to wait/signal a cpu stream;
  // 2. signal value other than 1 has been specified.
  std::unique_ptr<cu::CudaEvent> cuda;
  std::unique_ptr<cu::SharedEvent> shared;

  bool is_created() const {
    return cuda || shared;
  }

  void ensure_created(Stream s, uint64_t signal_value) {
    if (is_created()) {
      return;
    }
    if (s.device == mlx::core::Device::cpu || signal_value > 1) {
      nvtx3::mark("Using slow SharedEvent");
      shared = std::make_unique<cu::SharedEvent>();
    } else {
      cuda = std::make_unique<cu::CudaEvent>();
    }
  }
};

} // namespace

Event::Event(Stream s) : stream_(s) {
  event_ = std::shared_ptr<void>(
      new EventImpl(), [](void* ptr) { delete static_cast<EventImpl*>(ptr); });
}

void Event::wait() {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait();
  } else {
    event->shared->wait(value());
  }
}

void Event::wait(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait(s);
  } else {
    event->shared->wait(s, value());
  }
}

void Event::signal(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  event->ensure_created(s, value());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->record(s);
  } else {
    event->shared->signal(s, value());
  }
}

bool Event::is_signaled() const {
  auto* event = static_cast<EventImpl*>(event_.get());
  if (!event->is_created()) {
    return false;
  }
  if (event->cuda) {
    assert(value() == 1);
    return event->cuda->recorded() && event->cuda->completed();
  } else {
    return event->shared->is_signaled(value());
  }
}

} // namespace mlx::core
