#include "hip/hip_runtime.h"
// Copyright © 2024 Apple Inc.

#include "mlx/backend/cuda/allocator.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/backend/cuda/utils.h"
#include "mlx/event.h"
#include "mlx/scheduler.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

///////////////////////////////////////////////////////////////////////////////
// CudaEvent implementations
///////////////////////////////////////////////////////////////////////////////

// Wrapper of native cuda event. It can synchronize between GPU streams, or wait
// on GPU stream in CPU stream, but can not wait on CPU stream.
class CudaEventWrapper {
 public:
  CudaEventWrapper()
      : event_(std::make_shared<CudaEvent>(
            hipEventDisableTiming | hipEventBlockingSync)) {}

  void wait() {
    event_->wait();
  }

  void wait(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      scheduler::enqueue(s, [*this]() mutable {
        check_recorded();
        event_->wait();
      });
    } else {
      check_recorded();
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->wait(encoder.stream());
    }
  }

  void record(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      throw std::runtime_error("CudaEvent can not wait on CPU stream.");
    } else {
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->record(encoder.stream());
      recorded_ = true;
    }
  }

  bool is_signaled() const {
    return recorded_ && event_->completed();
  }

 private:
  void check_recorded() const {
    if (!recorded_) {
      throw std::runtime_error(
          "Should not wait on a CudaEvent before recording.");
    }
  }

  std::shared_ptr<CudaEvent> event_;
  bool recorded_{false};
};

///////////////////////////////////////////////////////////////////////////////
// SharedEvent implementations
///////////////////////////////////////////////////////////////////////////////

__host__ __device__ void event_wait(SharedEvent::Atomic* ac, uint64_t value) {
  uint64_t current;
  while ((current = ac->load()) < value) {
    ac->wait(current);
  }
}

__host__ __device__ void event_signal(SharedEvent::Atomic* ac, uint64_t value) {
  ac->store(value);
  ac->notify_all();
}

__global__ void event_wait_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_wait(ac, value);
}

__global__ void event_signal_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_signal(ac, value);
}

SharedEvent::Atomic* to_atomic(std::shared_ptr<Buffer> buf) {
  return static_cast<SharedEvent::Atomic*>(buf->raw_ptr());
}

SharedEvent::SharedEvent() {
  buf_ = std::shared_ptr<Buffer>(
      new Buffer{allocator().malloc(sizeof(Atomic))}, [](Buffer* ptr) {
        allocator().free(*ptr);
        delete ptr;
      });
  *static_cast<uint64_t*>(buf_->raw_ptr()) = 0;
}

void SharedEvent::wait(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait");
  event_wait(to_atomic(buf_), value);
}

void SharedEvent::wait(hipStream_t stream, uint64_t value) {
  event_wait_kernel<<<1, 1, 0, stream>>>(to_atomic(buf_), value);
}

void SharedEvent::wait(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { wait(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    wait(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

void SharedEvent::signal(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal");
  event_signal(to_atomic(buf_), value);
}

void SharedEvent::signal(hipStream_t stream, uint64_t value) {
  event_signal_kernel<<<1, 1, 0, stream>>>(to_atomic(buf_), value);
}

void SharedEvent::signal(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal(s)");
  if (s.device == mlx::core::Device::cpu) {
    // Signal through a GPU stream so the atomic is updated in GPU - updating
    // the atomic in CPU sometimes does not get GPU notified.
    static CudaStream stream(device(mlx::core::Device::gpu));
    scheduler::enqueue(s, [*this, value]() mutable { signal(stream, value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    signal(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

bool SharedEvent::is_signaled(uint64_t value) const {
  nvtx3::scoped_range r("cu::SharedEvent::is_signaled");
  return to_atomic(buf_)->load() >= value;
}

uint64_t SharedEvent::value() const {
  nvtx3::scoped_range r("cu::SharedEvent::value");
  return to_atomic(buf_)->load();
}

} // namespace cu

///////////////////////////////////////////////////////////////////////////////
// Event implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

struct EventImpl {
  // CudaEvent is preferred when possible because it is fast, however we have
  // to fallback to SharedEvent in following cases:
  // 1. the event is used to wait/signal a cpu stream;
  // 2. signal value other than 1 has been specified.
  std::unique_ptr<cu::CudaEventWrapper> cuda;
  std::unique_ptr<cu::SharedEvent> shared;

  bool is_created() const {
    return cuda || shared;
  }

  void ensure_created(Stream s, uint64_t signal_value) {
    if (is_created()) {
      return;
    }
    if (s.device == mlx::core::Device::cpu || signal_value > 1) {
      nvtx3::mark("Using slow SharedEvent");
      shared = std::make_unique<cu::SharedEvent>();
    } else {
      cuda = std::make_unique<cu::CudaEventWrapper>();
    }
  }
};

} // namespace

Event::Event(Stream s) : stream_(s) {
  event_ = std::shared_ptr<void>(
      new EventImpl(), [](void* ptr) { delete static_cast<EventImpl*>(ptr); });
}

void Event::wait() {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait();
  } else {
    event->shared->wait(value());
  }
}

void Event::wait(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait(s);
  } else {
    event->shared->wait(s, value());
  }
}

void Event::signal(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  event->ensure_created(s, value());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->record(s);
  } else {
    event->shared->signal(s, value());
  }
}

bool Event::is_signaled() const {
  auto* event = static_cast<EventImpl*>(event_.get());
  if (!event->is_created()) {
    return false;
  }
  if (event->cuda) {
    assert(value() == 1);
    return event->cuda->is_signaled();
  } else {
    return event->shared->is_signaled(value());
  }
}

} // namespace mlx::core
