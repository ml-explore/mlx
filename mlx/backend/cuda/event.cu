#include "hip/hip_runtime.h"
// Copyright © 2024 Apple Inc.

#include "mlx/backend/cuda/allocator.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/backend/cuda/utils.h"
#include "mlx/event.h"
#include "mlx/scheduler.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

///////////////////////////////////////////////////////////////////////////////
// CudaEvent implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

// Manage cached hipEvent_t objects.
struct CudaEventFactory {
  static RawCudaEvent create(int flags) {
    auto& cache = cache_for(flags);
    if (cache.empty()) {
      return RawCudaEvent(flags);
    } else {
      RawCudaEvent ret = std::move(cache.back());
      cache.pop_back();
      return ret;
    }
  }

  static void release(RawCudaEvent event) {
    assert(event != nullptr);
    cache_for(event.flags).push_back(std::move(event));
  }

  static std::vector<RawCudaEvent>& cache_for(int flags) {
    static std::map<int, std::vector<RawCudaEvent>> cache;
    return cache[flags];
  }
};

} // namespace

RawCudaEvent::RawCudaEvent(int flags) : flags(flags) {
  CHECK_CUDA_ERROR(hipEventCreateWithFlags(&handle_, flags));
}

CudaEvent::CudaEvent(int flags) : event_(CudaEventFactory::create(flags)) {
  assert(event_ != nullptr);
}

CudaEvent::~CudaEvent() {
  if (event_) {
    CudaEventFactory::release(std::move(event_));
  }
}

void CudaEvent::wait() {
  nvtx3::scoped_range r("cu::CudaEvent::wait");
  assert(event_ != nullptr);
  hipEventSynchronize(event_);
}

void CudaEvent::wait(hipStream_t stream) {
  assert(event_ != nullptr);
  hipStreamWaitEvent(stream, event_);
}

void CudaEvent::record(hipStream_t stream) {
  assert(event_ != nullptr);
  hipEventRecord(event_, stream);
}

bool CudaEvent::completed() const {
  assert(event_ != nullptr);
  return hipEventQuery(event_) == hipSuccess;
}

// Wraps CudaEvent with a few features:
// 1. The class can be copied.
// 2. Make wait/record work with CPU streams.
// 3. Add checks for waiting on un-recorded event.
class CudaEventWrapper {
 public:
  CudaEventWrapper()
      : event_(std::make_shared<CudaEvent>(
            hipEventDisableTiming | hipEventBlockingSync)) {}

  void wait() {
    event_->wait();
  }

  void wait(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      scheduler::enqueue(s, [*this]() mutable {
        check_recorded();
        event_->wait();
      });
    } else {
      check_recorded();
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->wait(encoder.stream());
    }
  }

  void record(Stream s) {
    if (s.device == mlx::core::Device::cpu) {
      throw std::runtime_error("CudaEvent can not wait on CPU stream.");
    } else {
      auto& encoder = cu::get_command_encoder(s);
      encoder.commit();
      event_->record(encoder.stream());
      recorded_ = true;
    }
  }

  bool is_signaled() const {
    return recorded_ && event_->completed();
  }

 private:
  void check_recorded() const {
    if (!recorded_) {
      throw std::runtime_error(
          "Should not wait on a CudaEvent before recording.");
    }
  }

  std::shared_ptr<CudaEvent> event_;
  bool recorded_{false};
};

///////////////////////////////////////////////////////////////////////////////
// SharedEvent implementations
///////////////////////////////////////////////////////////////////////////////

__host__ __device__ void event_wait(SharedEvent::Atomic* ac, uint64_t value) {
  uint64_t current;
  while ((current = ac->load()) < value) {
    ac->wait(current);
  }
}

__host__ __device__ void event_signal(SharedEvent::Atomic* ac, uint64_t value) {
  ac->store(value);
  ac->notify_all();
}

__global__ void event_wait_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_wait(ac, value);
}

__global__ void event_signal_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_signal(ac, value);
}

SharedEvent::SharedEvent() {
  buf_ = std::shared_ptr<Buffer>(
      new Buffer{allocator().malloc(sizeof(Atomic))}, [](Buffer* ptr) {
        allocator().free(*ptr);
        delete ptr;
      });
  *static_cast<uint64_t*>(buf_->raw_ptr()) = 0;
}

void SharedEvent::wait(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait");
  event_wait(atomic(), value);
}

void SharedEvent::wait(hipStream_t stream, uint64_t value) {
  event_wait_kernel<<<1, 1, 0, stream>>>(atomic(), value);
}

void SharedEvent::wait(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { wait(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    wait(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

void SharedEvent::signal(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal");
  event_signal(atomic(), value);
}

void SharedEvent::signal(hipStream_t stream, uint64_t value) {
  event_signal_kernel<<<1, 1, 0, stream>>>(atomic(), value);
}

void SharedEvent::signal(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal(s)");
  if (s.device == mlx::core::Device::cpu) {
    // Signal through a GPU stream so the atomic is updated in GPU - updating
    // the atomic in CPU sometimes does not get GPU notified.
    static CudaStream stream(device(mlx::core::Device::gpu));
    scheduler::enqueue(s, [*this, value]() mutable { signal(stream, value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    signal(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

bool SharedEvent::is_signaled(uint64_t value) const {
  nvtx3::scoped_range r("cu::SharedEvent::is_signaled");
  return atomic()->load() >= value;
}

uint64_t SharedEvent::value() const {
  nvtx3::scoped_range r("cu::SharedEvent::value");
  return atomic()->load();
}

} // namespace cu

///////////////////////////////////////////////////////////////////////////////
// Event implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

struct EventImpl {
  // CudaEvent is preferred when possible because it is fast, however we have
  // to fallback to SharedEvent in following cases:
  // 1. the event is used to wait/signal a cpu stream;
  // 2. signal value other than 1 has been specified.
  std::unique_ptr<cu::CudaEventWrapper> cuda;
  std::unique_ptr<cu::SharedEvent> shared;

  bool is_created() const {
    return cuda || shared;
  }

  void ensure_created(Stream s, uint64_t signal_value) {
    if (is_created()) {
      return;
    }
    if (s.device == mlx::core::Device::cpu || signal_value > 1) {
      nvtx3::mark("Using slow SharedEvent");
      shared = std::make_unique<cu::SharedEvent>();
    } else {
      cuda = std::make_unique<cu::CudaEventWrapper>();
    }
  }
};

} // namespace

Event::Event(Stream s) : stream_(s) {
  event_ = std::shared_ptr<void>(
      new EventImpl(), [](void* ptr) { delete static_cast<EventImpl*>(ptr); });
}

void Event::wait() {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait();
  } else {
    event->shared->wait(value());
  }
}

void Event::wait(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait(s);
  } else {
    event->shared->wait(s, value());
  }
}

void Event::signal(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  event->ensure_created(s, value());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->record(s);
  } else {
    event->shared->signal(s, value());
  }
}

bool Event::is_signaled() const {
  auto* event = static_cast<EventImpl*>(event_.get());
  if (!event->is_created()) {
    return false;
  }
  if (event->cuda) {
    assert(value() == 1);
    return event->cuda->is_signaled();
  } else {
    return event->shared->is_signaled(value());
  }
}

} // namespace mlx::core
