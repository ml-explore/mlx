#include "hip/hip_runtime.h"
// Copyright © 2024 Apple Inc.

#include "mlx/backend/cuda/allocator.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/event.h"
#include "mlx/backend/cuda/utils.h"
#include "mlx/event.h"
#include "mlx/scheduler.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

///////////////////////////////////////////////////////////////////////////////
// CudaEvent implementations
///////////////////////////////////////////////////////////////////////////////

// Cuda event managed with RAII.
class CudaEventHandle {
 public:
  CudaEventHandle() {
    CHECK_CUDA_ERROR(hipEventCreateWithFlags(
        &event_, hipEventDisableTiming | hipEventBlockingSync));
  }

  ~CudaEventHandle() {
    CHECK_CUDA_ERROR(hipEventDestroy(event_));
  }

  CudaEventHandle(const CudaEventHandle&) = delete;
  CudaEventHandle& operator=(const CudaEventHandle&) = delete;

  operator hipEvent_t() const {
    return event_;
  }

 private:
  hipEvent_t event_;
};

CudaEvent::CudaEvent() : event_(std::make_shared<CudaEventHandle>()) {}

void CudaEvent::wait() {
  nvtx3::scoped_range r("cu::CudaEvent::wait");
  if (!recorded_) {
    throw std::runtime_error("Should not wait on a CudaEvent before record.");
  }
  hipEventSynchronize(*event_);
}

void CudaEvent::wait(hipStream_t stream) {
  if (!recorded_) {
    throw std::runtime_error("Should not wait on a CudaEvent before record.");
  }
  hipStreamWaitEvent(stream, *event_);
}

void CudaEvent::wait(Stream s) {
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this]() mutable { wait(); });
  } else {
    auto& enc = cu::get_command_encoder(s);
    enc.commit();
    wait(enc.stream());
  }
}

void CudaEvent::record(hipStream_t stream) {
  hipEventRecord(*event_, stream);
  recorded_ = true;
}

void CudaEvent::record(Stream s) {
  if (s.device == mlx::core::Device::cpu) {
    throw std::runtime_error("CudaEvent can not wait on cpu stream.");
  } else {
    auto& enc = cu::get_command_encoder(s);
    enc.commit();
    record(enc.stream());
  }
}

bool CudaEvent::completed() const {
  return hipEventQuery(*event_) == hipSuccess;
}

///////////////////////////////////////////////////////////////////////////////
// SharedEvent implementations
///////////////////////////////////////////////////////////////////////////////

__host__ __device__ void event_wait(SharedEvent::Atomic* ac, uint64_t value) {
  uint64_t current;
  while ((current = ac->load()) < value) {
    ac->wait(current);
  }
}

__host__ __device__ void event_signal(SharedEvent::Atomic* ac, uint64_t value) {
  ac->store(value);
  ac->notify_all();
}

__global__ void event_wait_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_wait(ac, value);
}

__global__ void event_signal_kernel(SharedEvent::Atomic* ac, uint64_t value) {
  event_signal(ac, value);
}

SharedEvent::Atomic* to_atomic(std::shared_ptr<Buffer> buf) {
  return static_cast<SharedEvent::Atomic*>(buf->raw_ptr());
}

SharedEvent::SharedEvent() {
  buf_ = std::shared_ptr<Buffer>(
      new Buffer{allocator().malloc(sizeof(Atomic))}, [](Buffer* ptr) {
        allocator().free(*ptr);
        delete ptr;
      });
  *static_cast<uint64_t*>(buf_->raw_ptr()) = 0;
}

void SharedEvent::wait(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait");
  event_wait(to_atomic(buf_), value);
}

void SharedEvent::wait(hipStream_t stream, uint64_t value) {
  event_wait_kernel<<<1, 1, 0, stream>>>(to_atomic(buf_), value);
}

void SharedEvent::wait(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::wait(s)");
  if (s.device == mlx::core::Device::cpu) {
    scheduler::enqueue(s, [*this, value]() mutable { wait(value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    wait(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

void SharedEvent::signal(uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal");
  event_signal(to_atomic(buf_), value);
}

void SharedEvent::signal(hipStream_t stream, uint64_t value) {
  event_signal_kernel<<<1, 1, 0, stream>>>(to_atomic(buf_), value);
}

void SharedEvent::signal(Stream s, uint64_t value) {
  nvtx3::scoped_range r("cu::SharedEvent::signal(s)");
  if (s.device == mlx::core::Device::cpu) {
    // Signal through a GPU stream so the atomic is updated in GPU - updating
    // the atomic in CPU sometimes does not get GPU notified.
    static CudaStream stream(device(mlx::core::Device::gpu));
    scheduler::enqueue(s, [*this, value]() mutable { signal(stream, value); });
  } else {
    auto& encoder = get_command_encoder(s);
    encoder.commit();
    signal(encoder.stream(), value);
    encoder.add_completed_handler([buf = buf_]() {});
  }
}

bool SharedEvent::is_signaled(uint64_t value) const {
  nvtx3::scoped_range r("cu::SharedEvent::is_signaled");
  return to_atomic(buf_)->load() >= value;
}

uint64_t SharedEvent::value() const {
  nvtx3::scoped_range r("cu::SharedEvent::value");
  return to_atomic(buf_)->load();
}

} // namespace cu

///////////////////////////////////////////////////////////////////////////////
// Event implementations
///////////////////////////////////////////////////////////////////////////////

namespace {

struct EventImpl {
  // CudaEvent is preferred when possible because it is fast, however we have
  // to fallback to SharedEvent in following cases:
  // 1. the event is used to wait/signal a cpu stream;
  // 2. signal value other than 1 has been specified.
  std::unique_ptr<cu::CudaEvent> cuda;
  std::unique_ptr<cu::SharedEvent> shared;

  bool is_created() const {
    return cuda || shared;
  }

  void ensure_created(Stream s, uint64_t signal_value) {
    if (is_created()) {
      return;
    }
    if (s.device == mlx::core::Device::cpu || signal_value > 1) {
      nvtx3::mark("Using slow SharedEvent");
      shared = std::make_unique<cu::SharedEvent>();
    } else {
      cuda = std::make_unique<cu::CudaEvent>();
    }
  }
};

} // namespace

Event::Event(Stream s) : stream_(s) {
  event_ = std::shared_ptr<void>(
      new EventImpl(), [](void* ptr) { delete static_cast<EventImpl*>(ptr); });
}

void Event::wait() {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait();
  } else {
    event->shared->wait(value());
  }
}

void Event::wait(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  assert(event->is_created());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->wait(s);
  } else {
    event->shared->wait(s, value());
  }
}

void Event::signal(Stream s) {
  auto* event = static_cast<EventImpl*>(event_.get());
  event->ensure_created(s, value());
  if (event->cuda) {
    assert(value() == 1);
    event->cuda->record(s);
  } else {
    event->shared->signal(s, value());
  }
}

bool Event::is_signaled() const {
  auto* event = static_cast<EventImpl*>(event_.get());
  if (!event->is_created()) {
    return false;
  }
  if (event->cuda) {
    assert(value() == 1);
    return event->cuda->recorded() && event->cuda->completed();
  } else {
    return event->shared->is_signaled(value());
  }
}

} // namespace mlx::core
