#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/iterators/strided_iterator.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/reduce/reduce.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"

#include <hip/hip_cooperative_groups.h>
#include <nvtx3/nvtx3.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>

namespace mlx::core {

namespace cu {

namespace cg = cooperative_groups;

inline __device__ float2 plus(const float2& a, const float2& b) {
  return float2{a.x + b.x, a.y + b.y};
}

template <typename T, int BLOCK_DIM, int N_READS = 4>
__global__ void layer_norm(
    const T* x,
    const T* w,
    const T* b,
    T* out,
    float eps,
    uint32_t axis_size,
    uint32_t w_stride,
    uint32_t b_stride) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();

  x += grid.block_rank() * axis_size;
  out += grid.block_rank() * axis_size;

  float2 sum = {};
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    float xn[N_READS] = {};
    hipcub::LoadDirectBlocked(index, x, xn, axis_size);
    for (int i = 0; i < N_READS; i++) {
      float xi = xn[i];
      sum = plus(sum, float2{xi, xi * xi});
    }
  }

  using BlockReduceT = hipcub::BlockReduce<float2, BLOCK_DIM>;
  __shared__ typename BlockReduceT::TempStorage temp;
  sum = BlockReduceT(temp).Reduce(sum, plus);

  __shared__ float local_mean;
  __shared__ float local_normalizer;
  if (block.thread_rank() == 0) {
    float mean = sum.x / axis_size;
    float variance = sum.y / axis_size - mean * mean;
    local_mean = mean;
    local_normalizer = rsqrt(variance + eps);
  }
  block.sync();

  float mean = local_mean;
  float normalizer = local_normalizer;

  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    T xn[N_READS];
    T wn[N_READS];
    T bn[N_READS];
    hipcub::LoadDirectBlocked(index, x, xn, axis_size);
    hipcub::LoadDirectBlocked(index, strided_iterator(w, w_stride), wn, axis_size);
    hipcub::LoadDirectBlocked(index, strided_iterator(b, b_stride), bn, axis_size);
    for (int i = 0; i < N_READS; i++) {
      float norm = (static_cast<float>(xn[i]) - mean) * normalizer;
      xn[i] = wn[i] * static_cast<T>(norm) + bn[i];
    }
    hipcub::StoreDirectBlocked(index, out, xn, axis_size);
  }
}

struct SumVJP {
  float x;
  float x2;
  float wg;
  float wgx;
};

inline __device__ SumVJP plus_vjp(const SumVJP& a, const SumVJP& b) {
  return SumVJP{a.x + b.x, a.x2 + b.x2, a.wg + b.wg, a.wgx + b.wgx};
}

template <typename T, bool HAS_W, int BLOCK_DIM, int N_READS = 4>
__global__ void layer_norm_vjp(
    const T* x,
    const T* w,
    const T* g,
    T* gx,
    T* gw,
    float eps,
    uint32_t axis_size,
    uint32_t w_stride) {
  auto grid = cg::this_grid();
  auto block = cg::this_thread_block();

  x += grid.block_rank() * axis_size;
  g += grid.block_rank() * axis_size;
  gx += grid.block_rank() * axis_size;
  gw += grid.block_rank() * axis_size;

  SumVJP sum = {};
  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    T xn[N_READS] = {};
    T wn[N_READS] = {};
    T gn[N_READS] = {};
    int index = r * BLOCK_DIM + block.thread_rank();
    hipcub::LoadDirectBlocked(index, x, xn, axis_size);
    hipcub::LoadDirectBlocked(index, g, gn, axis_size);
    hipcub::LoadDirectBlocked(index, strided_iterator(w, w_stride), wn, axis_size);
    for (int i = 0; i < N_READS; i++) {
      float xi = xn[i];
      float wi = wn[i];
      float gi = gn[i];
      float wg = wi * gi;
      sum = plus_vjp(sum, SumVJP{xi, xi * xi, wg, wg * xi});
    }
  }

  using BlockReduceT = hipcub::BlockReduce<SumVJP, BLOCK_DIM>;
  __shared__ typename BlockReduceT::TempStorage temp;
  sum = BlockReduceT(temp).Reduce(sum, plus_vjp);

  __shared__ float local_mean;
  __shared__ float local_normalizer;
  __shared__ float local_meanwg;
  __shared__ float local_meanwgx;
  if (block.thread_rank() == 0) {
    float mean = sum.x / axis_size;
    float variance = sum.x2 / axis_size - mean * mean;
    local_mean = mean;
    local_normalizer = rsqrt(variance + eps);
    local_meanwg = sum.wg / axis_size;
    local_meanwgx = sum.wgx / axis_size;
  }
  block.sync();

  float mean = local_mean;
  float normalizer = local_normalizer;
  float meanwg = local_meanwg;
  float meanwgxc = local_meanwgx - meanwg * mean;
  ;
  float normalizer2 = normalizer * normalizer;

  for (int r = 0; r < cuda::ceil_div(axis_size, BLOCK_DIM * N_READS); r++) {
    auto index = r * BLOCK_DIM + block.thread_rank();
    T xn[N_READS];
    T wn[N_READS];
    T gn[N_READS];
    hipcub::LoadDirectBlocked(index, x, xn, axis_size);
    hipcub::LoadDirectBlocked(index, g, gn, axis_size);
    hipcub::LoadDirectBlocked(index, strided_iterator(w, w_stride), wn, axis_size);
    for (int i = 0; i < N_READS; i++) {
      float xi = (static_cast<float>(xn[i]) - mean) * normalizer;
      float wi = wn[i];
      float gi = gn[i];
      xn[i] = normalizer * (wi * gi - meanwg) - xi * meanwgxc * normalizer2;
      if constexpr (HAS_W) {
        wn[i] = gi * xi;
      }
    }
    hipcub::StoreDirectBlocked(index, gx, xn, axis_size);
    if constexpr (HAS_W) {
      hipcub::StoreDirectBlocked(index, gw, wn, axis_size);
    }
  }
}

} // namespace cu

namespace fast {

// TODO: The implementation is similar to backend/metal/normalization.cpp
void LayerNorm::eval_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs) {
  nvtx3::scoped_range r("LayerNorm::eval_gpu");
  auto& s = stream();
  auto& out = outputs[0];

  // Make sure that the last dimension is contiguous.
  auto set_output = [&s, &out](const array& x) {
    bool no_copy = x.flags().contiguous && x.strides()[x.ndim() - 1] == 1;
    if (no_copy && x.ndim() > 1) {
      auto s = x.strides()[x.ndim() - 2];
      no_copy &= (s == 0 || s == x.shape().back());
    }
    if (no_copy) {
      if (x.is_donatable()) {
        out.copy_shared_buffer(x);
      } else {
        out.set_data(
            allocator::malloc(x.data_size() * x.itemsize()),
            x.data_size(),
            x.strides(),
            x.flags());
      }
      return x;
    } else {
      auto x_copy = array(x.shape(), x.dtype(), nullptr, {});
      copy_gpu(x, x_copy, CopyType::General, s);
      out.copy_shared_buffer(x_copy);
      return x_copy;
    }
  };

  array o = set_output(inputs[0]);
  const array& x = o.data_shared_ptr() ? o : out;
  const array& w = inputs[1];
  const array& b = inputs[2];

  uint32_t axis_size = x.shape().back();
  uint32_t n_rows = x.data_size() / axis_size;
  uint32_t w_stride = (w.ndim() == 1) ? w.strides()[0] : 0;
  uint32_t b_stride = (b.ndim() == 1) ? b.strides()[0] : 0;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(x);
  encoder.set_input_array(w);
  encoder.set_input_array(b);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_FLOAT_TYPES_CHECKED(out.dtype(), "layernorm", CTYPE, {
      using DataType = cuda_type_t<CTYPE>;
      constexpr uint32_t N_READS = 4;
      MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
        auto kernel = cu::layer_norm<DataType, BLOCK_DIM, N_READS>;
        kernel<<<n_rows, BLOCK_DIM, 0, stream>>>(
            x.data<DataType>(),
            w.data<DataType>(),
            b.data<DataType>(),
            out.data<DataType>(),
            eps_,
            axis_size,
            w_stride,
            b_stride);
      });
    });
  });
}

void LayerNormVJP::eval_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs) {
  nvtx3::scoped_range r("LayerNormVJP::eval_gpu");
  auto& s = stream();
  auto& encoder = cu::get_command_encoder(s);

  // Ensure row contiguity. We could relax this step by checking that the array
  // is contiguous (no broadcasts or holes) and that the input strides are the
  // same as the cotangent strides but for now this is simpler.
  auto check_input = [&s](const array& x) -> std::pair<array, bool> {
    if (x.flags().row_contiguous) {
      return {x, false};
    }
    array x_copy(x.shape(), x.dtype(), nullptr, {});
    copy_gpu(x, x_copy, CopyType::General, s);
    return {x_copy, true};
  };
  bool donate_x = inputs[0].is_donatable();
  bool donate_g = inputs[3].is_donatable();
  auto [x, copied] = check_input(inputs[0]);
  donate_x |= copied;
  const array& w = inputs[1];
  const array& b = inputs[2];
  auto [g, g_copied] = check_input(inputs[3]);
  donate_g |= g_copied;
  array& gx = outputs[0];
  array& gw = outputs[1];
  array& gb = outputs[2];

  // Check whether we had a weight.
  bool has_w = w.ndim() != 0;

  // Allocate space for the outputs.
  bool g_in_gx = false;
  if (donate_x) {
    gx.copy_shared_buffer(x);
  } else if (donate_g) {
    gx.copy_shared_buffer(g);
    g_in_gx = true;
  } else {
    gx.set_data(allocator::malloc(gx.nbytes()));
  }
  if (g_copied && !g_in_gx) {
    encoder.add_temporary(g);
  }

  uint32_t axis_size = x.shape().back();
  int32_t n_rows = x.data_size() / axis_size;
  uint32_t w_stride = (w.ndim() == 1) ? w.strides()[0] : 0;

  // Allocate a temporary to store the gradients for w and allocate the output
  // gradient accumulators.
  array gw_temp =
      (has_w) ? array({n_rows, x.shape().back()}, gw.dtype(), nullptr, {}) : w;
  if (has_w) {
    if (!g_in_gx && donate_g) {
      gw_temp.copy_shared_buffer(g);
    } else {
      gw_temp.set_data(allocator::malloc(gw_temp.nbytes()));
      encoder.add_temporary(gw_temp);
    }
  }
  gw.set_data(allocator::malloc(gw.nbytes()));
  gb.set_data(allocator::malloc(gb.nbytes()));

  // Finish with the gradient for b in case we had a b.
  if (gb.ndim() == 1 && gb.size() == axis_size) {
    ReductionPlan plan(
        ReductionOpType::ContiguousStridedReduce, {n_rows}, {axis_size});
    col_reduce(encoder, g, gb, Reduce::ReduceType::Sum, {0}, plan);
  }

  encoder.set_input_array(x);
  encoder.set_input_array(w);
  encoder.set_input_array(g);
  encoder.set_output_array(gx);
  encoder.set_output_array(gw_temp);
  encoder.launch_kernel([&, x = x, g = g](hipStream_t stream) {
    MLX_SWITCH_FLOAT_TYPES_CHECKED(gx.dtype(), "layernorm_vjp", CTYPE, {
      using DataType = cuda_type_t<CTYPE>;
      constexpr int N_READS = 4;
      MLX_SWITCH_BOOL(has_w, HAS_W, {
        MLX_SWITCH_BLOCK_DIM(cuda::ceil_div(axis_size, N_READS), BLOCK_DIM, {
          auto kernel = cu::layer_norm_vjp<DataType, HAS_W, BLOCK_DIM, N_READS>;
          kernel<<<n_rows, BLOCK_DIM, 0, stream>>>(
              x.data<DataType>(),
              w.data<DataType>(),
              g.data<DataType>(),
              gx.data<DataType>(),
              gw_temp.data<DataType>(),
              eps_,
              axis_size,
              w_stride);
        });
      });
    });
  });

  if (has_w) {
    ReductionPlan plan(
        ReductionOpType::ContiguousStridedReduce, {n_rows}, {axis_size});
    col_reduce(encoder, gw_temp, gw, Reduce::ReduceType::Sum, {0}, plan);
  }
}

} // namespace fast

} // namespace mlx::core
