// Copyright © 2025 Apple Inc.

#include "mlx/backend/common/binary.h"
#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/iterators/general_iterator.cuh"
#include "mlx/backend/cuda/iterators/repeat_iterator.cuh"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/cuda/kernels/binary_ops.cuh"
#include "mlx/backend/cuda/kernels/cucomplex_math.cuh"
#include "mlx/dtype_utils.h"
#include "mlx/primitives.h"

#include <nvtx3/nvtx3.hpp>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

namespace mlx::core {

namespace cu {

template <typename Op, typename In, typename Out>
constexpr bool supports_binary_op() {
  if (std::is_same_v<Op, Add> || std::is_same_v<Op, Divide> ||
      std::is_same_v<Op, Maximum> || std::is_same_v<Op, Minimum> ||
      std::is_same_v<Op, Multiply> || std::is_same_v<Op, Subtract> ||
      std::is_same_v<Op, Power> || std::is_same_v<Op, Remainder>) {
    return std::is_same_v<In, Out>;
  }
  if (std::is_same_v<Op, Equal> || std::is_same_v<Op, Greater> ||
      std::is_same_v<Op, GreaterEqual> || std::is_same_v<Op, Less> ||
      std::is_same_v<Op, LessEqual> || std::is_same_v<Op, NotEqual>) {
    return std::is_same_v<Out, bool>;
  }
  if (std::is_same_v<Op, LogicalAnd> || std::is_same_v<Op, LogicalOr>) {
    return std::is_same_v<Out, bool> && std::is_same_v<In, bool>;
  }
  if (std::is_same_v<Op, NaNEqual>) {
    return std::is_same_v<Out, bool> &&
        (is_floating_v<In> || std::is_same_v<In, complex64_t>);
  }
  if (std::is_same_v<Op, LogAddExp> || std::is_same_v<Op, ArcTan2>) {
    return std::is_same_v<In, Out> && is_floating_v<In>;
  }
  if (std::is_same_v<Op, BitwiseAnd> || std::is_same_v<Op, BitwiseOr> ||
      std::is_same_v<Op, BitwiseXor>) {
    return std::is_same_v<In, Out> && std::is_integral_v<In>;
  }
  if (std::is_same_v<Op, LeftShift> || std::is_same_v<Op, RightShift>) {
    return std::is_same_v<In, Out> && std::is_integral_v<In> &&
        !std::is_same_v<In, bool>;
  }
  return false;
}

} // namespace cu

template <typename Op>
void binary_op_gpu_inplace(
    const std::vector<array>& inputs,
    std::vector<array>& outputs,
    std::string_view op,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto& out = outputs[0];
  if (out.size() == 0) {
    return;
  }

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(a);
  encoder.set_input_array(b);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_ALL_TYPES(a.dtype(), CTYPE_IN, {
      MLX_SWITCH_ALL_TYPES(out.dtype(), CTYPE_OUT, {
        if constexpr (cu::supports_binary_op<Op, CTYPE_IN, CTYPE_OUT>()) {
          using InType = cuda_type_t<CTYPE_IN>;
          using OutType = cuda_type_t<CTYPE_OUT>;
          auto policy = cu::thrust_policy(stream);
          auto a_ptr = thrust::device_pointer_cast(a.data<InType>());
          auto b_ptr = thrust::device_pointer_cast(b.data<InType>());
          auto out_ptr = thrust::device_pointer_cast(out.data<OutType>());

          auto bopt = get_binary_op_type(a, b);
          if (bopt == BinaryOpType::ScalarScalar) {
            auto a_begin = cu::repeat_iterator(a_ptr);
            auto a_end = a_begin + out.data_size();
            auto b_begin = cu::repeat_iterator(b_ptr);
            thrust::transform(policy, a_begin, a_end, b_begin, out_ptr, Op());
          } else if (bopt == BinaryOpType::ScalarVector) {
            auto a_begin = cu::repeat_iterator(a_ptr);
            auto a_end = a_begin + out.data_size();
            auto b_begin = b_ptr;
            thrust::transform(policy, a_begin, a_end, b_begin, out_ptr, Op());
          } else if (bopt == BinaryOpType::VectorScalar) {
            auto a_begin = a_ptr;
            auto a_end = a_begin + out.data_size();
            auto b_begin = cu::repeat_iterator(b_ptr);
            thrust::transform(policy, a_begin, a_end, b_begin, out_ptr, Op());
          } else if (bopt == BinaryOpType::VectorVector) {
            auto a_begin = a_ptr;
            auto a_end = a_begin + out.data_size();
            auto b_begin = b_ptr;
            thrust::transform(policy, a_begin, a_end, b_begin, out_ptr, Op());
          } else {
            auto [shape, strides] = collapse_contiguous_dims(a, b, out);
            auto [a_begin, a_end] = cu::make_general_iterators<int64_t>(
                a_ptr, out.data_size(), shape, strides[0]);
            auto [b_begin, b_end] = cu::make_general_iterators<int64_t>(
                b_ptr, out.data_size(), shape, strides[1]);
            thrust::transform(policy, a_begin, a_end, b_begin, out_ptr, Op());
          }
        } else {
          throw std::runtime_error(fmt::format(
              "Can not do binary op {} on inputs of {} with result of {}.",
              op,
              dtype_to_string(a.dtype()),
              dtype_to_string(out.dtype())));
        }
      });
    });
  });
}

template <typename Op>
void binary_op_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs,
    std::string_view op,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto bopt = get_binary_op_type(a, b);
  set_binary_op_output_data(a, b, outputs[0], bopt);
  set_binary_op_output_data(a, b, outputs[1], bopt);
  binary_op_gpu_inplace<Op>(inputs, outputs, op, s);
}

template <typename Op>
void binary_op_gpu(
    const std::vector<array>& inputs,
    array& out,
    std::string_view op,
    const Stream& s) {
  auto& a = inputs[0];
  auto& b = inputs[1];
  auto bopt = get_binary_op_type(a, b);
  set_binary_op_output_data(a, b, out, bopt);
  std::vector<array> outputs{out};
  binary_op_gpu_inplace<Op>(inputs, outputs, op, s);
}

#define BINARY_GPU(func)                                                 \
  void func::eval_gpu(const std::vector<array>& inputs, array& out) {    \
    nvtx3::scoped_range r(#func "::eval_gpu");                           \
    auto& s = out.primitive().stream();                                  \
    binary_op_gpu<cu::func>(inputs, out, get_primitive_string(this), s); \
  }

#define BINARY_GPU_MULTI(func)                                               \
  void func::eval_gpu(                                                       \
      const std::vector<array>& inputs, std::vector<array>& outputs) {       \
    nvtx3::scoped_range r(#func "::eval_gpu");                               \
    auto& s = outputs[0].primitive().stream();                               \
    binary_op_gpu<cu::func>(inputs, outputs, get_primitive_string(this), s); \
  }

BINARY_GPU(Add)
BINARY_GPU(ArcTan2)
BINARY_GPU(Divide)
BINARY_GPU(Remainder)
BINARY_GPU(Equal)
BINARY_GPU(Greater)
BINARY_GPU(GreaterEqual)
BINARY_GPU(Less)
BINARY_GPU(LessEqual)
BINARY_GPU(LogicalAnd)
BINARY_GPU(LogicalOr)
BINARY_GPU(LogAddExp)
BINARY_GPU(Maximum)
BINARY_GPU(Minimum)
BINARY_GPU(Multiply)
BINARY_GPU(NotEqual)
BINARY_GPU(Power)
BINARY_GPU(Subtract)

void BitwiseBinary::eval_gpu(const std::vector<array>& inputs, array& out) {
  nvtx3::scoped_range r("BitwiseBinary::eval_gpu");
  auto& s = out.primitive().stream();
  auto op = get_primitive_string(this);
  switch (op_) {
    case BitwiseBinary::And:
      binary_op_gpu<cu::BitwiseAnd>(inputs, out, op, s);
      break;
    case BitwiseBinary::Or:
      binary_op_gpu<cu::BitwiseOr>(inputs, out, op, s);
      break;
    case BitwiseBinary::Xor:
      binary_op_gpu<cu::BitwiseXor>(inputs, out, op, s);
      break;
    case BitwiseBinary::LeftShift:
      binary_op_gpu<cu::LeftShift>(inputs, out, op, s);
      break;
    case BitwiseBinary::RightShift:
      binary_op_gpu<cu::RightShift>(inputs, out, op, s);
      break;
  }
}

} // namespace mlx::core
