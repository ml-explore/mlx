#include "hip/hip_runtime.h"
// Copyright © 2025 Apple Inc.

#include "mlx/backend/cuda/device.h"
#include "mlx/backend/cuda/kernel_utils.cuh"
#include "mlx/backend/gpu/copy.h"
#include "mlx/dtype_utils.h"
#include "mlx/fast_primitives.h"

#include <nvtx3/nvtx3.hpp>

namespace mlx::core {

namespace cu {

template <typename T, bool traditional, bool forward>
__device__ void rope_single_impl(
    const T* in,
    T* out,
    int32_t offset,
    float inv_freq,
    float scale,
    int64_t stride,
    uint2 pos,
    uint2 dims) {
  float L = scale * static_cast<float>(offset);

  // Compute costheta, sintheta
  float theta = L * inv_freq;
  float costheta = cos(theta);
  float sintheta = sin(theta);

  // Compute the input and output indices
  uint index_1, index_2;
  if (traditional) {
    index_1 = 2 * pos.x + pos.y * stride;
    index_2 = index_1 + 1;
  } else {
    index_1 = pos.x + pos.y * stride;
    index_2 = index_1 + dims.x;
  }

  // Read and write the output
  float x1 = static_cast<float>(in[index_1]);
  float x2 = static_cast<float>(in[index_2]);
  float rx1;
  float rx2;
  if (forward) {
    rx1 = x1 * costheta - x2 * sintheta;
    rx2 = x1 * sintheta + x2 * costheta;
  } else {
    rx1 = x2 * sintheta + x1 * costheta;
    rx2 = x2 * costheta - x1 * sintheta;
  }
  out[index_1] = static_cast<T>(rx1);
  out[index_2] = static_cast<T>(rx2);
}

template <typename T, bool traditional, bool forward>
__global__ void rope_single(
    const T* in,
    T* out,
    const int32_t* offset,
    float scale,
    float base,
    int64_t stride,
    uint2 dims) {
  uint2 pos = make_uint2(
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y);
  if (pos.x >= dims.x || pos.y >= dims.y) {
    return;
  }

  float d = static_cast<float>(pos.x) / static_cast<float>(dims.x);
  float inv_freq = exp2(-d * base);
  rope_single_impl<T, traditional, forward>(
      in, out, *offset, inv_freq, scale, stride, pos, dims);
}

template <typename T, bool traditional, bool forward>
__global__ void rope_single_freqs(
    const T* in,
    T* out,
    const int32_t* offset,
    const float* freqs,
    float scale,
    int64_t stride,
    uint2 dims,
    int64_t freq_stride) {
  uint2 pos = make_uint2(
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y);
  if (pos.x >= dims.x || pos.y >= dims.y) {
    return;
  }

  float inv_freq = 1.0 / freqs[freq_stride * pos.x];
  rope_single_impl<T, traditional, forward>(
      in, out, *offset, inv_freq, scale, stride, pos, dims);
}

template <typename T, bool traditional, bool forward, int N = 4>
__device__ void rope_impl(
    const T* in,
    T* out,
    int offset,
    float inv_freq,
    float scale,
    const cuda::std::array<int64_t, 3> strides,
    const cuda::std::array<int64_t, 3> out_strides,
    int64_t n_batch,
    uint3 pos,
    uint3 dims) {
  float L = scale * static_cast<float>(pos.y + offset);

  // Compute costheta, sintheta
  float theta = L * inv_freq;
  float costheta = cos(theta);
  float sintheta = sin(theta);

  // Compute the input and output indices
  size_t in_index_1, in_index_2;
  size_t out_index_1, out_index_2;
  if (traditional) {
    out_index_1 = 2 * pos.x * out_strides[2] + pos.y * out_strides[1] +
        N * pos.z * out_strides[0];
    out_index_2 = out_index_1 + 1;
    in_index_1 =
        2 * pos.x * strides[2] + pos.y * strides[1] + N * pos.z * strides[0];
    in_index_2 = in_index_1 + strides[2];
  } else {
    out_index_1 = pos.x * out_strides[2] + pos.y * out_strides[1] +
        N * pos.z * out_strides[0];
    out_index_2 = out_index_1 + dims.x * out_strides[2];
    in_index_1 =
        pos.x * strides[2] + pos.y * strides[1] + N * pos.z * strides[0];
    in_index_2 = in_index_1 + dims.x * strides[2];
  }
  for (int i = 0; i < N && pos.z * N + i < n_batch; ++i) {
    // Read and write the output
    float x1 = static_cast<float>(in[in_index_1]);
    float x2 = static_cast<float>(in[in_index_2]);
    float rx1;
    float rx2;
    if (forward) {
      rx1 = x1 * costheta - x2 * sintheta;
      rx2 = x1 * sintheta + x2 * costheta;
    } else {
      rx1 = x2 * sintheta + x1 * costheta;
      rx2 = x2 * costheta - x1 * sintheta;
    }
    out[out_index_1] = static_cast<T>(rx1);
    out[out_index_2] = static_cast<T>(rx2);
    in_index_1 += strides[0];
    in_index_2 += strides[0];
    out_index_1 += out_strides[0];
    out_index_2 += out_strides[0];
  }
}

template <typename T, bool traditional, bool forward>
__global__ void rope(
    const T* in,
    T* out,
    const int32_t* offset,
    float scale,
    float base,
    const __grid_constant__ cuda::std::array<int64_t, 3> strides,
    const __grid_constant__ cuda::std::array<int64_t, 3> out_strides,
    int64_t n_batch,
    uint3 dims) {
  uint3 pos = make_uint3(
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y,
      blockIdx.z * blockDim.z + threadIdx.z);
  if (pos.x >= dims.x || pos.y >= dims.y || pos.z >= dims.z) {
    return;
  }

  float d = static_cast<float>(pos.x) / static_cast<float>(dims.x);
  float inv_freq = exp2(-d * base);
  rope_impl<T, traditional, forward>(
      in,
      out,
      *offset,
      inv_freq,
      scale,
      strides,
      out_strides,
      n_batch,
      pos,
      dims);
}

template <typename T, bool traditional, bool forward>
__global__ void rope_freqs(
    const T* in,
    T* out,
    const int32_t* offset,
    const float* freqs,
    float scale,
    float base,
    const __grid_constant__ cuda::std::array<int64_t, 3> strides,
    const __grid_constant__ cuda::std::array<int64_t, 3> out_strides,
    int64_t n_batch,
    uint3 dims,
    int64_t freq_stride) {
  uint3 pos = make_uint3(
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y,
      blockIdx.z * blockDim.z + threadIdx.z);
  if (pos.x >= dims.x || pos.y >= dims.y || pos.z >= dims.z) {
    return;
  }

  float inv_freq = 1.0 / freqs[freq_stride * pos.x];
  rope_impl<T, traditional, forward>(
      in,
      out,
      *offset,
      inv_freq,
      scale,
      strides,
      out_strides,
      n_batch,
      pos,
      dims);
}

} // namespace cu

namespace fast {

bool RoPE::use_fallback(Stream s) {
  return s.device == Device::cpu;
}

void RoPE::eval_gpu(
    const std::vector<array>& inputs,
    std::vector<array>& outputs) {
  nvtx3::scoped_range r("RoPE::eval_gpu");

  auto& s = stream();
  auto& in = inputs[0];
  auto& offset = inputs[1];
  auto& out = outputs[0];

  if (in.ndim() < 3) {
    throw std::runtime_error("[RoPE] Input must have at least 3 dimensions");
  }

  cuda::std::array<int64_t, 3> strides;
  cuda::std::array<int64_t, 3> out_strides;
  bool donated = false;
  int ndim = in.ndim();
  int dispatch_ndim = in.ndim();
  while (in.shape(-dispatch_ndim) == 1 && dispatch_ndim > 3) {
    dispatch_ndim--;
  }
  size_t mat_size = in.shape(-2) * in.shape(-1);

  // We apply rope to less that the whole vector so copy to output and then
  // apply in-place.
  if (dims_ < in.shape(-1)) {
    donated = true;
    auto ctype =
        (in.flags().row_contiguous) ? CopyType::Vector : CopyType::General;
    copy_gpu(in, out, ctype, s);
    strides[0] = mat_size;
    strides[1] = out.strides()[ndim - 2];
    strides[2] = out.strides()[ndim - 1];
  }

  // Either copy or apply in-place
  else if (in.flags().row_contiguous) {
    if (in.is_donatable()) {
      donated = true;
      out.copy_shared_buffer(in);
    } else {
      out.set_data(allocator::malloc(out.nbytes()));
    }
    strides[0] = mat_size;
    strides[1] = in.strides()[ndim - 2];
    strides[2] = in.strides()[ndim - 1];
  } else if (dispatch_ndim == 3) {
    // Handle non-contiguous 3D inputs
    out.set_data(allocator::malloc(out.nbytes()));
    strides[0] = in.strides()[ndim - 3];
    strides[1] = in.strides()[ndim - 2];
    strides[2] = in.strides()[ndim - 1];
  } else {
    // Copy non-contiguous > 3D inputs into the output and treat
    // input as donated
    donated = true;
    copy_gpu(in, out, CopyType::General, s);
    strides[0] = mat_size;
    strides[1] = out.strides()[ndim - 2];
    strides[2] = out.strides()[ndim - 1];
  }
  out_strides[0] = mat_size;
  out_strides[1] = out.strides()[ndim - 2];
  out_strides[2] = out.strides()[ndim - 1];

  // Some flags to help us dispatch below
  bool single = in.flags().row_contiguous && (mat_size == in.shape(-1));
  bool with_freqs = inputs.size() == 3;

  auto& encoder = cu::get_command_encoder(s);
  encoder.set_input_array(donated ? out : in);
  encoder.set_input_array(offset);
  encoder.set_output_array(out);
  encoder.launch_kernel([&](hipStream_t stream) {
    MLX_SWITCH_FLOAT_TYPES_CHECKED(in.dtype(), "rope", CTYPE, {
      using DataType = cuda_type_t<CTYPE>;
      MLX_SWITCH_BOOL(traditional_, TRADITIONAL, {
        MLX_SWITCH_BOOL(forward_, FORWARD, {
          if (single && !with_freqs) {
            auto kernel = cu::rope_single<DataType, TRADITIONAL, FORWARD>;
            uint2 dims = make_uint2(dims_ / 2, in.size() / mat_size);
            auto [grid, block] = get_grid_and_block(dims.x, dims.y, 1);
            kernel<<<grid, block, 0, stream>>>(
                (donated ? out : in).data<DataType>(),
                out.data<DataType>(),
                offset.data<int32_t>(),
                scale_,
                std::log2(base_),
                mat_size,
                dims);
          } else if (single) {
            auto kernel = cu::rope_single_freqs<DataType, TRADITIONAL, FORWARD>;
            uint2 dims = make_uint2(dims_ / 2, in.size() / mat_size);
            auto [grid, block] = get_grid_and_block(dims.x, dims.y, 1);
            kernel<<<grid, block, 0, stream>>>(
                (donated ? out : in).data<DataType>(),
                out.data<DataType>(),
                offset.data<int32_t>(),
                inputs[2].data<float>(),
                scale_,
                mat_size,
                dims,
                inputs[2].strides(0));
          } else if (with_freqs) {
            auto kernel = cu::rope_freqs<DataType, TRADITIONAL, FORWARD>;
            uint3 dims =
                make_uint3(dims_ / 2, in.shape(-2), in.size() / mat_size);
            dims.z = (dims.z + 3) / 4;
            auto [grid, block] = get_grid_and_block(dims.x, dims.y, dims.z);
            kernel<<<grid, block, 0, stream>>>(
                (donated ? out : in).data<DataType>(),
                out.data<DataType>(),
                offset.data<int32_t>(),
                inputs[2].data<float>(),
                scale_,
                std::log2(base_),
                strides,
                out_strides,
                in.size() / mat_size,
                dims,
                inputs[2].strides(0));
          } else {
            auto kernel = cu::rope<DataType, TRADITIONAL, FORWARD>;
            uint3 dims =
                make_uint3(dims_ / 2, in.shape(-2), in.size() / mat_size);
            dims.z = (dims.z + 3) / 4;
            auto [grid, block] = get_grid_and_block(dims.x, dims.y, dims.z);
            kernel<<<grid, block, 0, stream>>>(
                (donated ? out : in).data<DataType>(),
                out.data<DataType>(),
                offset.data<int32_t>(),
                scale_,
                std::log2(base_),
                strides,
                out_strides,
                in.size() / mat_size,
                dims);
          }
        });
      });
    });
  });
}

} // namespace fast

} // namespace mlx::core
